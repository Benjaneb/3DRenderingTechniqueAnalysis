#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define SCREEN_WIDTH 800
#define SCREEN_HEIGHT 500

#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "olcPixelGameEngine.h"
#include "MathUtilities.cuh"
#include "WorldDatatypes.h"

// Global variables

Player g_player = { { 0, 1, 0 }, { 0, { 0, 0, 1 } }, PI * 0.5f };

//one dimensional instead of 2 dimensional because maybe faster
olc::Pixel pixels[SCREEN_HEIGHT * SCREEN_WIDTH];
float depthBuffer[SCREEN_HEIGHT * SCREEN_WIDTH];

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;
std::vector<Light> g_lights;

olc::Sprite* textureAtlas;

class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

public:
	bool OnUserCreate() override
	{
		//textureAtlas = new olc::Sprite("textureAtlas.png");

		Sphere sphere1 = { { 1, 1, 10 }, 4, olc::BLUE };
		g_spheres = { sphere1 };

		Triangle triangle1 = { { { -2, 1, 3 }, { 0, 2, 3 }, { 1, 1.5, 3 } }, { { 0, 0 }, { 1, 1 }, { 0, 1 } } };
		g_triangles = { triangle1 };

		Light sun = { { 0, 13, 0 }, { 255, 255, 190 } };
		g_lights = { sun };

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		//Controlls(fElapsedTime);
		RayTracing();

		return true;
	}

	void RayTracing()
	{
		std::cout << "egg" << std::endl;

		float zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (int y = -SCREEN_HEIGHT * 0.5f; y < SCREEN_HEIGHT * 0.5f; y++)
		{
			for (int x = -SCREEN_WIDTH * 0.5f; x < SCREEN_WIDTH * 0.5f; x++)
			{
				Vec3D v_direction = { x, y, zFar };
				NormalizeVec3D(&v_direction);

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = (SCREEN_HEIGHT - 1) - (y + SCREEN_HEIGHT * 0.5f);

				//clearing the buffers
				pixels[SCREEN_WIDTH * screenY + screenX] = { 0, 0, 0 };
				depthBuffer[SCREEN_WIDTH * screenY + screenX] = INFINITY;

				//RenderGround(g_player.coords, v_direction, screenX, screenY);

				//RenderSpheres(g_player.coords, v_direction, screenX, screenY);

				RenderTriangles(g_player.coords, v_direction, screenX, screenY);

				Draw(x, y, pixels[SCREEN_WIDTH * screenY + screenX]);
			}
		}

		//std::cout << "fuck" << std::endl;

		//DrawSprite(0, 0, textureAtlas);
	}

	olc::Pixel RenderGround(Vec3D v_start, Vec3D v_direction)
	{
		// Ground properties
		float y = 0;
		olc::Pixel color = olc::Pixel(216, 192, 121);
		float renderDistance = 100;

		// Calculation
		Vec3D extendedRay = AddVec3D(v_start, VecScalarMultiplication3D(v_direction, renderDistance));

		if (extendedRay.y < y && g_player.coords.y > y || extendedRay.y > y && g_player.coords.y < y)
			return color;
	}

	olc::Pixel RenderSpheres(Vec3D v_start, Vec3D v_direction)
	{
		Vec3D v_intersection = { 0, 0, 0 };
		float minDistance_RM = 0;
		bool shadow;
		olc::Pixel color;

		for (int i = 0; i < g_spheres.size(); i++)
		{

			//bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, &v_intersection);

			bool intersectionExists = SphereIntersection_RM(g_spheres[i], v_start, v_direction, &v_intersection, &minDistance_RM);

			// Hard shadows
			if (g_spheres[i].luminance > 0)
			{
				for (int i = 0; i < g_lights.size(); i++)
				{
					Vec3D v_offset = SubtractVec3D(v_intersection, g_spheres[i].coords);
					NormalizeVec3D(&v_offset);
					v_offset = VecScalarMultiplication3D(v_offset, 0.05);

					Vec3D v_offsetIntersection = AddVec3D(v_offset, v_intersection);

					Vec3D v_direction = ReturnNormalizedVec3D(SubtractVec3D(g_lights[i].coords, v_intersection));

					shadow = !SphereIntersection_RM(g_spheres[i], v_offsetIntersection, v_direction);
				}
				// Color calculation
				float glowBrightness = 1 / (minDistance_RM + 1);
				color.r = g_spheres[i].color.r * shadow * glowBrightness;
				color.g = g_spheres[i].color.g * shadow * glowBrightness;
				color.b = g_spheres[i].color.b * shadow * glowBrightness;
			}
		}

		return color;
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr)
	{
		float k1 = (v_direction.x != 0) ? (v_direction.y / v_direction.x) : FLT_MAX;
		float k2 = (v_direction.x != 0) ? (v_direction.z / v_direction.x) : FLT_MAX;

		float a = 1 + k1 * k1 + k2 * k2;
		float b = 2 * (v_start.x - sphere.coords.x) + 2 * k1 * (v_start.y - sphere.coords.y) + 2 * k2 * (v_start.z - sphere.coords.z);
		float c = (v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) + (v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) + 
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// There exists no intersections (no real answer)
		if (b * b - 4 * a * c < 0) return false;

		// If there exists an intersection but we don't care *where* the intersection is
		if (v_intersection == nullptr) return true;
		
		// If we do care where the intersection is:
		Vec3D v_alternative1;
		v_alternative1.x = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		v_alternative1.y = k1 * v_alternative1.x + v_start.y;
		v_alternative1.z = k2 * v_alternative1.x + v_start.z;
		v_alternative1.x += v_start.x;

		Vec3D v_alternative2;
		v_alternative2.x = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		v_alternative2.y = k1 * v_alternative2.x + v_start.y;
		v_alternative2.z = k2 * v_alternative2.x + v_start.z;
		v_alternative2.x += v_start.x;

		// Check which intersection is the closest and choose that one
		float dist1 = DistanceSquared3D(v_alternative1, v_start);
		float dist2 = DistanceSquared3D(v_alternative2, v_start);
		Vec3D v_correctHit = (dist1 < dist2) ? v_alternative1 : v_alternative2;

		// Check if the intersection is behind the player. if so, discard it
		float dotProduct = DotProduct3D(v_correctHit, v_start);
		if (dotProduct < 0) return false;

		*v_intersection = v_correctHit;
		return true;
	}

	// Ray marching for spheres
	bool SphereIntersection_RM(Sphere sphere, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr, float* minDistance = nullptr)
	{
		float touchingDistance = 0.01;
		float renderDistance = 10;
		float distanceTravelled = 0;
		float currentMin = INFINITY;

		while (distanceTravelled < renderDistance)
		{
			float distance = Distance3D(v_start, sphere.coords) - sphere.radius;
			currentMin = Min(currentMin, distance); // For glow
			distanceTravelled += distance;
			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distance));

			if (distance <= touchingDistance)
			{
				if (v_intersection != nullptr) *v_intersection = v_start;
				return true;
			}
		}

		if (minDistance != nullptr && sphere.luminance > 0) *minDistance = currentMin;

		return false;
	}

	void RenderTriangles(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		olc::Pixel pixelColor = { 0, 0, 0 };

		bool intersectionExists;
		Vec3D v_intersection = { 0, 0, 0 };
		float depth = 0;

		for (int i = 0; i < g_triangles.size(); i++)
		{
			intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, &v_intersection, &depth, &pixelColor);
		}

		if (intersectionExists && depth < depthBuffer[SCREEN_WIDTH * screenY + screenX])
		{
			pixels[SCREEN_WIDTH * screenY + screenX] = { 255, 255, 255 };// pixelColor;
		}
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr, float* depth = nullptr, olc::Pixel* pixelColor = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if (DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) return false;
		if (DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) return false;
		if (DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0) return false;

		//if we don't care where the intersection is we just return true before setting v_intersection
		if (v_intersection == nullptr)
		{
			return true;
		}

		*v_intersection = v_trianglePlaneIntersection;
		*depth = v_trianglePlaneIntersection.z;

		if (pixelColor != nullptr)
		{
			Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
			Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

			Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

			Matrix3D triangleMatrix =
			{
				v_triangleEdge1,
				v_triangleEdge2,
				v_triangleNormal
			};

			Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

			Vec2D textureCoordinates = { 0, 0 };

			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
			AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

			*pixelColor = textureAtlas->Sample(textureCoordinates.x, textureCoordinates.y);
		}

		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, float f_planeOffset)
	{
		float f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		float f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	bool TriangleIntersection_RM(Triangle triangle, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);
		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		float f_maxDistance = 10;
		float f_totalDistanceTravelled = 0;

		while (f_totalDistanceTravelled < f_maxDistance)
		{
			float f_signedDistanceToPlane = f_trianglePlaneOffset - DotProduct3D(v_start, v_triangleNormal);

			// the start vector projected onto the trianglePlane
			Vec3D vecProjectedOnPlane = AddVec3D(v_start, VecScalarMultiplication3D(v_triangleNormal, f_signedDistanceToPlane));

			float f_distanceToTriangle;

			// if the projectedPoint is inside the triangle then the distance to the triangle is just the distance to the plane
			if (DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[0])) < 0 &&
				DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[1])) < 0 &&
				DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[2])) < 0)
			{
				f_distanceToTriangle = abs(f_signedDistanceToPlane);
			}
			//otherwise, the distance to the triangle is the distance to the closest edge of the triangle
			else
			{
				float distanceToEdge1 = DistanceToEdge(v_start, triangle.vertices[1], triangle.vertices[0]);
				float distanceToEdge2 = DistanceToEdge(v_start, triangle.vertices[2], triangle.vertices[1]);
				float distanceToEdge3 = DistanceToEdge(v_start, triangle.vertices[0], triangle.vertices[2]);

				float minDistance = distanceToEdge1;

				if (distanceToEdge2 < minDistance) minDistance = distanceToEdge2;
				if (distanceToEdge3 < minDistance) minDistance = distanceToEdge3;

				f_distanceToTriangle = minDistance;
			}
			
			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, f_distanceToTriangle));

			if (f_distanceToTriangle < 0.01)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;

				return true;
			}
			
			f_totalDistanceTravelled += f_distanceToTriangle;
		}

		return false;
	}

	float DistanceToEdge(Vec3D v_point, Vec3D v_vertex1, Vec3D v_vertex2)
	{
		Vec3D v_edgeDirection = SubtractVec3D(v_vertex2, v_vertex1);
		NormalizeVec3D(&v_edgeDirection);
		float f_edgeLength = Distance3D(v_vertex1, v_vertex2);

		float f_projectedPointOnEdgelength = DotProduct3D(SubtractVec3D(v_point, v_vertex1), v_edgeDirection);

		Vec3D v_closestPoint = VecScalarMultiplication3D(v_edgeDirection, Clamp(f_projectedPointOnEdgelength, 0, f_edgeLength));

		return Distance3D(v_point, v_closestPoint);
	}
};

int main()
{
	Engine rayTracer;
	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}