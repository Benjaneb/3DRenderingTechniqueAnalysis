#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define SCREEN_WIDTH 1280
#define SCREEN_HEIGHT 720
#define RENDER_DISTANCE 50
#define TOUCHING_DISTANCE 0.01f

#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "olcPixelGameEngine.h"
#include "MathUtilities.cuh"
#include "WorldDatatypes.h"

// Global variables

Player g_player = { { 0, 1, 0 }, { 1, { 0, 0, 0 } }, TAU * 0.25f };

olc::Pixel g_pixels[SCREEN_HEIGHT * SCREEN_WIDTH]; // Pixel buffer that contains all pixels that'll be drawn on screen
float g_depthBuffer[SCREEN_HEIGHT * SCREEN_WIDTH]; // Contains the most recent distance to the drawn object on every pixel

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;
std::vector<LightSource> g_lights;

olc::Sprite* g_textureAtlas;

enum ControlsType
{
	TYPE_1,
	TYPE_2
};

ControlsType g_controlsType = TYPE_2;

class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

public:
	bool OnUserCreate() override
	{
		g_textureAtlas = new olc::Sprite("../Assets/textureAtlas.png");

		Sphere sphere1 = { { 1, 1, 10 }, 4, olc::BLUE, 2 };
		g_spheres = { sphere1 };

		Triangle triangle1 = {
			{ { -2, 1, 3 }, { 0, 2, 3 }, { 1, 1.5, 3 } },
			{ { 0, 0 }, { 1, 1 }, { 0, 1 } }
		};
		g_triangles = { triangle1 };

		LightSource sun = { { 0, 13, 0 }, { 255, 255, 190 } };
		g_lights = { sun };

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		Controlls(fElapsedTime);
		RayTracing();

		return true;
	}

	void Controlls(float fElapsedTime)
	{
		float movementSpeed = 7 * fElapsedTime;
		float rotationSpeed = 2.5 * fElapsedTime;

		// Movement
		
		if (GetKey(olc::Key::W).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { 0, 0, 1 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}
		
		if (GetKey(olc::Key::A).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { -1, 0, 0 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}
		
		if (GetKey(olc::Key::S).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { 0, 0, -1 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}
		
		if (GetKey(olc::Key::D).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { 1, 0, 0 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}

		if (GetKey(olc::Key::SPACE).bHeld)
		{
			g_player.coords.y += movementSpeed;
		}

		if (GetKey(olc::Key::SHIFT).bHeld)
		{
			g_player.coords.y -= movementSpeed;
		}

		// Rotation
		
		if (GetKey(olc::Key::RIGHT).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion q_newRotationAxis = QuaternionMultiplication(ConjugateQuaternion(g_player.q_orientation), { 0, { 0, 1, 0 } }, g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion(q_newRotationAxis.vecPart, rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}
		
		if (GetKey(olc::Key::LEFT).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion q_newRotationAxis = QuaternionMultiplication(ConjugateQuaternion(g_player.q_orientation), { 0, { 0, 1, 0 } }, g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion(q_newRotationAxis.vecPart, -rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}
		
		if (GetKey(olc::Key::UP).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion({ 1, 0, 0 }, -rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}
		
		if (GetKey(olc::Key::DOWN).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion({ 1, 0, 0 }, rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}
	}

	void RayTracing()
	{
		float zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (int y = -SCREEN_HEIGHT * 0.5f; y < SCREEN_HEIGHT * 0.5f; y++)
		{
			for (int x = -SCREEN_WIDTH * 0.5f; x < SCREEN_WIDTH * 0.5f; x++)
			{
				Vec3D v_direction = { x, y, zFar };
				NormalizeVec3D(&v_direction);

				Vec3D v_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, v_direction }, ConjugateQuaternion(g_player.q_orientation)).vecPart;

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = (SCREEN_HEIGHT - 1) - (y + SCREEN_HEIGHT * 0.5f);

				// Clearing the buffers
				g_pixels[SCREEN_WIDTH * screenY + screenX] = { 137, 250, 255 };
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = INFINITY;

				RenderGround(g_player.coords, v_newDirection, screenX, screenY);

				RenderSpheres(g_player.coords, v_newDirection, screenX, screenY);

				//RenderTriangles(g_player.coords, v_newDirection, screenX, screenY);

				Draw(screenX, screenY, g_pixels[SCREEN_WIDTH * screenY + screenX]);
			}
		}
	}

	void RenderGround(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		olc::Pixel pixelColor = { 0, 0, 0 };

		bool intersectionExists;
		Vec3D v_intersection = { 0, 0, 0 };
		float depth = 0;

		float groundLevel = -1;
		VertexPair2D textureVertexPair = { { { 0, 0 }, { 1, 1 } } };
		float textureScalar = 10;

		intersectionExists = GroundIntersectionRT(groundLevel, textureVertexPair, textureScalar, v_start, v_direction, &v_intersection, &depth, &pixelColor);

		if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
		{
			g_pixels[SCREEN_WIDTH * screenY + screenX] = pixelColor;
			g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
		}
	}

	bool GroundIntersectionRT(float groundLevel, VertexPair2D textureVertexPair, float textureScalar, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr, olc::Pixel* pixelColor = nullptr)
	{
		if (v_direction.y >= 0)
		{
			return false;
		}

		if (v_intersection == nullptr)
		{
			return true;
		}

		ScaleVec3D(&v_direction, (groundLevel - v_start.y) / v_direction.y);

		Vec3D rayGroundIntersection = AddVec3D(v_start, v_direction);

		*v_intersection = rayGroundIntersection;
		*depth = Distance3D(g_player.coords, rayGroundIntersection);

		if (pixelColor == nullptr)
		{
			return true;
		}

		float signedTextureWidth = (textureVertexPair.vertices[1].x - textureVertexPair.vertices[0].x) * textureScalar;
		float signedTextureHeight = (textureVertexPair.vertices[1].y - textureVertexPair.vertices[0].y) * textureScalar;

		float textureX = fmod(rayGroundIntersection.x, signedTextureWidth) / signedTextureWidth;
		float textureY = fmod(rayGroundIntersection.z, signedTextureHeight) / signedTextureHeight;

		// if the textureCoordinates are negative, we need to flip them around the center of the texture and make them positive
		if (textureX < 0) textureX += 1;
		if (textureY < 0) textureY += 1;

		*pixelColor = g_textureAtlas->Sample(textureX + textureVertexPair.vertices[0].x, textureY + textureVertexPair.vertices[0].y);

		return true;
	}

	bool GroundIntersectionRM(float groundLevel, VertexPair2D textureVertexPair, float textureScalar, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr, olc::Pixel* pixelColor = nullptr)
	{
		float totalDistanceTravelled = 0;

		while (totalDistanceTravelled < RENDER_DISTANCE)
		{
			float distanceToGround = abs(v_start.y - groundLevel);

			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distanceToGround));

			if (distanceToGround < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				float signedTextureWidth = (textureVertexPair.vertices[1].x - textureVertexPair.vertices[0].x) * textureScalar;
				float signedTextureHeight = (textureVertexPair.vertices[1].y - textureVertexPair.vertices[0].y) * textureScalar;

				float textureX = fmod(v_start.x, signedTextureWidth) / signedTextureWidth;
				float textureY = fmod(v_start.z, signedTextureHeight) / signedTextureHeight;

				// if the textureCoordinates are negative, we need to flip them around the center of the texture and make them positive
				if (textureX < 0) textureX += 1;
				if (textureY < 0) textureY += 1;

				*pixelColor = g_textureAtlas->Sample(textureX + textureVertexPair.vertices[0].x, textureY + textureVertexPair.vertices[0].y);

				return true;
			}

			totalDistanceTravelled += distanceToGround;
		}

		return false;
	}

	void RenderSpheres(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersection = { 0, 0, 0 };
		float minDistance_RM = 0;
		bool shadow = 1;
		float depth = 0;

		for (int i = 0; i < g_spheres.size(); i++)
		{
			//bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, &v_intersection);

			bool intersectionExists = SphereIntersection_RM(g_spheres[i], v_start, v_direction, &v_intersection, &depth, &minDistance_RM);

			// Hard shadows
			if (g_spheres[i].luminance <= 0)
			{
				for (int i = 0; i < g_lights.size(); i++)
				{
					Vec3D v_offset = SubtractVec3D(v_intersection, g_spheres[i].coords);
					NormalizeVec3D(&v_offset);
					v_offset = VecScalarMultiplication3D(v_offset, 0.05);

					Vec3D v_offsetIntersection = AddVec3D(v_offset, v_intersection);

					Vec3D v_direction = ReturnNormalizedVec3D(SubtractVec3D(g_lights[i].coords, v_intersection));

					shadow = !SphereIntersection_RM(g_spheres[i], v_offsetIntersection, v_direction);
				}
			}

			// No glow
			if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
			{
				olc::Pixel pixelColor;
				pixelColor.r = g_spheres[i].color.r * shadow;
				pixelColor.b = g_spheres[i].color.b * shadow;
				pixelColor.g = g_spheres[i].color.g * shadow;

				g_pixels[SCREEN_WIDTH * screenY + screenX] = pixelColor;
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
			}
			// Glow (mix background color with glow color)
			else if (!intersectionExists && g_spheres[i].luminance > 0)
			{
				olc::Pixel glowColor;

				float glowBrightness = 1 / ((minDistance_RM / g_spheres[i].luminance) + 1);

				glowColor = ColorScalarMultiplication(g_spheres[i].color, glowBrightness);

				AddToColor(&glowColor, g_pixels[SCREEN_WIDTH * screenY + screenX]);

				// Makes sure the background doesn't get darker when the glow decreases
				glowColor = ColorScalarMultiplication(glowColor, 1 / (1 + glowBrightness));

				g_pixels[SCREEN_WIDTH * screenY + screenX] = glowColor;
			}
		}
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction, float* depth = nullptr, Vec3D* v_intersection = nullptr)
	{
		float k1 = (v_direction.x != 0) ? (v_direction.y / v_direction.x) : FLT_MAX;
		float k2 = (v_direction.x != 0) ? (v_direction.z / v_direction.x) : FLT_MAX;

		float a = 1 + k1 * k1 + k2 * k2;
		float b = 2 * (v_start.x - sphere.coords.x) + 2 * k1 * (v_start.y - sphere.coords.y) + 2 * k2 * (v_start.z - sphere.coords.z);
		float c = (v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) + (v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) + 
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// There exists no intersections (no real answer)
		if (b * b - 4 * a * c < 0) return false;

		// If there exists an intersection but we don't care *where* the intersection is
		if (v_intersection == nullptr) return true;
		
		// If we do care where the intersection is:
		Vec3D v_alternative1;
		v_alternative1.x = (-b + sqrt(b * b - 4 * a * c)) / (2 * a);
		v_alternative1.y = k1 * v_alternative1.x + v_start.y;
		v_alternative1.z = k2 * v_alternative1.x + v_start.z;
		v_alternative1.x += v_start.x;

		Vec3D v_alternative2;
		v_alternative2.x = (-b - sqrt(b * b - 4 * a * c)) / (2 * a);
		v_alternative2.y = k1 * v_alternative2.x + v_start.y;
		v_alternative2.z = k2 * v_alternative2.x + v_start.z;
		v_alternative2.x += v_start.x;

		// Check which intersection is the closest and choose that one
		float dist1 = DistanceSquared3D(v_alternative1, v_start);
		float dist2 = DistanceSquared3D(v_alternative2, v_start);
		Vec3D v_correctHit = (dist1 < dist2) ? v_alternative1 : v_alternative2;

		// Check if the intersection is behind the player. if so, discard it
		float dotProduct = DotProduct3D(v_correctHit, v_start);
		if (dotProduct < 0) return false;

		*v_intersection = v_correctHit;
		*depth = Distance3D(g_player.coords, v_correctHit);

		return true;
	}

	// Ray marching for spheres
	bool SphereIntersection_RM(Sphere sphere, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr, float* minDistance = nullptr)
	{
		float distanceTravelled = 0;
		float currentMin = INFINITY;

		while (distanceTravelled < RENDER_DISTANCE)
		{
			float distance = Distance3D(v_start, sphere.coords) - sphere.radius;
			currentMin = Min(currentMin, distance); // For glow
			distanceTravelled += distance;
			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distance));

			if (distance < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr) 
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				return true;
			}
		}

		if (minDistance != nullptr && sphere.luminance > 0) *minDistance = currentMin;

		return false;
	}

	void RenderTriangles(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		olc::Pixel pixelColor = { 0, 0, 0 };

		bool intersectionExists;
		Vec3D v_intersection = { 0, 0, 0 };
		float depth = 0;

		for (int i = 0; i < g_triangles.size(); i++)
		{
			intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, &v_intersection, &depth, &pixelColor);
		}

		if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
		{
			g_pixels[SCREEN_WIDTH * screenY + screenX] = pixelColor;
			g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
		}
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr, float* depth = nullptr, olc::Pixel* pixelColor = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if ((DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) ||
			(DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) ||
			(DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0))
		{
			return false;
		}

		// if we don't care where the intersection is we just return true before setting v_intersection
		if (v_intersection == nullptr)
		{
			return true;
		}

		*v_intersection = v_trianglePlaneIntersection;
		*depth = Distance3D(g_player.coords, v_trianglePlaneIntersection);

		// calculating the texture coordinates
		if (pixelColor == nullptr)
		{
			return true;
		}

		Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
		Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

		Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

		Matrix3D triangleMatrix =
		{
			v_triangleEdge1,
			v_triangleEdge2,
			v_triangleNormal
		};

		Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

		Vec2D textureCoordinates = { 0, 0 };

		AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
		AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
		AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

		*pixelColor = g_textureAtlas->Sample(textureCoordinates.x, textureCoordinates.y);
		
		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, float f_planeOffset)
	{
		float f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		float f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	bool TriangleIntersection_RM(Triangle triangle, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr, float* depth = nullptr, olc::Pixel* pixelColor = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);
		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		float f_totalDistanceTravelled = 0;

		while (f_totalDistanceTravelled < RENDER_DISTANCE)
		{
			float f_signedDistanceToPlane = f_trianglePlaneOffset - DotProduct3D(v_start, v_triangleNormal);

			// the start vector projected onto the trianglePlane
			Vec3D vecProjectedOnPlane = AddVec3D(v_start, VecScalarMultiplication3D(v_triangleNormal, f_signedDistanceToPlane));

			float f_distanceToTriangle;

			// If the projectedPoint is inside the triangle then the distance to the triangle is just the distance to the plane
			if (DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[0])) <= 0 &&
				DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[1])) <= 0 &&
				DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[2])) <= 0)
			{
				f_distanceToTriangle = abs(f_signedDistanceToPlane);
			}
			// Otherwise, the distance to the triangle is the distance to the closest edge of the triangle
			else
			{
				float distanceToEdge1 = DistanceToEdge(v_start, triangle.vertices[1], triangle.vertices[0]);
				float distanceToEdge2 = DistanceToEdge(v_start, triangle.vertices[2], triangle.vertices[1]);
				float distanceToEdge3 = DistanceToEdge(v_start, triangle.vertices[0], triangle.vertices[2]);

				float minDistance = distanceToEdge1;

				if (distanceToEdge2 < minDistance) minDistance = distanceToEdge2;
				if (distanceToEdge3 < minDistance) minDistance = distanceToEdge3;

				f_distanceToTriangle = minDistance;
			}
			
			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, f_distanceToTriangle));

			if (f_distanceToTriangle < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				// Calculating the texture coordinates

				Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
				Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

				Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_start, triangle.vertices[0]);

				Matrix3D triangleMatrix =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					v_triangleNormal
				};

				Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

				Vec2D textureCoordinates = { 0, 0 };

				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
				AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

				*pixelColor = g_textureAtlas->Sample(textureCoordinates.x, textureCoordinates.y);

				return true;
			}
			
			f_totalDistanceTravelled += f_distanceToTriangle;
		}

		return false;
	}

	float DistanceToEdge(Vec3D v_point, Vec3D v_vertex1, Vec3D v_vertex2)
	{
		Vec3D v_edgeDirection = SubtractVec3D(v_vertex2, v_vertex1);
		NormalizeVec3D(&v_edgeDirection);
		float f_edgeLength = Distance3D(v_vertex1, v_vertex2);

		float f_projectedPointOnEdgelength = DotProduct3D(SubtractVec3D(v_point, v_vertex1), v_edgeDirection);

		Vec3D v_closestPoint = VecScalarMultiplication3D(v_edgeDirection, Clamp(f_projectedPointOnEdgelength, 0, f_edgeLength));

		return Distance3D(v_point, v_closestPoint);
	}

	void MixColor(olc::Pixel* color1, olc::Pixel color2)
	{
		color1->r = (color2.r + color1->r) / 2;
		color1->g = (color2.g + color1->g) / 2;
		color1->b = (color2.b + color1->b) / 2;
	}
};

int main()
{
	Engine rayTracer;
	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}