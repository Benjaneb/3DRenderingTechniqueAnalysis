#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define RAY_TRACER
#define PATH_TRACING 1
#define ASYNC 0
#define SCREEN_WIDTH 1000
#define SCREEN_HEIGHT 750
#define RENDER_DISTANCE 50
#define TOUCHING_DISTANCE 0.01f
#define OFFSET_DISTANCE 0.002f
#define MAX_BOUNCES 3
#define SAMPLES_PER_PIXEL 1
#define SAMPLES_PER_RAY 1
#define WHITE_COLOR { 255, 255, 255 }

#include <iostream>
#include <random>
#include <future>
#include <hip/hip_runtime.h>
#include <>
#include "olcPixelGameEngine.h"
#include "MathUtilities.cuh"
#include "WorldDatatypes.h"
#include "ParseOBJ.h"

// Global variables

Player g_player;

Vec3D g_pixels[SCREEN_HEIGHT * SCREEN_WIDTH]; // Pixel buffer that contains all pixels that'll be drawn on screen
float g_depthBuffer[SCREEN_HEIGHT * SCREEN_WIDTH]; // Contains the distance to each point represented by a pixel

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;

Ground g_ground;

olc::Sprite* g_basketball_texture;
olc::Sprite* g_planks_texture;
olc::Sprite* g_concrete_texture;
olc::Sprite* g_tiledfloor_texture;
olc::Sprite* g_worldmap_texture;
olc::Sprite* g_gold_texture;

olc::Sprite* g_basketball_normalmap;
olc::Sprite* g_planks_normalmap;
olc::Sprite* g_concrete_normalmap;
olc::Sprite* g_tiledfloor_normalmap;

std::default_random_engine randEngine;

namespace Options
{
	bool mcControls = false;
}

class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

public:

	bool OnUserCreate() override
	{
		g_player = { { 1.5, 1.5, -2.064 }, { 1, ZERO_VEC3D }, TAU * 0.2f };

		g_basketball_texture = new olc::Sprite("../Assets/basketball.png");
		g_planks_texture = new olc::Sprite("../Assets/planks.png");
		g_concrete_texture = new olc::Sprite("../Assets/concrete.png");
		g_tiledfloor_texture = new olc::Sprite("../Assets/tiledfloor.png");
		g_worldmap_texture = new olc::Sprite("../Assets/worldmap.png");
		g_gold_texture = new olc::Sprite("../Assets/gold.png");

		g_basketball_normalmap = new olc::Sprite("../Assets/basketball_normalmap.png");
		g_planks_normalmap = new olc::Sprite("../Assets/planks_normalmap.png");
		g_concrete_normalmap = new olc::Sprite("../Assets/concrete_normalmap.png");
		g_tiledfloor_normalmap = new olc::Sprite("../Assets/tiledfloor_normalmap.png");

		g_spheres = 
		{
			// Lightsource
			{ { 1.5, 3, 1.5 }, 0.5, { 0.965, 0.795, 0.3333 }, { LAMBERTIAN, 17, 0 } },
			// Glossy ball
			{ { 1.5, 1.4, 1.5 }, 0.4, { 0.965, 0.795, 0.3333 }, { GLOSSY, 0.1, 0.75, 0.05 } },
			// Basket ball
			{ { 2.5, 0.5, 0.8 }, 0.5, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.45 }, g_basketball_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ 1, 0, 1 }), PI / 2), g_basketball_normalmap },
			// World atlas globe
			{ { 1.75, 0.3, 0.5 }, 0.3, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, g_worldmap_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ -1, 0.5, -2 }), PI / 2) },
			// Magenta lightsource
			{ { 0.5, 0.4, 0.8 }, 0.4, { 1, 0.2, 0.4157 }, { LAMBERTIAN, 7.5, 0 } },
			// Another glossy ball
			{ { 1.1, 0.3, 0.4 }, 0.3, { 0.6, 0.8, 0.9 }, { GLOSSY, 0.1, 0.6, 0.2 } }
		};

		g_triangles =
		{
			// Walls first face
			{ { { 0, 0, 3 }, { 0, 3, 3 }, { 3, 3, 3 } }, { 0.8, 1.2, 0.8 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
			{ { { 0, 0, 3 }, { 3, 3, 3 }, { 3, 0, 3 } }, { 0.8, 1.2, 0.8 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } } },
			// Walls second face
			{ { { 0, 0, 0 }, { 0, 3, 0 }, { 0, 3, 3 } }, { 0.8, 1.1, 1.1 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
			{ { { 0, 0, 0 }, { 0, 3, 3 }, { 0, 0, 3 } }, { 0.8, 1.1, 1.1 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } } },
			// Walls third face
			{ { { 3, 0, 3 }, { 3, 3, 3 }, { 3, 3, 0 } }, { 1.1, 0.8, 1.1 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
			{ { { 3, 0, 3 }, { 3, 3, 0 }, { 3, 0, 0 } }, { 1.1, 0.8, 1.1 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } } },
			// Walls fourth face
			{ { { 0, 3, 0 }, { 3, 3, 3 }, { 0, 3, 3 } }, { 1, 1, 1 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
			{ { { 0, 3, 0 }, { 3, 3, 0 }, { 3, 3, 3 } }, { 1, 1, 1 }, STANDARD_MATERIAL, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } } },

			// Box first face
			{ { { 1, 0, 2 }, { 2, 1, 2 }, { 1, 1, 2 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 2 }, { 2, 0, 2 }, { 2, 1, 2 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box second face
			{ { { 1, 0, 1 }, { 1, 1, 1 }, { 2, 1, 1 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 1 }, { 2, 1, 1 }, { 2, 0, 1 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box third face
			{ { { 1, 0, 1 }, { 1, 1, 2 }, { 1, 1, 1 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 1 }, { 1, 0, 2 }, { 1, 1, 2 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box fourth face							   
			{ { { 2, 0, 1 }, { 2, 1, 1 }, { 2, 1, 2 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 2, 0, 1 }, { 2, 1, 2 }, { 2, 0, 2 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box fifth face							   
			{ { { 1, 1, 1 }, { 1, 1, 2 }, { 2, 1, 2 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 1, 1 }, { 2, 1, 2 }, { 2, 1, 1 } }, { 1, 1, 1 }, { LAMBERTIAN, 0.15, 0.3 }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },

			// Lonely pyramid
			{ { { 0.8, 0, 2.8 }, { 0.5, 1.4, 2.5 }, { 0.2, 0, 2.8 } }, { 1, 1, 1 }, STANDARD_MATERIAL, "", g_gold_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
			{ { { 0.2, 0, 2.8 }, { 0.5, 1.4, 2.5 }, { 0.2, 0, 2.2 } }, { 1, 1, 1 }, STANDARD_MATERIAL, "", g_gold_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
			{ { { 0.2, 0, 2.2 }, { 0.5, 1.4, 2.5 }, { 0.8, 0, 2.2 } }, { 1, 1, 1 }, STANDARD_MATERIAL, "", g_gold_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
			{ { { 0.8, 0, 2.2 }, { 0.5, 1.4, 2.5 }, { 0.8, 0, 2.8 } }, { 1, 1, 1 }, STANDARD_MATERIAL, "", g_gold_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } } },
		};

		//ImportScene(&g_triangles, "../Assets/IsakBenjaminMunk.obj", { { { LAMBERTIAN, 0.7, 0.3 }, "Material" }, { { LAMBERTIAN, 0.7, 0.3 }, "Material.001" } }, { 1.5, 0.2, 1.5 });

		//ImportScene(&g_triangles, "../Assets/BananaLow_OBJ.obj", 0.5, { 1, 0, 0 });
#if ASYNC == 1
		std::async(std::launch::async, ImportScene, &g_triangles, "../Assets/RubberDuck.obj", 0.4, Vec3D({ 0.8, 0.5, 0.5 }));
#else
		//ImportScene(&g_triangles, "../Assets/RubberDuck.obj", 0.4, { 0.8, 0.5, 0.5 });
#endif
		g_ground = { 0, { 1, 1, 1 }, { LAMBERTIAN, 0.1, 0.5 }, g_tiledfloor_texture, { 0, 0 }, { 1, 1 }, 1, g_tiledfloor_normalmap };

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		Timer timer("Rendering");
		Controlls(fElapsedTime);

#if ASYNC == 1
		// Screen split up into 4 quadrants running in parallell on seperate threads
		std::async(std::launch::async, &Engine::RayTracing, this, Vec2D({ 0, 0 }), Vec2D({ SCREEN_WIDTH / 2, SCREEN_HEIGHT / 2 }));
		std::async(std::launch::async, &Engine::RayTracing, this, Vec2D({ SCREEN_WIDTH / 2, 0 }), Vec2D({ SCREEN_WIDTH, SCREEN_HEIGHT / 2 }));
		std::async(std::launch::async, &Engine::RayTracing, this, Vec2D({ 0, SCREEN_HEIGHT / 2 }), Vec2D({ SCREEN_WIDTH / 2, SCREEN_HEIGHT }));
		std::async(std::launch::async, &Engine::RayTracing, this, Vec2D({ SCREEN_WIDTH / 2, SCREEN_HEIGHT / 2 }), Vec2D({ SCREEN_WIDTH, SCREEN_HEIGHT }));
#else
		RayTracing({ 0, 0 }, { SCREEN_WIDTH, SCREEN_HEIGHT });
#endif
		return true;
	}

	// Defined in Controlls.h
	void Controlls(float fElapsedTime);

	void RayTracing(Vec2D screenStart, Vec2D screenEnd)
	{
		float zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (float y = screenStart.y - SCREEN_HEIGHT * 0.5f + 0.5f; y < screenEnd.y - SCREEN_HEIGHT * 0.5f + 0.5f; y++)
		{
			for (float x = screenStart.x - SCREEN_WIDTH * 0.5f + 0.5f; x < screenEnd.x - SCREEN_WIDTH * 0.5f + 0.5f; x++)
			{
				Vec3D v_direction = { x, y, zFar };
				NormalizeVec3D(&v_direction);

				Vec3D v_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, v_direction }, QuaternionConjugate(g_player.q_orientation)).vecPart;

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = (SCREEN_HEIGHT - 1) - (y + SCREEN_HEIGHT * 0.5f);

				Vec3D pixelColor = ZERO_VEC3D;

				for (int i = 0; i < SAMPLES_PER_PIXEL; i++)
				{
					// Clearing the buffers
					g_pixels[SCREEN_WIDTH * screenY + screenX] = ZERO_VEC3D;
					g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = INFINITY;

					RenderGround(g_player.coords, v_newDirection, screenX, screenY);

					RenderSpheres(g_player.coords, v_newDirection, screenX, screenY);

					RenderTriangles(g_player.coords, v_newDirection, screenX, screenY);

					AddToVec3D(&pixelColor, g_pixels[SCREEN_WIDTH * screenY + screenX]);
				}

				ScaleVec3D(&pixelColor, 1 / float(SAMPLES_PER_PIXEL));

				pixelColor.x = Min(pixelColor.x, 255.0f);
				pixelColor.y = Min(pixelColor.y, 255.0f);
				pixelColor.z = Min(pixelColor.z, 255.0f);

				Draw(screenX, screenY, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
			std::cout << int((y + SCREEN_HEIGHT * 0.5f) / SCREEN_HEIGHT * 100) << "%" << std::endl;
		}
	}

	void RenderGround(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_intersectionColor = ZERO_VEC3D;
		Vec3D v_surfaceNormal = ZERO_VEC3D;
		float depth = 0;

		bool intersectionExists = GroundIntersection_RT(v_start, v_direction, &v_intersection, &v_intersectionColor, &v_surfaceNormal, &depth);

		if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
		{
			v_intersectionColor = CalculateLighting_PathTracing(
				v_intersectionColor, g_ground.material, v_surfaceNormal, v_direction, v_intersection, 0
			);

			g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
			g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
		}
	}

	bool GroundIntersection_RT(Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Vec3D* v_surfaceNormal = nullptr, float* depth = nullptr)
	{
		if (v_direction.y >= 0 || v_start.y < g_ground.level)
		{
			return false;
		}

		if (v_intersection == nullptr)
		{
			return true;
		}

		ScaleVec3D(&v_direction, (g_ground.level - v_start.y) / v_direction.y);

		Vec3D rayGroundIntersection = AddVec3D(v_start, v_direction);

		Vec3D v_normal = { 0, 1, 0 };

		Vec3D v_offset = VecScalarMultiplication3D(v_normal, OFFSET_DISTANCE);

		AddToVec3D(&rayGroundIntersection, v_offset);

		*v_intersection = rayGroundIntersection;

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, rayGroundIntersection);
		}

		if (v_surfaceNormal != nullptr)
		{
			*v_surfaceNormal = v_normal;
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (g_ground.texture != nullptr || g_ground.normalMap != nullptr)
		{
			float signedTextureWidth = (g_ground.textureCorner2.x - g_ground.textureCorner1.x) * g_ground.textureScalar;
			float signedTextureHeight = (g_ground.textureCorner2.y - g_ground.textureCorner1.y) * g_ground.textureScalar;

			float t1 = fmod(rayGroundIntersection.x, signedTextureWidth) / signedTextureWidth;
			float t2 = fmod(rayGroundIntersection.z, signedTextureHeight) / signedTextureHeight;

			// if the t values are negative, we need to flip them around the center of the texture and make them positive
			if (t1 < 0) t1 += 1;
			if (t2 < 0) t2 += 1;

			float textureX = Lerp(g_ground.textureCorner1.x, g_ground.textureCorner2.x, t1);
			float textureY = Lerp(g_ground.textureCorner1.y, g_ground.textureCorner2.y, t2);

			if (g_ground.texture != nullptr)
			{
				olc::Pixel texelColor = g_ground.texture->Sample(textureX, textureY);

				*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };
			}
			if (g_ground.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = g_ground.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				*v_surfaceNormal = ReturnNormalizedVec3D({ float(normalMapColor.r) * 2 - 255.0f, float(normalMapColor.b) * 2 - 255.0f, float(normalMapColor.g) * 2 - 255.0f });
			}
		}

		// Proof that the ConusProduct is the most useful function

		// Tint the color
		*v_intersectionColor = ConusProduct(*v_intersectionColor, g_ground.tint);

		return true;
	}

	/*bool GroundIntersection_RM(float groundLevel, VertexPair2D textureVertexPair, float textureScalar, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr, Vec3D* pixelColor = nullptr)
	{
		float totalDistanceTravelled = 0;

		while (totalDistanceTravelled < RENDER_DISTANCE)
		{
			float distanceToGround = abs(v_start.y - groundLevel);

			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distanceToGround));

			if (distanceToGround < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				float signedTextureWidth = (textureVertexPair.vertices[1].x - textureVertexPair.vertices[0].x) * textureScalar;
				float signedTextureHeight = (textureVertexPair.vertices[1].y - textureVertexPair.vertices[0].y) * textureScalar;

				float textureX = fmod(v_start.x, signedTextureWidth) / signedTextureWidth;
				float textureY = fmod(v_start.z, signedTextureHeight) / signedTextureHeight;

				// if the textureCoordinates are negative, we need to flip them around the center of the texture and make them positive
				if (textureX < 0) textureX += 1;
				if (textureY < 0) textureY += 1;

				olc::Pixel texelColor = g_ground.texture->Sample(textureX, textureY);

				*pixelColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };

				return true;
			}

			totalDistanceTravelled += distanceToGround;
		}

		return false;
	}*/

	void RenderSpheres(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_intersectionColor = ZERO_VEC3D;
		Vec3D v_surfaceNormal = ZERO_VEC3D;
		float depth = 0;

		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, &v_intersection, &v_intersectionColor, &v_surfaceNormal, &depth);

			//bool intersectionExists = SphereIntersection_RM(g_spheres[i], v_start, v_direction, &v_intersection, &depth);

			if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
			{
				v_intersectionColor = CalculateLighting_PathTracing(
					v_intersectionColor, g_spheres[i].material, v_surfaceNormal, v_direction, v_intersection, 0
				);

				g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
			}
		}
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Vec3D* v_surfaceNormal = nullptr, float* depth = nullptr)
	{
		float dxdz = v_direction.x / v_direction.z;
		float dydz = v_direction.y / v_direction.z;

		float a = dxdz * dxdz + dydz * dydz + 1;
		
		float b = 
			2 * dxdz * (v_start.x - sphere.coords.x) +
			2 * dydz * (v_start.y - sphere.coords.y) +
			2 * (v_start.z - sphere.coords.z);

		float c = 
			(v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) +
			(v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) +
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// ISAK: There wasn't any need to recalculate this multiple times
		float rootContent = b * b - 4 * a * c;

		// There exists no intersections (no real answer)
		if (rootContent < 0) return false;

		float z1 = (-b + sqrt(rootContent)) / (2 * a);
		float z2 = (-b - sqrt(rootContent)) / (2 * a);

		Vec3D v_alternative1 = { z1 * dxdz, z1 * dydz, z1 };
		AddToVec3D(&v_alternative1, v_start);

		Vec3D v_alternative2 = { z2 * dxdz, z2 * dydz, z2 };
		AddToVec3D(&v_alternative2, v_start);

		// Check which intersection is the closest and choose that one
		float dist1 = DistanceSquared3D(v_alternative1, v_start);
		float dist2 = DistanceSquared3D(v_alternative2, v_start);

		Vec3D v_correctHit = (dist1 < dist2) ? v_alternative1 : v_alternative2;

		// Check if the intersection is behind the ray. if so, discard it
		if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0) return false;

		Vec3D v_normal = SubtractVec3D(v_correctHit, sphere.coords);
		NormalizeVec3D(&v_normal);

		// There exists an intersection which is not behind the ray, but we don't care about returning where the intersection was
		if (v_intersection != nullptr)
		{
			Vec3D v_offset = VecScalarMultiplication3D(v_normal, OFFSET_DISTANCE);

			// ISAK: Better to offset the intersection here so we don't have to do it anywere else
			*v_intersection = AddVec3D(v_correctHit, v_offset);
		}

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, v_correctHit);
		}

		if (v_surfaceNormal != nullptr)
		{
			*v_surfaceNormal = v_normal;
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (sphere.texture != nullptr || v_surfaceNormal != nullptr)
		{
			Vec3D i_Hat = { 1, 0, 0 };
			Vec3D j_Hat = { 0, 1, 0 };
			Vec3D k_Hat = { 0, 0, 1 };

			// Rotating axies by sphere rotation quaternion
			i_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, i_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			j_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, j_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			k_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, k_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;

			// Translate normal into new coordinate system
			v_normal = { DotProduct3D(v_normal, i_Hat), DotProduct3D(v_normal, j_Hat), DotProduct3D(v_normal, k_Hat) };

			// UV coordinates
			float u = 0.5 + atan2(v_normal.x, v_normal.z) / TAU;
			float v = 0.5 - asin(v_normal.y) / PI;

			float textureX = Lerp(sphere.textureCorner1.x, sphere.textureCorner2.x, u);
			float textureY = Lerp(sphere.textureCorner1.y, sphere.textureCorner2.y, v);

			if (sphere.texture != nullptr)
			{
				// Interpolating between assigned texture coordinates
				olc::Pixel texelColor = sphere.texture->Sample(textureX, textureY);

				*v_intersectionColor = { (float)texelColor.r, (float)texelColor.g, (float)texelColor.b };
			}
			if (sphere.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = sphere.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ float(normalMapColor.r) * 2 - 255.0f, float(normalMapColor.b) * 2 - 255.0f, float(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the sphere at the intersection point
				Vec3D v_sidewaysTangent = ReturnNormalizedVec3D({ -v_normal.z, 0, v_normal.x });
				Vec3D v_forwardTangent = ReturnNormalizedVec3D({ 0, -v_normal.z, v_normal.y });

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					v_sidewaysTangent,
					v_normal,
					v_forwardTangent
				};

				*v_surfaceNormal = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		// Tint the color
		*v_intersectionColor = ConusProduct(*v_intersectionColor, sphere.tint);

		return true;
	}

	// Ray marching for spheres
	/*bool SphereIntersection_RM(Sphere sphere, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr)
	{
		float distanceTravelled = 0;

		while (distanceTravelled < RENDER_DISTANCE)
		{
			float distance = Distance3D(v_start, sphere.coords) - sphere.radius;

			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distance));
			distanceTravelled += distance;

			if (distance < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr) 
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				return true;
			}
		}

		return false;
	}*/

	void RenderTriangles(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_intersectionColor = ZERO_VEC3D;
		Vec3D v_surfaceNormal = ZERO_VEC3D;
		float depth = 0;

		for (int i = 0; i < g_triangles.size(); i++)
		{
			bool intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, &v_intersection, &v_intersectionColor, &v_surfaceNormal, &depth);

			if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
			{
#if PATH_TRACING == 1
				v_intersectionColor = CalculateLighting_PathTracing(
					v_intersectionColor, g_triangles[i].material, v_surfaceNormal, v_direction, v_intersection, 0
				);
#else

#endif

				g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
			}
		}
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Vec3D* v_surfaceNormal = nullptr, float* depth = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if ((DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) ||
			(DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) ||
			(DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0))
		{
			return false;
		}

		Vec3D v_offset = VecScalarMultiplication3D(v_triangleNormal, OFFSET_DISTANCE);

		AddToVec3D(&v_trianglePlaneIntersection, v_offset);

		if (DotProduct3D(SubtractVec3D(v_trianglePlaneIntersection, v_start), v_direction) < 0)
		{
			return false;
		}

		// if we don't care where the intersection is we just return true before setting v_intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_trianglePlaneIntersection;
		}

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, v_trianglePlaneIntersection);
		}

		if (v_surfaceNormal != nullptr)
		{
			*v_surfaceNormal = v_triangleNormal;
		}
		
		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (triangle.texture != nullptr || triangle.normalMap != nullptr)
		{
			// from here on we calculate the texture coordinates

			Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

			Matrix3D triangleMatrix =
			{
				v_triangleEdge1,
				v_triangleEdge2,
				v_triangleNormal
			};

			Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

			Vec2D textureCoordinates = { 0, 0 };

			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]), triangleEdgeScalars.x));
			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]), triangleEdgeScalars.y));
			AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

			if (triangle.texture != nullptr)
			{
				olc::Pixel texelColor = triangle.texture->Sample(textureCoordinates.x, textureCoordinates.y);

				*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };
			}
			if (triangle.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = triangle.normalMap->Sample(textureCoordinates.x, textureCoordinates.y);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ float(normalMapColor.r) * 2 - 255.0f, float(normalMapColor.b) * 2 - 255.0f, float(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the triangle for finding the normal in object space

				// { u1, v1 }, { u2, v2 }, { u3, v3 } are the normalMapVertices
				// T is the tangent
				// B is the bitangent
				
				//                       | T.x  B.x  0 |   
				// { v_triangleEdge1 } = | T.y  B.y  0 | * { u2 - u1, v2 - v1, 0 }
				//                       | T.z  B.z  0 |   

				//                       | T.x  B.x  0 |   
				// { v_triangleEdge2 } = | T.y  B.y  0 | * { u3 - u1, v3 - v1, 0 }
				//                       | T.z  B.z  0 |   

				// | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | T.x  B.x  0 |   | u2 - u1  u3 - u1  0 |
				// | v_triangleEdge1.y  v_triangleEdge2.y  0 | = | T.y  B.y  0 | * | v2 - v1  v3 - v1  0 |
				// | v_triangleEdge1.z  v_triangleEdge2.z  0 |   | T.z  B.z  0 |   |    0        0     1 |

				//                                                                                       -1
				// | T.x  B.x  0 |   | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | u2 - u1  u3 - u1  0 |
				// | T.y  B.y  0 | = | v_triangleEdge1.y  v_triangleEdge2.y  0 | * | v2 - v1  v3 - v1  0 |
				// | T.z  B.z  0 |	 | v_triangleEdge1.z  v_triangleEdge2.z  0 |   |    0        0     1 |

				Matrix3D m1 =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					ZERO_VEC3D
				};

				Matrix3D m2 =
				{
					{ triangle.textureVertices[1].x - triangle.textureVertices[0].x, triangle.textureVertices[1].y - triangle.textureVertices[0].y, 0 },
					{ triangle.textureVertices[2].x - triangle.textureVertices[0].x, triangle.textureVertices[2].y - triangle.textureVertices[0].y, 0 },
					{ 0, 0, 1 }
				};

				Matrix3D tangentsMatrix = MatrixMultiplication3D(InverseMatrix3D(m2), m1);

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					ReturnNormalizedVec3D(tangentsMatrix.i_Hat),
					v_triangleNormal,
					ReturnNormalizedVec3D(tangentsMatrix.j_Hat)
				};

				*v_surfaceNormal = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		// Tint the color
		*v_intersectionColor = ConusProduct(*v_intersectionColor, triangle.tint);
		
		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, float f_planeOffset)
	{
		float f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		float f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	/*bool TriangleIntersection_RM(Triangle triangle, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr, float* depth = nullptr, Vec3D* pixelColor = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);
		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		float f_totalDistanceTravelled = 0;

		while (f_totalDistanceTravelled < RENDER_DISTANCE)
		{
			float f_signedDistanceToPlane = f_trianglePlaneOffset - DotProduct3D(v_start, v_triangleNormal);

			// the start vector projected onto the trianglePlane
			Vec3D vecProjectedOnPlane = AddVec3D(v_start, VecScalarMultiplication3D(v_triangleNormal, f_signedDistanceToPlane));

			float f_distanceToTriangle;

			// If the projectedPoint is inside the triangle then the distance to the triangle is just the distance to the plane
			if (DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[0])) <= 0 &&
				DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[1])) <= 0 &&
				DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[2])) <= 0)
			{
				f_distanceToTriangle = abs(f_signedDistanceToPlane);
			}
			// Otherwise, the distance to the triangle is the distance to the closest edge of the triangle
			else
			{
				float distanceToEdge1 = DistanceToEdge(v_start, triangle.vertices[1], triangle.vertices[0]);
				float distanceToEdge2 = DistanceToEdge(v_start, triangle.vertices[2], triangle.vertices[1]);
				float distanceToEdge3 = DistanceToEdge(v_start, triangle.vertices[0], triangle.vertices[2]);

				float minDistance = distanceToEdge1;

				if (distanceToEdge2 < minDistance) minDistance = distanceToEdge2;
				if (distanceToEdge3 < minDistance) minDistance = distanceToEdge3;

				f_distanceToTriangle = minDistance;
			}
			
			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, f_distanceToTriangle));

			if (f_distanceToTriangle < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				// Calculating the texture coordinates

				Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
				Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

				Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_start, triangle.vertices[0]);

				Matrix3D triangleMatrix =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					v_triangleNormal
				};

				Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

				Vec2D textureCoordinates = { 0, 0 };

				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
				AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

				olc::Pixel texelColor = triangle.texture->Sample(textureCoordinates.x, textureCoordinates.y);

				*pixelColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };

				return true;
			}
			
			f_totalDistanceTravelled += f_distanceToTriangle;
		}

		return false;
	}

	float DistanceToEdge(Vec3D v_point, Vec3D v_vertex1, Vec3D v_vertex2)
	{
		Vec3D v_edgeDirection = SubtractVec3D(v_vertex2, v_vertex1);
		NormalizeVec3D(&v_edgeDirection);
		float f_edgeLength = Distance3D(v_vertex1, v_vertex2);

		float f_projectedPointOnEdgelength = DotProduct3D(SubtractVec3D(v_point, v_vertex1), v_edgeDirection);

		Vec3D v_closestPoint = VecScalarMultiplication3D(v_edgeDirection, Clamp(f_projectedPointOnEdgelength, 0, f_edgeLength));

		return Distance3D(v_point, v_closestPoint);
	}*/

	Vec3D CalculateLighting_PathTracing(Vec3D v_objectColor, Material material, Vec3D v_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, int i_bounceCount)
	{
		Vec3D v_outgoingLightColor = VecScalarMultiplication3D(v_objectColor, material.emittance);

		if (i_bounceCount > MAX_BOUNCES)
		{
			return v_outgoingLightColor;
		}

		Vec3D v_outgoingDirection;

		if (material.materialType == LAMBERTIAN)
		{
			float randX = int64_t(randEngine()) - int64_t(randEngine.max()) / 2;
			float randY = int64_t(randEngine()) - int64_t(randEngine.max()) / 2;
			float randZ = int64_t(randEngine()) - int64_t(randEngine.max()) / 2;

			v_outgoingDirection = ReturnNormalizedVec3D({ randX, randY, randZ });

			if (DotProduct3D(v_outgoingDirection, v_surfaceNormal) < 0)
			{
				// The vector is in the wrong hemisphere, so we flip it
				ScaleVec3D(&v_outgoingDirection, -1);
			}
		}
		else if (material.materialType == GLOSSY)
		{
			// Law of reflection for specular surfaces
			v_outgoingDirection = SubtractVec3D(v_incomingDirection, VecScalarMultiplication3D(v_surfaceNormal, 2 * DotProduct3D(v_incomingDirection, v_surfaceNormal)));

			float randX = float(int64_t(randEngine()) - int64_t(randEngine.max()) / 2) / float(int64_t(randEngine.max()) / 2) * material.roughness;
			float randY = float(int64_t(randEngine()) - int64_t(randEngine.max()) / 2) / float(int64_t(randEngine.max()) / 2) * material.roughness;
			float randZ = float(int64_t(randEngine()) - int64_t(randEngine.max()) / 2) / float(int64_t(randEngine.max()) / 2) * material.roughness;

			Vec3D randomOffset = { randX, randY, randZ };

			AddToVec3D(&v_outgoingDirection, randomOffset);
		}

		Vec3D v_nextIntersection = ZERO_VEC3D;
		Vec3D v_intersectionColor = ZERO_VEC3D;
		Vec3D v_normal = ZERO_VEC3D;

		// Checking for an intersection with any of the spheres

		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_intersection, v_outgoingDirection, &v_nextIntersection, &v_intersectionColor, &v_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_intersection, v_outgoingDirection, v_nextIntersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				Vec3D v_incomingLightColor = CalculateLighting_PathTracing(
					v_intersectionColor, g_spheres[i].material, v_normal, v_outgoingDirection, v_nextIntersection, i_bounceCount + 1
				);

				AddToVec3D(
					&v_outgoingLightColor,
					VecScalarMultiplication3D(v_incomingLightColor, 2 * material.reflectance * DotProduct3D(v_surfaceNormal, v_outgoingDirection))
				);

				return v_outgoingLightColor;
			}
		}

		// Checking for an intersection with any of the triangles

		for (int i = 0; i < g_triangles.size(); i++)
		{
			bool intersectionExists = TriangleIntersection_RT(g_triangles[i], v_intersection, v_outgoingDirection, &v_nextIntersection, &v_intersectionColor, &v_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_intersection, v_outgoingDirection, v_nextIntersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				Vec3D v_incomingLightColor = CalculateLighting_PathTracing(
					v_intersectionColor, g_triangles[i].material, v_normal, v_outgoingDirection, v_nextIntersection, i_bounceCount + 1
				);

				AddToVec3D(
					&v_outgoingLightColor,
					VecScalarMultiplication3D(v_incomingLightColor, 2 * material.reflectance * DotProduct3D(v_surfaceNormal, v_outgoingDirection))
				);

				return v_outgoingLightColor;
			}
		}

		// Checking for an intersection with the ground

		bool intersectionExists = GroundIntersection_RT(v_intersection, v_outgoingDirection, &v_nextIntersection, &v_intersectionColor, &v_normal);

		bool b_rayIsBlocked = false;

		if (intersectionExists)
		{
			b_rayIsBlocked = IsRayBlocked(v_intersection, v_outgoingDirection, v_nextIntersection);
		}

		if (intersectionExists && b_rayIsBlocked == false)
		{
			Vec3D v_incomingLightColor = CalculateLighting_PathTracing(
				v_intersectionColor, g_ground.material, v_normal, v_outgoingDirection, v_nextIntersection, i_bounceCount + 1
			);

			AddToVec3D(
				&v_outgoingLightColor,
				VecScalarMultiplication3D(v_incomingLightColor, 2 * material.reflectance * DotProduct3D(v_surfaceNormal, v_outgoingDirection))
			);

			return v_outgoingLightColor;
		}

		// Nothing was hit
		return v_outgoingLightColor;
	}

	Vec3D CalculateLightning_DistributionTracing(Vec3D v_objectColor, Material material, Vec3D v_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, int i_bounceCount)
	{

	}

	bool IsRayBlocked(Vec3D v_start, Vec3D v_direction, Vec3D v_intersection)
	{
		Vec3D v_otherIntersection = ZERO_VEC3D;

		for (int j = 0; j < g_spheres.size(); j++)
		{
			bool otherIntersectionExists = SphereIntersection_RT(g_spheres[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		for (int j = 0; j < g_triangles.size(); j++)
		{
			bool otherIntersectionExists = TriangleIntersection_RT(g_triangles[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		bool otherIntersectionExists = GroundIntersection_RT(v_start, v_direction, &v_otherIntersection);

		// If there exists a closer intersection to the ray start vector it means the ray is blocked
		if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
		{
			return true;
		}

		// The ray is not blocked
		return false;
	}

	void Refraction(Vec3D v_direction, Vec3D v_intersection, Vec3D v_normal, float refractionIndex)
	{

	}

	Vec3D RefractRay(Vec3D v_incomingDirection, Vec3D v_normal, float refractionIndex1, float refractionIndex2)
	{
		float sinOutgoingAngle = refractionIndex1 * -DotProduct3D(v_incomingDirection, v_normal) / refractionIndex2;
		float cosOutgoingAngle = sqrt(1 - sinOutgoingAngle * sinOutgoingAngle);
		
		Vec3D v_tangent = CrossProduct(v_normal, CrossProduct(v_normal, v_incomingDirection));

		return AddVec3D(
			VecScalarMultiplication3D(v_normal, -sinOutgoingAngle),
			VecScalarMultiplication3D(v_tangent, cosOutgoingAngle)
		);
	}
};

int main()
{
	Engine rayTracer;
	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}

#include "Controlls.h"