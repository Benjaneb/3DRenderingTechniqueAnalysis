#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define RAY_TRACER
#define PATH_TRACING 0 // 0: distribution tracing, 1: path tracing

// Startup settings (cannot be changed during runtime)
#define ASYNC 1
#define THREAD_COUNT 4
#define SCREEN_WIDTH 900
#define SCREEN_HEIGHT 720
#define OFFSET_DISTANCE 0.0001
#define MOVE_DISTANCE 0.01
#define SAMPLES_PER_PIXEL 1000 // for path tracing
#define AMBIENT_LIGHT { 0, 0, 0 } //{ 27.5, 35, 55 } // sky light basically
#define GAUSSIAN_BLUR 1 // blur for denoising
#define MEDIAN_FILTER 0 // used for firefly reduction and denoising, bad for low spp
#define REFLECTIONS 1 // ON or OFF
#define MAX_COLOR_VALUE 1000000 // used for reducing fireflies, introduces bias
#define MAX_BOUNCES 2 // For distribution ray tracing
#define SAMPLES_PER_RAY 5 // for distribution ray tracing
#define WHITE_COLOR { 255, 255, 255 }
#define REFRACTION_INDEX_AIR 1

#include <iostream>
#include <random>
#include <future>

#include <hip/hip_runtime.h>
#include <>

#include "olcPixelGameEngine.h"

#include "MathUtilities.cuh"
#include "WorldDatatypes.h"
#include "ParseOBJ.h"

// Global variables

Vec3D screenBuffer[SCREEN_HEIGHT * SCREEN_WIDTH];

Player g_player;

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;
std::vector<Light> g_lights;

Ground g_ground;

// Textures
olc::Sprite* g_basketball_texture;
olc::Sprite* g_planks_texture;
olc::Sprite* g_concrete_texture;
olc::Sprite* g_tiledfloor_texture;
olc::Sprite* g_worldmap_texture;
olc::Sprite* g_bricks_texture;

olc::Sprite* g_basketball_normalmap;
olc::Sprite* g_planks_normalmap;
olc::Sprite* g_concrete_normalmap;
olc::Sprite* g_tiledfloor_normalmap;
olc::Sprite* g_worldmap_normalmap;
olc::Sprite* g_bricks_normalmap;

std::random_device seedEngine;
std::uniform_real_distribution<> uniformDistribution(-1, 1);
std::uniform_real_distribution<> uniform_zero_to_one(0, 1);

// Ingame options (can be changed during runtime)
namespace Options
{
	bool mcControls = true;
}

class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

	bool OnUserCreate() override
	{
		g_player = { { 1.5, 1.5, -2.064 }, { 1, ZERO_VEC3D }, TAU * 0.2f };
		//g_player = { { 1.5, 0.5, -0.5 }, { 1, ZERO_VEC3D }, TAU * 0.2f };

		g_basketball_texture = new olc::Sprite("../Assets/basketball.png");
		g_planks_texture = new olc::Sprite("../Assets/planks.png");
		g_concrete_texture = new olc::Sprite("../Assets/concrete.png");
		g_tiledfloor_texture = new olc::Sprite("../Assets/tiledfloor.png");
		g_worldmap_texture = new olc::Sprite("../Assets/worldmap.png");
		g_bricks_texture = new olc::Sprite("../Assets/bricks.png");

		g_basketball_normalmap = new olc::Sprite("../Assets/basketball_normalmap.png");
		g_planks_normalmap = new olc::Sprite("../Assets/planks_normalmap.png");
		g_concrete_normalmap = new olc::Sprite("../Assets/concrete_normalmap.png");
		g_tiledfloor_normalmap = new olc::Sprite("../Assets/tiledfloor_normalmap.png");
		g_worldmap_normalmap = new olc::Sprite("../Assets/tiledfloor_normalmap.png");
		g_bricks_normalmap = new olc::Sprite("../Assets/bricks_normalmap.png");

		g_spheres =
		{
			/* FIRST BALLS */

			/*// Lightsource
			{ { 1.5, 3, 1.5 }, 0.5, { { 45, 40, 30 }, { 0.9, 0.7, 0.1 }, { 0.9, 0.7, 0.1 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Glossy ball
			{ { 1.5, 1.4, 1.5 }, 0.4, { { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, { 0.8, 0.8, 0.8 }, 0.05, 12.5, { 500, 500, 500 } } },
			// Other lightsource
			{ { 0.6, 0.3, 0.85 }, 0.3, { { 30, 5, 10 }, { 0.9, 0.2, 0.4 }, { 0.9, 0.2, 0.4 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Other lightsource
			{ { 1.9, 0.3, 0.5 }, 0.3, { { 2.25, 13.1, 18.7 }, { 0.9, 0.2, 0.4 }, { 0.9, 0.2, 0.4 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Refractive ball
			{ { 2.5, 0.5, 2.2 }, 0.5, { { 0, 0, 0 }, { 0.02, 0.02, 0.02 }, { 0.4, 0.4, 0.4 }, 0.05, 1.52, { 0, 0, 0 } } }*/

			/* SECOND BALLS */

			//{ { 1.5, 3, 1.5 }, 0.5, { { 45, 40, 30 }, { 0.9, 0.7, 0.1 }, 0.5, 0.6, 1.6, { 500, 500, 500 }, 0, DIELECTRIC } },

			{ { 1.5, 0.7, 1.5 }, 0.7, { { 0, 0, 0 }, { 0, 0, 0 }, 0.8, 0.002, 1.04, { 0, 1, 0.666 }, 0, DIELECTRIC } }, // old IOR = 1.04
			//{ { 1.5, 0.7, 1.5 }, 0.7, { { 0, 0, 0 }, { 0.8, 0.2, 0.4 }, 0.8, 0.05, 12.5, { 500, 500, 500 }, 0, PLASTIC } },
			//{ { 1.5, 0.7, 1.5 }, 0.7, { { 0, 0, 0 }, { 0.843, 0.7176, 0.251 }, 0.8, 0.01, 10, { 500, 500, 500 }, 2.92, DIELECTRIC } },

			{ { 0.5, 0.45, 2.1 }, 0.45, { { 0, 0, 0 }, { 1.0, 0.851246, 0.301305 }, 0.8, 0.1, 0.277, { 500, 500, 500 }, 2.92, METAL } },

			{ { 2.5, 0.45, 2.1 }, 0.45, { { 0, 0, 0 }, { 0.31627, 0.95295, 0.56719 }, 0.85, 0.1, 3, { 500, 500, 500 }, 0, PLASTIC } },

			// Other Refractive ball
			//{ { 1.5, 2.3, 0.3 }, 0.5, { { 0, 0, 0 }, { 0.2, 0.2, 0.2 }, { 0.2, 0.2, 0.2 }, 0.3, 1.52, { 0, 0, 0 } } }
			// Basket ball
			//{ { 2.5, 0.5, 0.8 }, 0.5, { 1, 1, 1 }, { 0.2, 0.6, 0.8, 0.9, { -1, 0, 0 }, 500, 2 }, g_basketball_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ 1, 0, 1 }), PI / 2) },
			// World atlas globe
			//{ { 1.75, 0.3, 0.5 }, 0.3, { 1, 1, 1 }, { 0.35, 0.7, 0.7, 0.9, { 1, 0, 0 }, 500, 1.45 }, g_worldmap_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ -1, 0.5, -2 }), PI / 2) },
			// Magenta lightsource
			//{ { 0.5, 0.4, 0.8 }, 0.4, { 1, 0.2, 0.4157 }, { 35, 0.2, 0.5, 0.95, { -1, 0, 0 }, 500, 1.6 } },
			// Refractive ball
			//{ { 1.1, 0.3, 0.4 }, 0.3, { 1, 1, 1 }, { 0.2, 0.2, 0.2, 0.95, { 1, 0, 0 }, 0.5, 1.4 } },
			// Cyan lightsource
			//{ { 2.4, 0.3, 1.75 }, 0.3, { 0.3, 1.15, 1.15 }, { 45, 0.2, 0.5, 0.95, { 1, 0, 0 }, 500, 1.6 } }
			
		};

		g_triangles =
		{
			// Walls north face
			{ { { 0, 0, 3 }, { 0, 3, 3 }, { 3, 3, 3 } }, { { 0, 0, 0 }, { 0.9, 0.9, 0.9 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_bricks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } },     g_bricks_normalmap*/ },
			{ { { 0, 0, 3 }, { 3, 3, 3 }, { 3, 0, 3 } }, { { 0, 0, 0 }, { 0.9, 0.9, 0.9 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_bricks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } },     g_bricks_normalmap*/ },
			// Walls west face														   													  
			{ { { 0, 0, 0 }, { 0, 3, 0 }, { 0, 3, 3 } }, { { 0, 0, 0 }, { 0.9, 0.2, 0.1 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap*/ },
			{ { { 0, 0, 0 }, { 0, 3, 3 }, { 0, 0, 3 } }, { { 0, 0, 0 }, { 0.9, 0.2, 0.1 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap*/ },
			// Walls east face
			{ { { 3, 0, 3 }, { 3, 3, 3 }, { 3, 3, 0 } }, { { 0, 0, 0 }, { 0.1, 0.9, 0.3 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap*/ },
			{ { { 3, 0, 3 }, { 3, 3, 0 }, { 3, 0, 0 } }, { { 0, 0, 0 }, { 0.1, 0.9, 0.3 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap*/ },
			// Walls ceiling
			{ { { 0, 3, 0 }, { 3, 3, 3 }, { 0, 3, 3 } }, { { 0, 0, 0 }, { 0.9, 0.9, 0.9 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap*/ },
			{ { { 0, 3, 0 }, { 3, 3, 0 }, { 3, 3, 3 } }, { { 0, 0, 0 }, { 0.9, 0.9, 0.9 }, 0.2, 0.975, 1.3, { 500, 500, 500 }, 0, DIELECTRIC }/*, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap*/ },

			// Tall box north face
			{ { { 0.5, 0, 2.5 }, { 1.25, 1.58, 2.75 }, { 1.25, 0, 2.75 } },				{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 0.5, 0, 2.5 }, { 0.5, 1.58, 2.5 }, { 1.25, 1.58, 2.75 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Tall box south face
			{ { { 0.75, 0, 1.75 }, { 1.5, 1.58, 2 }, { 1.5, 0, 2 } },					{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 0.75, 0, 1.75 }, { 0.75, 1.58, 1.75 }, { 1.5, 1.58, 2 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Tall box west face
			{ { { 0.5, 0, 2.5 }, { 0.75, 1.58, 1.75 }, { 0.75, 0, 1.75 } },				{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 0.5, 0, 2.5 }, { 0.5, 1.58, 2.5 }, { 0.75, 1.58, 1.75 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Tall box east face
			{ { { 1.5, 0, 2 }, { 1.25, 1.58, 2.75 }, { 1.25, 0, 2.75 } },				{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 1.5, 0, 2 }, { 1.5, 1.58, 2 }, { 1.25, 1.58, 2.75 } }, 				{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Tall box top face
			{ { { 0.75, 1.58, 1.75 }, { 1.25, 1.58, 2.75 }, { 1.5, 1.58, 2 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 0.75, 1.58, 1.75 }, { 0.5, 1.58, 2.5 }, { 1.25, 1.58, 2.75 } },		{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },

			// Box north face															   
			{ { { 1.625, 0, 1.5 }, { 2.375, 0.79, 1.25 }, { 2.375, 0, 1.25 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 1.625, 0, 1.5 }, { 1.625, 0.79, 1.5 }, { 2.375, 0.79, 1.25 } },		{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Box south face
			{ { { 1.375, 0, 0.75 }, { 2.125, 0.79, 0.5 }, { 2.125, 0, 0.5 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 1.375, 0, 0.75 }, { 1.375, 0.79, 0.75 }, { 2.125, 0.79, 0.5 } },		{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Box west face
			{ { { 1.625, 0, 1.5 }, { 1.375, 0.79, 0.75 }, { 1.375, 0, 0.75 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 1.625, 0, 1.5 }, { 1.625, 0.79, 1.5 }, { 1.375, 0.79, 0.75 } },		{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Box east face
			{ { { 2.375, 0, 1.25 }, { 2.125, 0.79, 0.5 }, { 2.125, 0, 0.5 } },			{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 2.375, 0, 1.25 }, { 2.375, 0.79, 1.25 }, { 2.125, 0.79, 0.5 } },		{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			// Box top face
			{ { { 1.375, 0.79, 0.75 }, { 2.375, 0.79, 1.25 }, { 2.125, 0.79, 0.5 } },	{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },
			{ { { 1.375, 0.79, 0.75 }, { 1.625, 0.79, 1.5 }, { 2.375, 0.79, 1.25 } },	{ { 0, 0, 0 }, { 0.8, 0.8, 0.8 }, 0.4, 0.9, 1.7, { 500, 500, 500 }, 0, DIELECTRIC } },

			// refractive pyramid
			/*{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.1, 0 + 0.01, 2.9 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.1, 0 + 0.01, 2.9 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.1, 0 + 0.01, 2.1 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.9, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.1 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.9, 0 + 0.01, 2.9 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.1, 0 + 0.01, 2.9 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.9, 0 + 0.01, 2.1 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.52 } },

			// other refractive pyramid
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.1 + 2, 0 + 0.01, 2.9 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.1 + 2, 0 + 0.01, 2.9 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.1 + 2, 0 + 0.01, 2.1 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.9 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.1 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.9 + 2, 0 + 0.01, 2.9 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.1 + 2, 0 + 0.01, 2.9 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.9 + 2, 0 + 0.01, 2.1 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.7 } }*/
		};

		g_ground = { 0, { { 0, 0, 0 }, { 1, 1, 1 }, 0.3, 0.6, 2, { 500, 500, 500 }, 0, DIELECTRIC }, g_tiledfloor_texture, { 0, 0 }, { 1, 1 }, 1, g_tiledfloor_normalmap };

#if PATH_TRACING == 0
		g_lights =
		{
			//{ { 1.5, 2.7, 1.5 }, 0.3, { 45, 40, 30 } },
			{ { 1.5, 2.7, 1.5 }, 0.3, { 30, 26.67, 20 } },
			//{ { 1.5, 2.7, 1.5 }, 0.3, { 39.3, 45.3, 20 } },
			//{ { 0, 0, -1005 }, 1000, { 45, 40, 30 } } // Giant light source
		};

		// Insert all light sources into sphere list to make them visible
		for (int i = 0; i < g_lights.size(); i++)
			g_spheres.push_back({ g_lights[i].coords, g_lights[i].radius - 0.1, { g_lights[i].emittance, ZERO_VEC3D, 0, 0, 1, { 0, 0, 0 } } });
#endif

#if ASYNC == 1
	//std::async(std::launch::async, ImportScene, &g_triangles, "../Assets/RubberDuck.obj", 0.4, Vec3D({ 0.8, 0.5, 0.5 }));
#else
	//ImportScene(&g_triangles, "../Assets/RubberDuck.obj", 0.4, { 0.8, 0.5, 0.5 });
#endif

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		Timer timer("Rendering");

		Controlls(fElapsedTime);

		StartThreads();

#if GAUSSIAN_BLUR == 1
		GaussianBlur();
#endif

#if MEDIAN_FILTER == 1
		MedianFiltering();
#endif

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				Vec3D pixelColor = screenBuffer[y * SCREEN_WIDTH + x];

				Draw(x, y, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
		}

		std::cout << "\a" << std::endl;

		return true;
	}

	void StartThreads()
	{
#if ASYNC == 1
		// Screen split up into 4 quadrants running in parallell on seperate threads

		std::future<void> returnValues[THREAD_COUNT];

		for (int i = 0; i < THREAD_COUNT; i++)
		{
			int startX = i * ceil(SCREEN_WIDTH / double(THREAD_COUNT));
			int endX = (i + 1) * ceil(SCREEN_WIDTH / double(THREAD_COUNT));

			if (startX >= SCREEN_WIDTH)
			{
				break;
			}

			endX = Min(endX, SCREEN_WIDTH);

			std::mt19937 randomEngine(seedEngine());

			returnValues[i] = std::async(std::launch::async, &Engine::RayTracing, this, startX, endX, randomEngine);
		}
#else
		std::mt19937 randomEngine(seedEngine());
		RayTracing(0, SCREEN_WIDTH, randomEngine);
#endif
	}

private:
	// Defined in Controlls.h
	void Controlls(float fElapsedTime);

	void RayTracing(int startX, int endX, std::mt19937 randomEngine)
	{
		const double zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (double y = -SCREEN_HEIGHT * 0.5f + 0.5f; y < SCREEN_HEIGHT * 0.5f + 0.5f; y++)
		{
			for (double x = -SCREEN_WIDTH * 0.5f + 0.5f + startX; x < -SCREEN_WIDTH * 0.5f + 0.5f + endX; x++)
			{
				Vec3D v_direction = { x, y, zFar };

				Vec3D v_orientedDirection = QuaternionMultiplication(g_player.q_orientation, { 0, v_direction }, QuaternionConjugate(g_player.q_orientation)).vecPart;

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = SCREEN_HEIGHT - (y + SCREEN_HEIGHT * 0.5f);

				Vec3D pixelColor = ZERO_VEC3D;

				for (int i = 0; i < SAMPLES_PER_PIXEL; i++)
				{
#if PATH_TRACING == 1
					// For anti-aliasing
					Vec3D v_jitteredDirection = AddVec3D(v_orientedDirection, RandomVec_InUnitSphere(&randomEngine));
					NormalizeVec3D(&v_jitteredDirection);

					AddToVec3D(&pixelColor, RenderPixel(g_player.coords, v_jitteredDirection, &randomEngine));
#else
					NormalizeVec3D(&v_orientedDirection);

					AddToVec3D(&pixelColor, RenderPixel(g_player.coords, v_orientedDirection, &randomEngine));
#endif
				}

				ScaleVec3D(&pixelColor, 1 / double(SAMPLES_PER_PIXEL));

				pixelColor.x = Min(pixelColor.x, 255.0f);
				pixelColor.y = Min(pixelColor.y, 255.0f);
				pixelColor.z = Min(pixelColor.z, 255.0f);

				ScaleVec3D(&pixelColor, 1.0f / 255);

				pixelColor = { LINEAR_TO_SRGB(pixelColor.x), LINEAR_TO_SRGB(pixelColor.y), LINEAR_TO_SRGB(pixelColor.z) };

				ScaleVec3D(&pixelColor, 255.0f);

				screenBuffer[screenY * SCREEN_WIDTH + screenX] = pixelColor;

				//Draw(screenX, screenY, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
#if PATH_TRACING == 1
			std::cout << ((y + SCREEN_HEIGHT * 0.5f) / SCREEN_HEIGHT) * 100 << "%" << '\n';
#endif
		}
	}

	Vec3D RenderPixel(Vec3D v_start, Vec3D v_direction, std::mt19937* randomEngine)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_textureColor = ZERO_VEC3D;
		Quaternion q_surfaceNormal = IDENTITY_QUATERNION;
		Material material;

		bool intersectionExists = NextIntersection(v_start, v_direction, &v_intersection, &v_textureColor, &q_surfaceNormal, &material);

		if (intersectionExists)
		{
#if PATH_TRACING == 1
			v_textureColor = CalculateLighting_PathTracing(
				v_textureColor, material, q_surfaceNormal, v_direction, v_intersection, { 1, 1, 1 }, randomEngine
			);
#else
			v_textureColor = CalculateLighting_DistributionTracing(
				v_textureColor, material, q_surfaceNormal.vecPart, v_direction, v_intersection, 0, randomEngine
			);
#endif
		}

		return v_textureColor;
	}

	void MedianFilter()
	{
		auto AddColorToVector = [](std::vector<Vec3D>* colors, int x, int y)
		{
			if (x >= 0 && x < SCREEN_WIDTH && y >= 0 && y < SCREEN_HEIGHT)
			{
				colors->push_back(screenBuffer[y * SCREEN_WIDTH + x]);
			}
		};

		auto ColorSize = [](Vec3D color)
		{
			return Max(color.x, Max(color.y, color.z));
		};

		auto MedianColor = [ColorSize](std::vector<Vec3D>* colors)
		{
			for (int i = 0; i < colors->size(); i++)
			{
				for (int j = 0; j < colors->size() - 1; j++)
				{
					if (ColorSize(colors->at(j)) > ColorSize(colors->at(j + 1)))
					{
						SwapVec3D(&(colors->at(j)), &(colors->at(j + 1)));
					}
				}
			}

			return colors->at(colors->size() / 2);
		};

		Vec3D* screenBufferCopy = new Vec3D[SCREEN_HEIGHT * SCREEN_WIDTH];

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				std::vector<Vec3D> colors;

				AddColorToVector(&colors, x, y);
				AddColorToVector(&colors, x + 1, y);
				AddColorToVector(&colors, x - 1, y);
				AddColorToVector(&colors, x, y + 1);
				AddColorToVector(&colors, x, y - 1);

				screenBufferCopy[y * SCREEN_WIDTH + x] = MedianColor(&colors);
			}
		}

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				screenBuffer[y * SCREEN_WIDTH + x] = screenBufferCopy[y * SCREEN_WIDTH + x];
			}
		}

		delete[] screenBufferCopy;
	}

	void GaussianBlur()
	{
		auto WeightedPixel = [](double weight, int x, int y)
		{
			Vec3D weightedPixel = ZERO_VEC3D;

			if (x >= 0 && x < SCREEN_WIDTH && y >= 0 && y < SCREEN_HEIGHT)
			{
				weightedPixel = VecScalarMultiplication3D(screenBuffer[y * SCREEN_WIDTH + x], weight);
			}

			return weightedPixel;
		};

#define KERNEL_SIZE 3

		double gaussianKernel[KERNEL_SIZE * KERNEL_SIZE] =
		{
			0.0000, 0.0625, 0.0000,
			0.0625, 0.7500, 0.0625,
			0.0000, 0.0625, 0.0000,
		};

		Vec3D* screenBufferCopy = new Vec3D[SCREEN_HEIGHT * SCREEN_WIDTH];

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				Vec3D blurredPixel = ZERO_VEC3D;

				for (int i = 0; i < 3; i++)
				{
					for (int j = 0; j < 3; j++)
					{
						AddToVec3D(&blurredPixel, WeightedPixel(gaussianKernel[i * KERNEL_SIZE + j], x + j - KERNEL_SIZE / 2, y + i - KERNEL_SIZE / 2));
					}
				}

				screenBufferCopy[y * SCREEN_WIDTH + x] = blurredPixel;
			}
		}

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				screenBuffer[y * SCREEN_WIDTH + x] = screenBufferCopy[y * SCREEN_WIDTH + x];
			}
		}

		delete[] screenBufferCopy;
	}

	double LINEAR_TO_SRGB(double l)
	{
		if (l <= 0.0031308)
		{
			return l * 12.92;
		}

		return 1.055 * pow(l, 0.41666) - 0.055;
	}

	bool GroundIntersection_RT(Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		if (v_direction.y >= 0 || v_start.y < g_ground.level)
		{
			return false;
		}

		ScaleVec3D(&v_direction, (g_ground.level - v_start.y) / v_direction.y);

		Vec3D rayGroundIntersection = AddVec3D(v_start, v_direction);

		if (v_intersection != nullptr)
		{
			*v_intersection = rayGroundIntersection;
		}

		if (q_surfaceNormal != nullptr)
		{
			*q_surfaceNormal = { 1, { 0, 1, 0 } };
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (g_ground.texture != nullptr || g_ground.normalMap != nullptr)
		{
			double signedTextureWidth = (g_ground.textureCorner2.x - g_ground.textureCorner1.x) * g_ground.textureScalar;
			double signedTextureHeight = (g_ground.textureCorner2.y - g_ground.textureCorner1.y) * g_ground.textureScalar;

			double t1 = fmod(rayGroundIntersection.x, signedTextureWidth) / signedTextureWidth;
			double t2 = fmod(rayGroundIntersection.z, signedTextureHeight) / signedTextureHeight;

			// if the t values are negative, we need to flip them around the center of the texture and make them positive
			if (t1 < 0) t1 += 1;
			if (t2 < 0) t2 += 1;

			double textureX = Lerp(g_ground.textureCorner1.x, g_ground.textureCorner2.x, t1);
			double textureY = Lerp(g_ground.textureCorner1.y, g_ground.textureCorner2.y, t2);

			if (g_ground.texture != nullptr)
			{
				olc::Pixel texelColor = g_ground.texture->Sample(textureX, textureY);

				*v_intersectionColor = { double(texelColor.r), double(texelColor.g), double(texelColor.b) };
			}
			if (g_ground.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = g_ground.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				q_surfaceNormal->vecPart = ReturnNormalizedVec3D({ double(normalMapColor.r) * 2 - 255.0f, double(normalMapColor.b) * 2 - 255.0f, double(normalMapColor.g) * 2 - 255.0f });
			}
		}

		return true;
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		double dxdz = v_direction.x / v_direction.z;
		double dydz = v_direction.y / v_direction.z;

		double a = dxdz * dxdz + dydz * dydz + 1;
		
		double b = 
			2 * dxdz * (v_start.x - sphere.coords.x) +
			2 * dydz * (v_start.y - sphere.coords.y) +
			2 * (v_start.z - sphere.coords.z);

		double c = 
			(v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) +
			(v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) +
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// ISAK: There wasn't any need to recalculate this multiple times
		double rootContent = b * b - 4 * a * c;

		// There exists no intersections (no real answer)
		if (rootContent < 0) return false;

		double z1 = (-b + sqrt(rootContent)) / (2 * a);
		double z2 = (-b - sqrt(rootContent)) / (2 * a);

		Vec3D v_alternative1 = { z1 * dxdz, z1 * dydz, z1 };
		AddToVec3D(&v_alternative1, v_start);

		Vec3D v_alternative2 = { z2 * dxdz, z2 * dydz, z2 };
		AddToVec3D(&v_alternative2, v_start);

		// Check which intersection is the closest and choose that one
		double dist1 = DistanceSquared3D(v_alternative1, v_start);
		double dist2 = DistanceSquared3D(v_alternative2, v_start);

		bool dist1Closest = dist1 < dist2;

		Vec3D v_correctHit = dist1Closest ? v_alternative1 : v_alternative2;
		Vec3D v_otherHit = dist1Closest ? v_alternative2 : v_alternative1;

		// Check if the intersection is behind the ray. If so, choose the other one.
		if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0)
		{
			v_correctHit = v_otherHit;

			// Check if the other intersection is behind the ray. If so, discard it.
			if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0) return false;
		}

		// Checks whether or not to return the intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_correctHit;
		}

		// Calculating the normal of the sphere (without normalmap)
		Vec3D v_normal = SubtractVec3D(v_correctHit, sphere.coords);
		NormalizeVec3D(&v_normal);

		if (q_surfaceNormal != nullptr)
		{
			q_surfaceNormal->vecPart = v_normal;
			q_surfaceNormal->realPart = 1;

			if (DistanceSquared3D(v_start, sphere.coords) < sphere.radius * sphere.radius)
			{
				q_surfaceNormal->realPart = -1;
			}
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (sphere.texture != nullptr || sphere.normalMap != nullptr)
		{
			Vec3D i_Hat = { 1, 0, 0 };
			Vec3D j_Hat = { 0, 1, 0 };
			Vec3D k_Hat = { 0, 0, 1 };

			// Rotating axies by sphere rotation quaternion
			i_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, i_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			j_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, j_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			k_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, k_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;

			// Translate normal into new coordinate system
			v_normal = { DotProduct3D(v_normal, i_Hat), DotProduct3D(v_normal, j_Hat), DotProduct3D(v_normal, k_Hat) };

			// UV coordinates
			double u = 0.5 + atan2(v_normal.x, v_normal.z) / TAU;
			double v = 0.5 - asin(v_normal.y) / PI;

			double textureX = Lerp(sphere.textureCorner1.x, sphere.textureCorner2.x, u);
			double textureY = Lerp(sphere.textureCorner1.y, sphere.textureCorner2.y, v);

			if (sphere.texture != nullptr)
			{
				// Interpolating between assigned texture coordinates
				olc::Pixel texelColor = sphere.texture->Sample(textureX, textureY);

				*v_intersectionColor = { (double)texelColor.r, (double)texelColor.g, (double)texelColor.b };
			}
			if (sphere.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = sphere.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ double(normalMapColor.r) * 2 - 255.0f, double(normalMapColor.b) * 2 - 255.0f, double(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the sphere
				Vec3D v_sidewaysTangent = ReturnNormalizedVec3D({ -v_normal.z, 0, v_normal.x });
				Vec3D v_forwardTangent = ReturnNormalizedVec3D({ 0, -v_normal.z, v_normal.y });

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					v_sidewaysTangent,
					v_normal,
					v_forwardTangent
				};

				q_surfaceNormal->vecPart = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		return true;
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		double f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		if (DotProduct3D(SubtractVec3D(v_trianglePlaneIntersection, v_start), v_direction) < 0) return false;

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if ((DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) ||
			(DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) ||
			(DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0))
		{
			return false;
		}

		// Checks whether or not to return the intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_trianglePlaneIntersection;
		}

		if (q_surfaceNormal != nullptr)
		{
			q_surfaceNormal->vecPart = v_triangleNormal;

			q_surfaceNormal->realPart = 1;

			if (DotProduct3D(v_triangleNormal, v_direction) > 0)
			{
				// The triangle face is inside of the mesh, so the normal must be flipped
				q_surfaceNormal->realPart = -1;
			}
		}
		
		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (triangle.texture != nullptr || triangle.normalMap != nullptr)
		{
			// from here on we calculate the texture coordinates

			Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

			Matrix3D triangleMatrix =
			{
				v_triangleEdge1,
				v_triangleEdge2,
				v_triangleNormal
			};

			Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

			Vec2D textureCoordinates = { 0, 0 };

			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]), triangleEdgeScalars.x));
			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]), triangleEdgeScalars.y));
			AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

			if (triangle.texture != nullptr)
			{
				olc::Pixel texelColor = triangle.texture->Sample(textureCoordinates.x, textureCoordinates.y);

				*v_intersectionColor = { double(texelColor.r), double(texelColor.g), double(texelColor.b) };
			}
			if (triangle.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = triangle.normalMap->Sample(textureCoordinates.x, textureCoordinates.y);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ double(normalMapColor.r) * 2 - 255.0f, double(normalMapColor.b) * 2 - 255.0f, double(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the triangle for finding the normal in object space

				// { u1, v1 }, { u2, v2 }, { u3, v3 } are the normalMapVertices
				// T is the tangent
				// B is the bitangent
				
				//                       | T.x  B.x  0 |   
				// { v_triangleEdge1 } = | T.y  B.y  0 | * { u2 - u1, v2 - v1, 0 }
				//                       | T.z  B.z  0 |   

				//                       | T.x  B.x  0 |   
				// { v_triangleEdge2 } = | T.y  B.y  0 | * { u3 - u1, v3 - v1, 0 }
				//                       | T.z  B.z  0 |   

				// | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | T.x  B.x  0 |   | u2 - u1  u3 - u1  0 |
				// | v_triangleEdge1.y  v_triangleEdge2.y  0 | = | T.y  B.y  0 | * | v2 - v1  v3 - v1  0 |
				// | v_triangleEdge1.z  v_triangleEdge2.z  0 |   | T.z  B.z  0 |   |    0        0     1 |

				//                                                                                       -1
				// | T.x  B.x  0 |   | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | u2 - u1  u3 - u1  0 |
				// | T.y  B.y  0 | = | v_triangleEdge1.y  v_triangleEdge2.y  0 | * | v2 - v1  v3 - v1  0 |
				// | T.z  B.z  0 |	 | v_triangleEdge1.z  v_triangleEdge2.z  0 |   |    0        0     1 |

				Matrix3D m1 =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					ZERO_VEC3D
				};

				Matrix3D m2 =
				{
					{ triangle.textureVertices[1].x - triangle.textureVertices[0].x, triangle.textureVertices[1].y - triangle.textureVertices[0].y, 0 },
					{ triangle.textureVertices[2].x - triangle.textureVertices[0].x, triangle.textureVertices[2].y - triangle.textureVertices[0].y, 0 },
					{ 0, 0, 1 }
				};

				Matrix3D tangentsMatrix = MatrixMultiplication3D(InverseMatrix3D(m2), m1);

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					ReturnNormalizedVec3D(tangentsMatrix.i_Hat),
					v_triangleNormal,
					ReturnNormalizedVec3D(tangentsMatrix.j_Hat)
				};

				q_surfaceNormal->vecPart = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, double f_planeOffset)
	{
		double f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		double f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	enum ScatteringType
	{
		LAMBERTIAN,
		SPECULAR,
		TRANSMISSIVE
	};

	Vec3D CalculateLighting_PathTracing(Vec3D v_textureColor, Material material, Quaternion q_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, Vec3D accumulatedAttenuation, std::mt19937* randomEngine)
	{
		Vec3D v_outgoingLightColor = ConusProduct(v_textureColor, material.emittance);

		// counterintuitive, but the probability goes up when accumulatedAttenuation goes up
		double survivalProbability = Max(Sigmoid(2 * Max(accumulatedAttenuation.x, Max(accumulatedAttenuation.y, accumulatedAttenuation.z))), 0.1);

		// Randomly terminate paths with russian roulette
		if (uniform_zero_to_one(*randomEngine) > survivalProbability)
		{
			return v_outgoingLightColor;
		}

		double refractionIndex1 = REFRACTION_INDEX_AIR;
		double refractionIndex2 = material.refractionIndex;
		Vec3D attenuation = { 0, 0, 0 };

		if (q_surfaceNormal.realPart == -1)
		{
			refractionIndex1 = material.refractionIndex;
			refractionIndex2 = REFRACTION_INDEX_AIR;
			v_outgoingLightColor = ZERO_VEC3D;
		}

		ScaleVec3D(&v_incomingDirection, -1);

		// Scale the normal to be oriented in the hemisphere the material was hit from
		ScaleVec3D(&(q_surfaceNormal.vecPart), q_surfaceNormal.realPart);

		Vec3D v_outgoingDirection;
		ScatteringType scatteringType;

		Vec3D v_microscopicNormal = MicroscopicNormal(v_incomingDirection, q_surfaceNormal.vecPart, material.roughness, randomEngine); // for specular and transmissive scattering

		double scatteringTypeProbability; // will be assigned a value later on, used for energy conservation

		bool isMaterialDielectric = (material.type == DIELECTRIC);
		bool isMaterialMetallic = (material.type == METAL);

		double reflectionProbability = 1.0; // 1.0 for metals

		if (!isMaterialMetallic)
		{
			double normalisedAttenuation = -exp(-Min(material.attenuation.x, Min(material.attenuation.y, material.attenuation.z))) + 1.0; // between 0 and 1
			double fresnelDielectric = FresnelDielectric(v_incomingDirection, v_microscopicNormal, refractionIndex1, refractionIndex2) * 0.5;

			reflectionProbability = Max(fresnelDielectric, normalisedAttenuation);
		}

		if (uniform_zero_to_one(*randomEngine) <= reflectionProbability)
		{
			double specularProbability = 1.0; // 1.0 for non-dielectrics
			
			if (isMaterialDielectric)
			{
				specularProbability = material.specularValue / (material.specularValue + Max(material.diffuseTint.x, Max(material.diffuseTint.y, material.diffuseTint.z)));
			}

			if(uniform_zero_to_one(*randomEngine) <= specularProbability)
			{
				scatteringType = SPECULAR;

				v_outgoingDirection = SubtractVec3D(VecScalarMultiplication3D(v_microscopicNormal, 2 * DotProduct3D(v_incomingDirection, v_microscopicNormal)), v_incomingDirection);

				scatteringTypeProbability = specularProbability * reflectionProbability;
			}
			else
			{
				scatteringType = LAMBERTIAN;

				Vec3D v_tangent = ReturnNormalizedVec3D(SubtractVec3D(v_incomingDirection, VecScalarMultiplication3D(q_surfaceNormal.vecPart, DotProduct3D(v_incomingDirection, q_surfaceNormal.vecPart))));

				Matrix3D transformationMatrix =
				{
					v_tangent,
					q_surfaceNormal.vecPart,
					CrossProduct(q_surfaceNormal.vecPart, v_tangent)
				};

				double randVariable = uniform_zero_to_one(*randomEngine);
				double theta = uniform_zero_to_one(*randomEngine) * TAU;

				double r = sqrt(randVariable);

				v_outgoingDirection = VecMatrixMultiplication3D({ r * cos(theta), sqrt(1 - randVariable), r * sin(theta) }, transformationMatrix);

				scatteringTypeProbability = (1 - specularProbability) * reflectionProbability;
			}
		}
		else
		{
			scatteringType = TRANSMISSIVE;

			double n = refractionIndex1 / refractionIndex2;

			double incomingDotBisector = DotProduct3D(v_incomingDirection, v_microscopicNormal);

			double bisectorScalar = n * incomingDotBisector - Sign(DotProduct3D(v_incomingDirection, q_surfaceNormal.vecPart)) * sqrt(Max(1 + n * (incomingDotBisector * incomingDotBisector - 1), 0));

			v_outgoingDirection = SubtractVec3D(VecScalarMultiplication3D(v_microscopicNormal, bisectorScalar), VecScalarMultiplication3D(v_incomingDirection, n));

			scatteringTypeProbability = 1.0 - reflectionProbability;
		}

		NormalizeVec3D(&v_outgoingDirection);

		AddToVec3D(&v_intersection, VecScalarMultiplication3D(v_outgoingDirection, OFFSET_DISTANCE));

		if (DotProduct3D(v_outgoingDirection, q_surfaceNormal.vecPart) * q_surfaceNormal.realPart < 0)
		{
			// The ray is going through the object
			attenuation = material.attenuation;
		}

		Vec3D v_nextIntersection = ZERO_VEC3D;
		Vec3D v_nextTextureColor = ZERO_VEC3D;
		Quaternion q_nextNormal = IDENTITY_QUATERNION;
		Material nextMaterial;

		Vec3D v_diffuseTint = VecScalarMultiplication3D(ConusProduct(v_textureColor, material.diffuseTint), 1.0 / 255);

		Vec3D weight = ZERO_VEC3D;

		if (scatteringType == LAMBERTIAN)
		{
			weight = VecScalarMultiplication3D(BRDF_LAMBERTIAN(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, refractionIndex1, refractionIndex2, v_diffuseTint), PI / scatteringTypeProbability);
		}
		else if (scatteringType == SPECULAR)
		{
			weight = VecScalarMultiplication3D(
				BRDF_COOKTORRANCE(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, v_microscopicNormal, refractionIndex1, refractionIndex2, material.roughness, material.extinctionCoefficient, material.specularValue, isMaterialMetallic), 1.0 / scatteringTypeProbability
			);

			if (!isMaterialDielectric)
			{
				weight = ConusProduct(weight, v_diffuseTint);
			}
		}
		else
		{
			weight = VecScalarMultiplication3D(BTDF(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, v_microscopicNormal, refractionIndex1, refractionIndex2, material.roughness), 1.0 / scatteringTypeProbability);
		}

		Vec3D v_incomingLightColor = AMBIENT_LIGHT;

		bool intersectionExists = NextIntersection(v_intersection, v_outgoingDirection, &v_nextIntersection, &v_nextTextureColor, &q_nextNormal, &nextMaterial);

		double distance = Distance3D(v_intersection, v_nextIntersection);

		attenuation = { exp(-attenuation.x * distance), exp(-attenuation.y * distance), exp(-attenuation.z * distance) };

		weight = ConusProduct(weight, attenuation);

		if (intersectionExists)
		{
			v_incomingLightColor = CalculateLighting_PathTracing(
				v_nextTextureColor, nextMaterial, q_nextNormal, v_outgoingDirection, v_nextIntersection, accumulatedAttenuation, randomEngine
			);
		}

		v_incomingLightColor = { Min(v_incomingLightColor.x, MAX_COLOR_VALUE), Min(v_incomingLightColor.y, MAX_COLOR_VALUE), Min(v_incomingLightColor.z, MAX_COLOR_VALUE) }; // Introduces bias. To avoid bias MAX_COLOR_VALUE should be very high

		// Add the energy that is lost by randomly terminating paths
		ScaleVec3D(&v_incomingLightColor, 1.0 / survivalProbability);

		AddToVec3D(&v_outgoingLightColor, ConusProduct(v_incomingLightColor, weight));

		return v_outgoingLightColor;
	}

	bool NextIntersection(Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection, Vec3D* v_color, Quaternion* q_normal, Material* material)
	{
		// Check all spheres
		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool sphereIntersect = SphereIntersection_RT(g_spheres[i], v_start, v_direction, v_intersection, v_color, q_normal);

			if (sphereIntersect && !IsRayBlocked(v_start, v_direction, *v_intersection))
			{
				*material = g_spheres[i].material;
				return true;
			}
		}

		// Check all triangles
		for (int i = 0; i < g_triangles.size(); i++)
		{
			bool triangleIntersect = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, v_intersection, v_color, q_normal);

			if (triangleIntersect && !IsRayBlocked(v_start, v_direction, *v_intersection))
			{
				*material = g_triangles[i].material;
				return true;
			}
		}

		// Check ground
		bool groundIntersect = GroundIntersection_RT(v_start, v_direction, v_intersection, v_color, q_normal);

		if (groundIntersect)
		{
			*material = g_ground.material;
			return true;
		}
	}

	bool IsRayBlocked(Vec3D v_start, Vec3D v_direction, Vec3D v_intersection)
	{
		Vec3D v_otherIntersection = ZERO_VEC3D;

		for (int j = 0; j < g_spheres.size(); j++)
		{
			bool otherIntersectionExists = SphereIntersection_RT(g_spheres[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		for (int j = 0; j < g_triangles.size(); j++)
		{
			bool otherIntersectionExists = TriangleIntersection_RT(g_triangles[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		bool otherIntersectionExists = GroundIntersection_RT(v_start, v_direction, &v_otherIntersection);

		// If there exists a closer intersection to the ray start vector it means the ray is blocked
		if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
		{
			return true;
		}

		// The ray is not blocked
		return false;
	}

	// Cook-Torrance (cock tolerance) BRDF with GGX distribution function and GGX geometry function
	Vec3D BRDF_COOKTORRANCE(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, Vec3D v_microscopicNormal, double refractionIndex1, double refractionIndex2, double roughness, double extinctionCoefficient, double specularValue, bool isMaterialMetallic)
	{

		double fresnelFactor;
		
		if (isMaterialMetallic)
		{
			fresnelFactor = FresnelConductor(v_incomingDirection, v_microscopicNormal, refractionIndex1, refractionIndex2, extinctionCoefficient);
		}
		else
		{
			fresnelFactor = FresnelDielectric(v_incomingDirection, v_microscopicNormal, refractionIndex1, refractionIndex2);
		}

		// Some terms are not included because they are cancelled out bt the PDF
		double specularTerm = Abs(DotProduct3D(v_incomingDirection, v_microscopicNormal)) * fresnelFactor * GeometryBidirectional(v_incomingDirection, v_outgoingDirection, v_normal, v_microscopicNormal, roughness) /
			(Abs(DotProduct3D(v_incomingDirection, v_normal)) * Abs(DotProduct3D(v_microscopicNormal, v_normal)));

		return VecScalarMultiplication3D({ specularValue, specularValue, specularValue }, specularTerm);
	}

	Vec3D BRDF_LAMBERTIAN(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, double refractionIndex1, double refractionIndex2, Vec3D v_diffuseTint)
	{
		Vec3D v_bisectorVector = ReturnNormalizedVec3D(Lerp3D(v_incomingDirection, v_outgoingDirection, 0.5));

		double fresnelFactor = FresnelDielectric(v_incomingDirection, v_bisectorVector, refractionIndex1, refractionIndex2);

		double diffuseTerm = Chi(DotProduct3D(v_bisectorVector, v_normal)) * Square(1 - fresnelFactor) / PI;

		return VecScalarMultiplication3D(v_diffuseTint, diffuseTerm);
	}

	double Chi(double x)
	{
		return x > 0 ? 1 : 0;
	}

	double FresnelDielectric(Vec3D v_incomingDirection, Vec3D v_microscopicNormal, double refractionIndex1, double refractionIndex2)
	{
		double c = Abs(DotProduct3D(v_incomingDirection, v_microscopicNormal));

		double g = sqrt(Max((refractionIndex2 * refractionIndex2) / (refractionIndex1 * refractionIndex1) - 1 + c * c, 0));

		return 0.5 * Square((g - c) / (g + c)) * (1 + Square(c * (g + c) - 1) / Square(c * (g - c) + 1));
	}

	double FresnelConductor(Vec3D v_incomingDirection, Vec3D v_normal, double refractionIndex1, double refractionIndex2, double extinctionCoefficient)
	{
		// reference for this can be found here: https://seblagarde.wordpress.com/2013/04/29/memo-on-fresnel-equations/

		double eta2 = Square(refractionIndex2 / refractionIndex1);
		double etak2 = Square(extinctionCoefficient / refractionIndex1);

		double cosTheta = DotProduct3D(v_incomingDirection, v_normal);
		double cosTheta2 = cosTheta * cosTheta;

		double sinTheta2 = 1 - cosTheta2;
		double sinTheta4 = sinTheta2 * sinTheta2;

		double sumA2B2 = sqrt(Square(eta2 - etak2 - sinTheta2) + 4 * eta2 * etak2);

		double a = sqrt(0.5 * (sumA2B2 + eta2 - etak2 - sinTheta2));

		double sPolarizedReflection = (sumA2B2 - 2 * a * cosTheta + cosTheta2) / (sumA2B2 + 2 * a * cosTheta + cosTheta2);
		double pPolarizedReflection = sPolarizedReflection * (cosTheta2 * sumA2B2 - 2 * a * cosTheta * sinTheta2 + sinTheta4) / (cosTheta2 * sumA2B2 + 2 * a * cosTheta * sinTheta2 + sinTheta4);

		return 0.5 * (sPolarizedReflection + pPolarizedReflection);
	}

	double GeometryBidirectional(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, Vec3D v_microscopicNormal, double roughness)
	{
		return GeometryMonodirectional(v_incomingDirection, v_normal, v_microscopicNormal, roughness) * GeometryMonodirectional(v_outgoingDirection, v_normal, v_microscopicNormal, roughness);
	}

	double GeometryMonodirectional(Vec3D vec, Vec3D v_normal, Vec3D v_microscopicNormal, double roughness)
	{
		double VecDotNormal = DotProduct3D(vec, v_normal);
		double VecDotNormal2 = VecDotNormal * VecDotNormal;
		double a2 = VecDotNormal2 / (roughness * roughness * (1 - VecDotNormal2)); // a squared

		return Chi(DotProduct3D(vec, v_microscopicNormal) / DotProduct3D(vec, v_normal)) * 2 / (1 + sqrt(1 + 1 / a2));
	}

	Vec3D BTDF(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, Vec3D v_microscopicNormal, double refractionIndex1, double refractionIndex2, double roughness)
	{
		double btdf = Abs(DotProduct3D(v_incomingDirection, v_microscopicNormal)) * GeometryBidirectional(v_incomingDirection, v_outgoingDirection, v_normal, v_microscopicNormal, roughness) /
			(Abs(DotProduct3D(v_incomingDirection, v_normal)) * Abs(DotProduct3D(v_microscopicNormal, v_normal)));

		return { btdf, btdf, btdf };
	}

	// computing the bisector vector (microscopic normal) used for importance sampling
	Vec3D MicroscopicNormal(Vec3D v_incomingDirection, Vec3D v_normal, double roughness, std::mt19937* randomEngine)
	{
		double randVariable = uniform_zero_to_one(*randomEngine);

		double cosTheta = sqrt((1 - randVariable) / (randVariable * (roughness * roughness - 1) + 1));
		double sinTheta = sqrt(1 - cosTheta * cosTheta);

		double randAngle = uniform_zero_to_one(*randomEngine) * TAU;

		Vec3D v_bisectorVector = { sinTheta * cos(randAngle), cosTheta, sinTheta * sin(randAngle) };

		Vec3D v_tangent = ReturnNormalizedVec3D(SubtractVec3D(v_incomingDirection, VecScalarMultiplication3D(v_normal, DotProduct3D(v_incomingDirection, v_normal))));

		Matrix3D transformationMatrix =
		{
			v_tangent,
			v_normal,
			CrossProduct(v_normal, v_tangent)
		};

		return VecMatrixMultiplication3D(v_bisectorVector, transformationMatrix);
	}

	Vec3D CalculateLighting_DistributionTracing(Vec3D v_objectColor, Material material, Vec3D v_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, int i_bounceCount, std::mt19937* randomEngine)
	{
		Vec3D v_pixelColor = ZERO_VEC3D;

		if (i_bounceCount > MAX_BOUNCES)
			return v_pixelColor;

		// Temporary until refraction (it'll need to decide whether to offset in or out)
		AddToVec3D(&v_intersection, VecScalarMultiplication3D(v_surfaceNormal, MOVE_DISTANCE));

		Vec3D v_emittedColor = ConusProduct(v_objectColor, material.emittance);

		// Reflections
		Vec3D v_reflectionIntersection, v_reflectionColor, v_nextObjectColor;
		Quaternion q_reflectionIntersectionNormal;
		Material newMaterial;

#if REFLECTIONS == 1
		Vec3D v_specularDirection = SubtractVec3D(v_incomingDirection, VecScalarMultiplication3D(v_surfaceNormal, 2 * DotProduct3D(v_incomingDirection, v_surfaceNormal)));

		if (material.roughness == 0)
		{
			// Specular reflections
			bool b_foundIntersection = NextIntersection(v_intersection, v_specularDirection, &v_reflectionIntersection, &v_nextObjectColor, &q_reflectionIntersectionNormal, &newMaterial);

			if (b_foundIntersection)
			{
				v_reflectionColor = CalculateLighting_DistributionTracing(
					v_nextObjectColor, newMaterial, q_reflectionIntersectionNormal.vecPart, v_specularDirection, v_intersection, i_bounceCount + 1, randomEngine
				);
			}
		}
		else
		{
			// Diffuse reflections
			int hitCount = 1;

			for (int i = 0; i < SAMPLES_PER_RAY; i++)
			{
				Vec3D v_lambertianDirection = ReturnNormalizedVec3D(RandomVec_InUnitSphere(randomEngine));
				if (DotProduct3D(v_lambertianDirection, v_surfaceNormal) < 0)
					v_lambertianDirection = VecScalarMultiplication3D(v_lambertianDirection, -1);

				Vec3D v_diffuseDirection = Lerp3D(v_specularDirection, v_lambertianDirection, material.roughness);

				bool b_foundIntersection = NextIntersection(v_intersection, v_diffuseDirection, &v_reflectionIntersection, &v_nextObjectColor, &q_reflectionIntersectionNormal, &newMaterial);

				if (b_foundIntersection)
				{
					AddToVec3D(&v_reflectionColor, CalculateLighting_DistributionTracing(
						v_nextObjectColor, newMaterial, q_reflectionIntersectionNormal.vecPart, v_diffuseDirection, v_intersection, i_bounceCount + 1, randomEngine
					));
					hitCount++;
				}
			}

			v_reflectionColor = VecScalarMultiplication3D(v_reflectionColor, 1 / hitCount);
		}
#endif

		Vec3D v_lightingColor = ZERO_VEC3D;

		// Soft shadows
		for (int i = 0; i < g_lights.size(); i++)
		{
			double notBlockedProportion = 0;

			for (int j = 0; j < SAMPLES_PER_RAY; j++)
			{
				Vec3D v_displacement = ReturnNormalizedVec3D(RandomVec_InUnitSphere(randomEngine));
				v_displacement = VecScalarMultiplication3D(v_displacement, g_lights[i].radius);
				Vec3D randomPointLight = AddVec3D(g_lights[i].coords, v_displacement);

				Vec3D v_newDirection = ReturnNormalizedVec3D(SubtractVec3D(randomPointLight, v_intersection));

				notBlockedProportion += !IsRayBlocked(v_intersection, v_newDirection, g_lights[i].coords);
			}

			notBlockedProportion /= SAMPLES_PER_RAY;

			Vec3D v_lightColor;

			if (notBlockedProportion != 0)
			{
				float distance = Distance3D(v_intersection, g_lights[i].coords) - g_lights[i].radius;

				v_lightColor = VecScalarMultiplication3D(g_lights[i].emittance, notBlockedProportion / (distance * distance));
			}
			else
				v_lightColor = { 0.10, 0.13, 0.20 };
				//v_lightColor = { 0, 0, 0 };
			
			AddToVec3D(&v_lightingColor, v_lightColor);
		}


		// Refraction
		// Tangent inside of the plane defined by v_surfaceNormal and v_incomingDirection
		//Vec3D v_surfaceTangent = CrossProduct(ReturnNormalizedVec3D(CrossProduct(v_surfaceNormal, v_incomingDirection)), v_surfaceNormal);

		//float sinIncomingAngle = DotProduct3D(v_incomingDirection, v_surfaceTangent);

		//float sinRefractedAngle = Min(REFRACTION_INDEX_AIR * sinIncomingAngle / material.refractionIndex, 1.0f);

		//float cosRefractedAngle = sqrt(1 - sinRefractedAngle * sinRefractedAngle); // Pythagorean identity

		float cosIncomingAngle = Clamp(-DotProduct3D(v_incomingDirection, v_surfaceNormal), 0, 1);

		//bool b_foundIntersection = FindIntersection(v_intersection, v_reflectedDirecion, &v_reflectionIntersection, &v_reflectionColor, &q_reflectionIntersectionNormal, &newMaterial);

		//if (b_foundIntersection)
		//	v_reflectionColor = CalculateLighting_DistributionTracing(
		//		v_reflectionColor, newMaterial, q_reflectionIntersectionNormal.vecPart, v_reflectedDirecion, v_intersection, ++i_bounceCount
		//	);

		// Schlick's approximation
		float r0 = Square((REFRACTION_INDEX_AIR - material.refractionIndex) / (REFRACTION_INDEX_AIR + material.refractionIndex));
		float fresnel = r0 + (1 - r0) * (1 - cosIncomingAngle) * (1 - cosIncomingAngle) * (1 - cosIncomingAngle) * (1 - cosIncomingAngle) * (1 - cosIncomingAngle);

		v_pixelColor = ConusProduct(material.diffuseTint, AddVec3D(VecScalarMultiplication3D(v_reflectionColor, fresnel), VecScalarMultiplication3D(v_objectColor, 1 - fresnel)));

		v_pixelColor = ConusProduct(v_pixelColor, v_lightingColor);

		AddToVec3D(&v_pixelColor, ConusProduct(v_objectColor, material.emittance));
		
		return v_pixelColor;
	}

	Vec3D RandomVec_InUnitSphere(std::mt19937* randomEngine)
	{
		Vec3D randPoint;

		do
		{
			double randX = uniformDistribution(*randomEngine);
			double randY = uniformDistribution(*randomEngine);
			double randZ = uniformDistribution(*randomEngine);

			randPoint = { randX, randY, randZ };
		} while (VecLengthSquared(randPoint) > 1);

		return randPoint;
	}
};

__global__ void cum()
{

}

int main()
{
	cum<<<1, 1>>>();

	Engine rayTracer;

	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}

#include "Controlls.h"