#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define RAY_TRACER
#define PATH_TRACING 1

// Startup settings (cannot be changed during runtime)
#define ASYNC 1
#define THREAD_COUNT 4
#define SCREEN_WIDTH 900
#define SCREEN_HEIGHT 720
#define TOUCHING_DISTANCE 0.01f
#define OFFSET_DISTANCE 0.00001f
#define SAMPLES_PER_PIXEL 100 // for path tracing
#define GAUSSIAN_BLUR 1 // blur for denoising
#define MEDIAN_FILTER 0 // used for firefly reduction and denoising, bad for low spp
#define MAX_COLOR_VALUE 1000000 // used for reducing fireflies, introduces bias
#define MAX_BOUNCES 15 // For distribution ray tracing
#define SAMPLES_PER_RAY 1 // for distribution ray tracing
#define WHITE_COLOR { 255, 255, 255 }
#define REFRACTION_INDEX_AIR 1

#include <iostream>
#include <random>
#include <future>

#include <hip/hip_runtime.h>
#include <>

#include "olcPixelGameEngine.h"

#include "MathUtilities.cuh"
#include "WorldDatatypes.h"
#include "ParseOBJ.h"

// Global variables

Vec3D screenBuffer[SCREEN_HEIGHT * SCREEN_WIDTH];

Player g_player;

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;
std::vector<Light> g_lights;

Ground g_ground;

// Textures
olc::Sprite* g_basketball_texture;
olc::Sprite* g_planks_texture;
olc::Sprite* g_concrete_texture;
olc::Sprite* g_tiledfloor_texture;
olc::Sprite* g_worldmap_texture;
olc::Sprite* g_bricks_texture;

olc::Sprite* g_basketball_normalmap;
olc::Sprite* g_planks_normalmap;
olc::Sprite* g_concrete_normalmap;
olc::Sprite* g_tiledfloor_normalmap;
olc::Sprite* g_worldmap_normalmap;
olc::Sprite* g_bricks_normalmap;

std::random_device seedEngine;
std::uniform_real_distribution<> uniformDistribution(-1, 1);
std::uniform_real_distribution<> uniform_zero_to_one(0, 1);

// Ingame options (can be changed during runtime)
namespace Options
{
	bool mcControls = true;
}

class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

	bool OnUserCreate() override
	{
		g_player = { { 1.5, 1.5, -2.064 }, { 1, ZERO_VEC3D }, TAU * 0.2f };

		g_basketball_texture = new olc::Sprite("../Assets/basketball.png");
		g_planks_texture = new olc::Sprite("../Assets/planks.png");
		g_concrete_texture = new olc::Sprite("../Assets/concrete.png");
		g_tiledfloor_texture = new olc::Sprite("../Assets/tiledfloor.png");
		g_worldmap_texture = new olc::Sprite("../Assets/worldmap.png");
		g_bricks_texture = new olc::Sprite("../Assets/bricks.png");

		g_basketball_normalmap = new olc::Sprite("../Assets/basketball_normalmap.png");
		g_planks_normalmap = new olc::Sprite("../Assets/planks_normalmap.png");
		g_concrete_normalmap = new olc::Sprite("../Assets/concrete_normalmap.png");
		g_tiledfloor_normalmap = new olc::Sprite("../Assets/tiledfloor_normalmap.png");
		g_worldmap_normalmap = new olc::Sprite("../Assets/tiledfloor_normalmap.png");
		g_bricks_normalmap = new olc::Sprite("../Assets/bricks_normalmap.png");

		g_spheres =
		{
			// Lightsource
			{ { 1.5, 3, 1.5 }, 0.5, { { 45, 40, 30 }, { 0.9, 0.7, 0.1 }, { 0.9, 0.7, 0.1 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Glossy ball
			{ { 1.5, 1.4, 1.5 }, 0.4, { { 0, 0, 0 }, { 0.9, 0.9, 0.9 }, { 0.9, 0.9, 0.9 }, 0.05, 12.5, { 500, 500, 500 } } },
			// Other lightsource
			{ { 0.6, 0.3, 0.85 }, 0.3, { { 30, 5, 10 }, { 0.9, 0.2, 0.4 }, { 0.9, 0.2, 0.4 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Other lightsource
			{ { 1.9, 0.3, 0.5 }, 0.3, { { 2.25, 13.1, 18.7 }, { 0.9, 0.2, 0.4 }, { 0.9, 0.2, 0.4 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Refractive ball
			{ { 2.5, 0.5, 2.2 }, 0.5, { { 0, 0, 0 }, { 0.2, 0.2, 0.2 }, { 0.2, 0.2, 0.2 }, 0.1, 1.52, { 0, 0, 0 } } }
			// Other Refractive ball
			//{ { 1.5, 2.3, 0.3 }, 0.5, { { 0, 0, 0 }, { 0.2, 0.2, 0.2 }, { 0.2, 0.2, 0.2 }, 0.3, 1.52, { 0, 0, 0 } } }
			// Basket ball
			//{ { 2.5, 0.5, 0.8 }, 0.5, { 1, 1, 1 }, { 0.2, 0.6, 0.8, 0.9, { -1, 0, 0 }, 500, 2 }, g_basketball_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ 1, 0, 1 }), PI / 2) },
			// World atlas globe
			//{ { 1.75, 0.3, 0.5 }, 0.3, { 1, 1, 1 }, { 0.35, 0.7, 0.7, 0.9, { 1, 0, 0 }, 500, 1.45 }, g_worldmap_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ -1, 0.5, -2 }), PI / 2) },
			// Magenta lightsource
			//{ { 0.5, 0.4, 0.8 }, 0.4, { 1, 0.2, 0.4157 }, { 35, 0.2, 0.5, 0.95, { -1, 0, 0 }, 500, 1.6 } },
			// Refractive ball
			//{ { 1.1, 0.3, 0.4 }, 0.3, { 1, 1, 1 }, { 0.2, 0.2, 0.2, 0.95, { 1, 0, 0 }, 0.5, 1.4 } },
			// Cyan lightsource
			//{ { 2.4, 0.3, 1.75 }, 0.3, { 0.3, 1.15, 1.15 }, { 45, 0.2, 0.5, 0.95, { 1, 0, 0 }, 500, 1.6 } }
		};

		g_triangles =
		{
			// Walls first face
			{ { { 0, 0, 3 }, { 0, 3, 3 }, { 3, 3, 3 } }, { { 0, 0, 0 }, { 0.3, 0.2, 0.2 }, { 0.3, 0.2, 0.2 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_bricks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_bricks_normalmap },
			{ { { 0, 0, 3 }, { 3, 3, 3 }, { 3, 0, 3 } }, { { 0, 0, 0 }, { 0.3, 0.2, 0.2 }, { 0.3, 0.2, 0.2 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_bricks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_bricks_normalmap },
			// Walls second face														   
			{ { { 0, 0, 0 }, { 0, 3, 0 }, { 0, 3, 3 } }, { { 0, 0, 0 }, { 0.2, 0.4, 0.4 }, { 0.2, 0.4, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap },
			{ { { 0, 0, 0 }, { 0, 3, 3 }, { 0, 0, 3 } }, { { 0, 0, 0 }, { 0.2, 0.4, 0.4 }, { 0.2, 0.4, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap },
			// Walls third face															   
			{ { { 3, 0, 3 }, { 3, 3, 3 }, { 3, 3, 0 } }, { { 0, 0, 0 }, { 0.4, 0.2, 0.4 }, { 0.4, 0.2, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap },
			{ { { 3, 0, 3 }, { 3, 3, 0 }, { 3, 0, 0 } }, { { 0, 0, 0 }, { 0.4, 0.2, 0.4 }, { 0.4, 0.2, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap },
			// Walls fourth face														   
			{ { { 0, 3, 0 }, { 3, 3, 3 }, { 0, 3, 3 } }, { { 0, 0, 0 }, { 0.3, 0.3, 0.3 }, { 0.3, 0.3, 0.3 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap },
			{ { { 0, 3, 0 }, { 3, 3, 0 }, { 3, 3, 3 } }, { { 0, 0, 0 }, { 0.3, 0.3, 0.3 }, { 0.3, 0.3, 0.3 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap },

			// Box first face															   
			{ { { 1, 0, 2 }, { 2, 1, 2 }, { 1, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 2 }, { 2, 0, 2 }, { 2, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box second face											  				     
			{ { { 1, 0, 1 }, { 1, 1, 1 }, { 2, 1, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 1 }, { 2, 1, 1 }, { 2, 0, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box third face											 				    
			{ { { 1, 0, 1 }, { 1, 1, 2 }, { 1, 1, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 1 }, { 1, 0, 2 }, { 1, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box fourth face							   				  				     
			{ { { 2, 0, 1 }, { 2, 1, 1 }, { 2, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 2, 0, 1 }, { 2, 1, 2 }, { 2, 0, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box fifth face							   				  				     
			{ { { 1, 1, 1 }, { 1, 1, 2 }, { 2, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 1, 1 }, { 2, 1, 2 }, { 2, 1, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },

			// refractive pyramid
			/*{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.1, 0 + 0.01, 2.9 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.1, 0 + 0.01, 2.9 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.1, 0 + 0.01, 2.1 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.9, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.1 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.9, 0 + 0.01, 2.9 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.1, 0 + 0.01, 2.9 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.9, 0 + 0.01, 2.1 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.52 } },

			// other refractive pyramid
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.1 + 2, 0 + 0.01, 2.9 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.1 + 2, 0 + 0.01, 2.9 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.1 + 2, 0 + 0.01, 2.1 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.9 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.1 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.9 + 2, 0 + 0.01, 2.9 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.1 + 2, 0 + 0.01, 2.9 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.9 + 2, 0 + 0.01, 2.1 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.7 } }*/
		};

		g_ground = { 0, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.6, 2, 500 }, g_tiledfloor_texture, { 0, 0 }, { 1, 1 }, 1, g_tiledfloor_normalmap };

		g_lights =
		{
			{ { 1.5, 3, 1.5 }, 0.5, 500, { 1, 0.8, 0.6 } }
		};

#if ASYNC == 1
	//std::async(std::launch::async, ImportScene, &g_triangles, "../Assets/RubberDuck.obj", 0.4, Vec3D({ 0.8, 0.5, 0.5 }));
#else
	//ImportScene(&g_triangles, "../Assets/RubberDuck.obj", 0.4, { 0.8, 0.5, 0.5 });
#endif

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		Timer timer("Rendering");

		Controlls(fElapsedTime);

		StartThreads();

#if GAUSSIAN_BLUR == 1
		GaussianBlur();
#endif

#if MEDIAN_FILTER == 1
		MedianFiltering();
#endif

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				Vec3D pixelColor = screenBuffer[y * SCREEN_WIDTH + x];

				Draw(x, y, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
		}

		std::cout << "\a" << std::endl;

		return true;
	}

	void StartThreads()
	{
#if ASYNC == 1
		// Screen split up into 4 quadrants running in parallell on seperate threads

		std::future<void> returnValues[THREAD_COUNT];

		for (int i = 0; i < THREAD_COUNT; i++)
		{
			int startX = i * ceil(SCREEN_WIDTH / double(THREAD_COUNT));
			int endX = (i + 1) * ceil(SCREEN_WIDTH / double(THREAD_COUNT));

			if (startX >= SCREEN_WIDTH)
			{
				break;
			}

			endX = Min(endX, SCREEN_WIDTH);

			std::mt19937 randomEngine(seedEngine());

			returnValues[i] = std::async(std::launch::async, &Engine::RayTracing, this, startX, endX, randomEngine);
		}
#else
		std::mt19937 randomEngine(seedEngine());
		RayTracing(0, SCREEN_WIDTH, randomEngine);
#endif
	}

private:
	// Defined in Controlls.h
	void Controlls(float fElapsedTime);

	void RayTracing(int startX, int endX, std::mt19937 randomEngine)
	{
		const double zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (double y = -SCREEN_HEIGHT * 0.5f + 0.5f; y < SCREEN_HEIGHT * 0.5f + 0.5f; y++)
		{
			for (double x = -SCREEN_WIDTH * 0.5f + 0.5f + startX; x < -SCREEN_WIDTH * 0.5f + 0.5f + endX; x++)
			{
				Vec3D v_direction = { x, y, zFar };

				Vec3D v_orientedDirection = QuaternionMultiplication(g_player.q_orientation, { 0, v_direction }, QuaternionConjugate(g_player.q_orientation)).vecPart;

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = SCREEN_HEIGHT - (y + SCREEN_HEIGHT * 0.5f);

				Vec3D pixelColor = ZERO_VEC3D;

				for (int i = 0; i < SAMPLES_PER_PIXEL; i++)
				{
					// For anti-aliasing
					Vec3D v_jitteredDirection = AddVec3D(v_orientedDirection, RandomVec_InUnitSphere(&randomEngine));

					NormalizeVec3D(&v_jitteredDirection);

					AddToVec3D(&pixelColor, RenderPixel(g_player.coords, v_jitteredDirection, &randomEngine));
				}

				ScaleVec3D(&pixelColor, 1 / double(SAMPLES_PER_PIXEL));

				pixelColor.x = Min(pixelColor.x, 255.0f);
				pixelColor.y = Min(pixelColor.y, 255.0f);
				pixelColor.z = Min(pixelColor.z, 255.0f);

				ScaleVec3D(&pixelColor, 1.0f / 255);

				pixelColor = { LINEAR_TO_SRGB(pixelColor.x), LINEAR_TO_SRGB(pixelColor.y), LINEAR_TO_SRGB(pixelColor.z) };

				ScaleVec3D(&pixelColor, 255.0f);

				screenBuffer[screenY * SCREEN_WIDTH + screenX] = pixelColor;

				//Draw(screenX, screenY, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
#if PATH_TRACING == 1
			std::cout << ((y + SCREEN_HEIGHT * 0.5f) / SCREEN_HEIGHT) * 100 << "%" << std::endl;
#endif
		}
	}

	Vec3D RenderPixel(Vec3D v_start, Vec3D v_direction, std::mt19937* randomEngine)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_textureColor = ZERO_VEC3D;
		Quaternion q_surfaceNormal = IDENTITY_QUATERNION;
		Material material;

		bool intersectionExists = NextIntersection(v_start, v_direction, &v_intersection, &v_textureColor, &q_surfaceNormal, &material);

		if (intersectionExists)
		{
#if PATH_TRACING == 1
			v_textureColor = CalculateLighting_PathTracing(
				v_textureColor, material, q_surfaceNormal, v_direction, v_intersection, { 1, 1, 1 }, randomEngine
			);
#else
			v_textureColor = CalculateLighting_DistributionTracing(
				v_textureColor, material, q_surfaceNormal.vecPart, v_direction, v_intersection, 0
			);
#endif
		}

		return v_textureColor;
	}

	void MedianFilter()
	{
		auto AddColorToVector = [](std::vector<Vec3D>* colors, int x, int y)
		{
			if (x >= 0 && x < SCREEN_WIDTH && y >= 0 && y < SCREEN_HEIGHT)
			{
				colors->push_back(screenBuffer[y * SCREEN_WIDTH + x]);
			}
		};

		auto ColorSize = [](Vec3D color)
		{
			return Max(color.x, Max(color.y, color.z));
		};

		auto MedianColor = [ColorSize](std::vector<Vec3D>* colors)
		{
			for (int i = 0; i < colors->size(); i++)
			{
				for (int j = 0; j < colors->size() - 1; j++)
				{
					if (ColorSize(colors->at(j)) > ColorSize(colors->at(j + 1)))
					{
						SwapVec3D(&(colors->at(j)), &(colors->at(j + 1)));
					}
				}
			}

			return colors->at(colors->size() / 2);
		};

		Vec3D* screenBufferCopy = new Vec3D[SCREEN_HEIGHT * SCREEN_WIDTH];

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				std::vector<Vec3D> colors;

				AddColorToVector(&colors, x, y);
				AddColorToVector(&colors, x + 1, y);
				AddColorToVector(&colors, x - 1, y);
				AddColorToVector(&colors, x, y + 1);
				AddColorToVector(&colors, x, y - 1);

				screenBufferCopy[y * SCREEN_WIDTH + x] = MedianColor(&colors);
			}
		}

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				screenBuffer[y * SCREEN_WIDTH + x] = screenBufferCopy[y * SCREEN_WIDTH + x];
			}
		}

		delete[] screenBufferCopy;
	}

	void GaussianBlur()
	{
		auto WeightedPixel = [](double weight, int x, int y)
		{
			Vec3D weightedPixel = ZERO_VEC3D;

			if (x >= 0 && x < SCREEN_WIDTH && y >= 0 && y < SCREEN_HEIGHT)
			{
				weightedPixel = VecScalarMultiplication3D(screenBuffer[y * SCREEN_WIDTH + x], weight);
			}

			return weightedPixel;
		};

#define KERNEL_SIZE 3

		double gaussianKernel[KERNEL_SIZE * KERNEL_SIZE] =
		{
			0.0000, 0.0625, 0.0000,
			0.0625, 0.7500, 0.0625,
			0.0000, 0.0625, 0.0000,
		};

		Vec3D* screenBufferCopy = new Vec3D[SCREEN_HEIGHT * SCREEN_WIDTH];

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				Vec3D blurredPixel = ZERO_VEC3D;

				for (int i = 0; i < 3; i++)
				{
					for (int j = 0; j < 3; j++)
					{
						AddToVec3D(&blurredPixel, WeightedPixel(gaussianKernel[i * KERNEL_SIZE + j], x + j - KERNEL_SIZE / 2, y + i - KERNEL_SIZE / 2));
					}
				}

				screenBufferCopy[y * SCREEN_WIDTH + x] = blurredPixel;
			}
		}

		for (int y = 0; y < SCREEN_HEIGHT; y++)
		{
			for (int x = 0; x < SCREEN_WIDTH; x++)
			{
				screenBuffer[y * SCREEN_WIDTH + x] = screenBufferCopy[y * SCREEN_WIDTH + x];
			}
		}

		delete[] screenBufferCopy;
	}

	double LINEAR_TO_SRGB(double l)
	{
		if (l <= 0.0031308)
		{
			return l * 12.92;
		}

		return 1.055 * pow(l, 0.41666) - 0.055;
	}

	bool GroundIntersection_RT(Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		if (v_direction.y >= 0 || v_start.y < g_ground.level)
		{
			return false;
		}

		ScaleVec3D(&v_direction, (g_ground.level - v_start.y) / v_direction.y);

		Vec3D rayGroundIntersection = AddVec3D(v_start, v_direction);

		if (v_intersection != nullptr)
		{
			*v_intersection = rayGroundIntersection;
		}

		if (q_surfaceNormal != nullptr)
		{
			*q_surfaceNormal = { 1, { 0, 1, 0 } };
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (g_ground.texture != nullptr || g_ground.normalMap != nullptr)
		{
			double signedTextureWidth = (g_ground.textureCorner2.x - g_ground.textureCorner1.x) * g_ground.textureScalar;
			double signedTextureHeight = (g_ground.textureCorner2.y - g_ground.textureCorner1.y) * g_ground.textureScalar;

			double t1 = fmod(rayGroundIntersection.x, signedTextureWidth) / signedTextureWidth;
			double t2 = fmod(rayGroundIntersection.z, signedTextureHeight) / signedTextureHeight;

			// if the t values are negative, we need to flip them around the center of the texture and make them positive
			if (t1 < 0) t1 += 1;
			if (t2 < 0) t2 += 1;

			double textureX = Lerp(g_ground.textureCorner1.x, g_ground.textureCorner2.x, t1);
			double textureY = Lerp(g_ground.textureCorner1.y, g_ground.textureCorner2.y, t2);

			if (g_ground.texture != nullptr)
			{
				olc::Pixel texelColor = g_ground.texture->Sample(textureX, textureY);

				*v_intersectionColor = { double(texelColor.r), double(texelColor.g), double(texelColor.b) };
			}
			if (g_ground.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = g_ground.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				q_surfaceNormal->vecPart = ReturnNormalizedVec3D({ double(normalMapColor.r) * 2 - 255.0f, double(normalMapColor.b) * 2 - 255.0f, double(normalMapColor.g) * 2 - 255.0f });
			}
		}

		return true;
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		double dxdz = v_direction.x / v_direction.z;
		double dydz = v_direction.y / v_direction.z;

		double a = dxdz * dxdz + dydz * dydz + 1;
		
		double b = 
			2 * dxdz * (v_start.x - sphere.coords.x) +
			2 * dydz * (v_start.y - sphere.coords.y) +
			2 * (v_start.z - sphere.coords.z);

		double c = 
			(v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) +
			(v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) +
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// ISAK: There wasn't any need to recalculate this multiple times
		double rootContent = b * b - 4 * a * c;

		// There exists no intersections (no real answer)
		if (rootContent < 0) return false;

		double z1 = (-b + sqrt(rootContent)) / (2 * a);
		double z2 = (-b - sqrt(rootContent)) / (2 * a);

		Vec3D v_alternative1 = { z1 * dxdz, z1 * dydz, z1 };
		AddToVec3D(&v_alternative1, v_start);

		Vec3D v_alternative2 = { z2 * dxdz, z2 * dydz, z2 };
		AddToVec3D(&v_alternative2, v_start);

		// Check which intersection is the closest and choose that one
		double dist1 = DistanceSquared3D(v_alternative1, v_start);
		double dist2 = DistanceSquared3D(v_alternative2, v_start);

		bool dist1Closest = dist1 < dist2;

		Vec3D v_correctHit = dist1Closest ? v_alternative1 : v_alternative2;
		Vec3D v_otherHit = dist1Closest ? v_alternative2 : v_alternative1;

		// Check if the intersection is behind the ray. If so, choose the other one.
		if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0)
		{
			v_correctHit = v_otherHit;

			// Check if the other intersection is behind the ray. If so, discard it.
			if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0) return false;
		}

		// Checks whether or not to return the intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_correctHit;
		}

		// Calculating the normal of the sphere (without normalmap)
		Vec3D v_normal = SubtractVec3D(v_correctHit, sphere.coords);
		NormalizeVec3D(&v_normal);

		if (q_surfaceNormal != nullptr)
		{
			q_surfaceNormal->vecPart = v_normal;
			q_surfaceNormal->realPart = 1;

			if (DistanceSquared3D(v_start, sphere.coords) < sphere.radius * sphere.radius)
			{
				q_surfaceNormal->realPart = -1;
			}
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (sphere.texture != nullptr || sphere.normalMap != nullptr)
		{
			Vec3D i_Hat = { 1, 0, 0 };
			Vec3D j_Hat = { 0, 1, 0 };
			Vec3D k_Hat = { 0, 0, 1 };

			// Rotating axies by sphere rotation quaternion
			i_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, i_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			j_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, j_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			k_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, k_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;

			// Translate normal into new coordinate system
			v_normal = { DotProduct3D(v_normal, i_Hat), DotProduct3D(v_normal, j_Hat), DotProduct3D(v_normal, k_Hat) };

			// UV coordinates
			double u = 0.5 + atan2(v_normal.x, v_normal.z) / TAU;
			double v = 0.5 - asin(v_normal.y) / PI;

			double textureX = Lerp(sphere.textureCorner1.x, sphere.textureCorner2.x, u);
			double textureY = Lerp(sphere.textureCorner1.y, sphere.textureCorner2.y, v);

			if (sphere.texture != nullptr)
			{
				// Interpolating between assigned texture coordinates
				olc::Pixel texelColor = sphere.texture->Sample(textureX, textureY);

				*v_intersectionColor = { (double)texelColor.r, (double)texelColor.g, (double)texelColor.b };
			}
			if (sphere.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = sphere.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ double(normalMapColor.r) * 2 - 255.0f, double(normalMapColor.b) * 2 - 255.0f, double(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the sphere
				Vec3D v_sidewaysTangent = ReturnNormalizedVec3D({ -v_normal.z, 0, v_normal.x });
				Vec3D v_forwardTangent = ReturnNormalizedVec3D({ 0, -v_normal.z, v_normal.y });

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					v_sidewaysTangent,
					v_normal,
					v_forwardTangent
				};

				q_surfaceNormal->vecPart = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		return true;
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		double f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		if (DotProduct3D(SubtractVec3D(v_trianglePlaneIntersection, v_start), v_direction) < 0) return false;

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if ((DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) ||
			(DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) ||
			(DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0))
		{
			return false;
		}

		// Checks whether or not to return the intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_trianglePlaneIntersection;
		}

		if (q_surfaceNormal != nullptr)
		{
			q_surfaceNormal->vecPart = v_triangleNormal;

			q_surfaceNormal->realPart = 1;

			if (DotProduct3D(v_triangleNormal, v_direction) > 0)
			{
				// The triangle face is inside of the mesh, so the normal must be flipped
				q_surfaceNormal->realPart = -1;
			}
		}
		
		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (triangle.texture != nullptr || triangle.normalMap != nullptr)
		{
			// from here on we calculate the texture coordinates

			Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

			Matrix3D triangleMatrix =
			{
				v_triangleEdge1,
				v_triangleEdge2,
				v_triangleNormal
			};

			Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

			Vec2D textureCoordinates = { 0, 0 };

			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]), triangleEdgeScalars.x));
			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]), triangleEdgeScalars.y));
			AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

			if (triangle.texture != nullptr)
			{
				olc::Pixel texelColor = triangle.texture->Sample(textureCoordinates.x, textureCoordinates.y);

				*v_intersectionColor = { double(texelColor.r), double(texelColor.g), double(texelColor.b) };
			}
			if (triangle.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = triangle.normalMap->Sample(textureCoordinates.x, textureCoordinates.y);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ double(normalMapColor.r) * 2 - 255.0f, double(normalMapColor.b) * 2 - 255.0f, double(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the triangle for finding the normal in object space

				// { u1, v1 }, { u2, v2 }, { u3, v3 } are the normalMapVertices
				// T is the tangent
				// B is the bitangent
				
				//                       | T.x  B.x  0 |   
				// { v_triangleEdge1 } = | T.y  B.y  0 | * { u2 - u1, v2 - v1, 0 }
				//                       | T.z  B.z  0 |   

				//                       | T.x  B.x  0 |   
				// { v_triangleEdge2 } = | T.y  B.y  0 | * { u3 - u1, v3 - v1, 0 }
				//                       | T.z  B.z  0 |   

				// | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | T.x  B.x  0 |   | u2 - u1  u3 - u1  0 |
				// | v_triangleEdge1.y  v_triangleEdge2.y  0 | = | T.y  B.y  0 | * | v2 - v1  v3 - v1  0 |
				// | v_triangleEdge1.z  v_triangleEdge2.z  0 |   | T.z  B.z  0 |   |    0        0     1 |

				//                                                                                       -1
				// | T.x  B.x  0 |   | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | u2 - u1  u3 - u1  0 |
				// | T.y  B.y  0 | = | v_triangleEdge1.y  v_triangleEdge2.y  0 | * | v2 - v1  v3 - v1  0 |
				// | T.z  B.z  0 |	 | v_triangleEdge1.z  v_triangleEdge2.z  0 |   |    0        0     1 |

				Matrix3D m1 =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					ZERO_VEC3D
				};

				Matrix3D m2 =
				{
					{ triangle.textureVertices[1].x - triangle.textureVertices[0].x, triangle.textureVertices[1].y - triangle.textureVertices[0].y, 0 },
					{ triangle.textureVertices[2].x - triangle.textureVertices[0].x, triangle.textureVertices[2].y - triangle.textureVertices[0].y, 0 },
					{ 0, 0, 1 }
				};

				Matrix3D tangentsMatrix = MatrixMultiplication3D(InverseMatrix3D(m2), m1);

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					ReturnNormalizedVec3D(tangentsMatrix.i_Hat),
					v_triangleNormal,
					ReturnNormalizedVec3D(tangentsMatrix.j_Hat)
				};

				q_surfaceNormal->vecPart = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, double f_planeOffset)
	{
		double f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		double f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	enum ScatteringType
	{
		LAMBERTIAN,
		SPECULAR,
		TRANSMISSIVE
	};

	Vec3D CalculateLighting_PathTracing(Vec3D v_textureColor, Material material, Quaternion q_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, Vec3D accumulatedAttenuation, std::mt19937* randomEngine)
	{
		Vec3D v_outgoingLightColor = ConusProduct(v_textureColor, material.emittance);

		double survivalProbability = Max(Sigmoid(2 * Max(accumulatedAttenuation.x, Max(accumulatedAttenuation.y, accumulatedAttenuation.z))), 0.05);

		// Randomly terminate paths with russian roulette
		if (uniform_zero_to_one(*randomEngine) > survivalProbability)
		{
			return v_outgoingLightColor;
		}

		double refractionIndex1 = REFRACTION_INDEX_AIR;
		double refractionIndex2 = material.refractionIndex;
		Vec3D attenuation = { 0, 0, 0 };

		if (q_surfaceNormal.realPart == -1)
		{
			refractionIndex1 = material.refractionIndex;
			refractionIndex2 = REFRACTION_INDEX_AIR;
			v_outgoingLightColor = ZERO_VEC3D;
		}

		ScaleVec3D(&v_incomingDirection, -1);

		// Scale the normal to be oriented in the hemisphere the material was hit from
		ScaleVec3D(&(q_surfaceNormal.vecPart), q_surfaceNormal.realPart);

		Vec3D v_outgoingDirection;
		ScatteringType scatteringType;

		Vec3D v_microscopicNormal;

		double randNumber = uniform_zero_to_one(*randomEngine);

		if (randNumber < 0.33333)
		{
			scatteringType = LAMBERTIAN;

			Vec3D v_tangent = ReturnNormalizedVec3D(SubtractVec3D(v_incomingDirection, VecScalarMultiplication3D(q_surfaceNormal.vecPart, DotProduct3D(v_incomingDirection, q_surfaceNormal.vecPart))));

			Matrix3D transformationMatrix =
			{
				v_tangent,
				q_surfaceNormal.vecPart,
				CrossProduct(q_surfaceNormal.vecPart, v_tangent)
			};

			double randVariable = uniform_zero_to_one(*randomEngine);
			double theta = uniform_zero_to_one(*randomEngine) * TAU;

			double r = sqrt(randVariable);

			v_outgoingDirection = VecMatrixMultiplication3D({ r * cos(theta), sqrt(1 - randVariable), r * sin(theta) }, transformationMatrix);
		}
		else if (randNumber < 0.66666)
		{
			scatteringType = SPECULAR;

			v_microscopicNormal = MicroscopicNormal(v_incomingDirection, q_surfaceNormal.vecPart, material.roughness, randomEngine);

			v_outgoingDirection = SubtractVec3D(VecScalarMultiplication3D(v_microscopicNormal, 2 * DotProduct3D(v_incomingDirection, v_microscopicNormal)), v_incomingDirection);
		}
		else
		{
			scatteringType = TRANSMISSIVE;

			v_microscopicNormal = MicroscopicNormal(v_incomingDirection, q_surfaceNormal.vecPart, material.roughness, randomEngine);

			double n = refractionIndex1 / refractionIndex2;

			double incomingDotBisector = DotProduct3D(v_incomingDirection, v_microscopicNormal);

			double bisectorScalar = n * incomingDotBisector - Sign(DotProduct3D(v_incomingDirection, q_surfaceNormal.vecPart)) * sqrt(Max(1 + n * (incomingDotBisector * incomingDotBisector - 1), 0));

			v_outgoingDirection = SubtractVec3D(VecScalarMultiplication3D(v_microscopicNormal, bisectorScalar), VecScalarMultiplication3D(v_incomingDirection, n));
		}

		NormalizeVec3D(&v_outgoingDirection);

		AddToVec3D(&v_intersection, VecScalarMultiplication3D(v_outgoingDirection, OFFSET_DISTANCE));

		if (DotProduct3D(v_outgoingDirection, q_surfaceNormal.vecPart) < 0)
		{
			// The ray is going into the object
			attenuation = material.attenuation;
		}

		Vec3D v_nextIntersection = ZERO_VEC3D;
		Vec3D v_nextTextureColor = ZERO_VEC3D;
		Quaternion q_nextNormal = IDENTITY_QUATERNION;
		Material nextMaterial;

		bool intersectionExists = NextIntersection(v_intersection, v_outgoingDirection, &v_nextIntersection, &v_nextTextureColor, &q_nextNormal, &nextMaterial);

		if (intersectionExists)
		{
			Vec3D weight = ZERO_VEC3D;

			Vec3D v_diffuseTint = VecScalarMultiplication3D(ConusProduct(v_textureColor, material.diffuseTint), 1.0f / 255);
			Vec3D v_specularTint = VecScalarMultiplication3D(ConusProduct(v_textureColor, material.specularTint), 1.0f / 255);

			if (scatteringType == LAMBERTIAN)
			{
				weight = VecScalarMultiplication3D(BRDF_LAMBERTIAN(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, refractionIndex1, refractionIndex2, v_diffuseTint), 3 * PI);
			}
			else if (scatteringType == SPECULAR)
			{
				weight = VecScalarMultiplication3D(BRDF_COOKTORRANCE(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, v_microscopicNormal, refractionIndex1, refractionIndex2, material.roughness, v_specularTint), Abs(DotProduct3D(v_outgoingDirection, q_surfaceNormal.vecPart)) * 3 * PI);
			}
			else
			{
				weight = VecScalarMultiplication3D(BTDF(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, v_microscopicNormal, refractionIndex1, refractionIndex2, material.roughness), Abs(DotProduct3D(v_outgoingDirection, q_surfaceNormal.vecPart)) * 3 * PI);
			}

			double distance = Distance3D(v_intersection, v_nextIntersection);

			attenuation = { exp(-attenuation.x * distance), exp(-attenuation.y * distance), exp(-attenuation.z * distance) };

			weight = ConusProduct(weight, attenuation);

			accumulatedAttenuation = ConusProduct(accumulatedAttenuation, weight);

			Vec3D v_incomingLightColor = CalculateLighting_PathTracing(
				v_nextTextureColor, nextMaterial, q_nextNormal, v_outgoingDirection, v_nextIntersection, accumulatedAttenuation, randomEngine
			);

			v_incomingLightColor = { Min(v_incomingLightColor.x, MAX_COLOR_VALUE), Min(v_incomingLightColor.y, MAX_COLOR_VALUE), Min(v_incomingLightColor.z, MAX_COLOR_VALUE) }; // Introduces bias. To avoid bias MAX_COLOR_VALUE should be very high

			// Add the energy that is lost by randomly terminating paths
			ScaleVec3D(&v_incomingLightColor, 1.0 / survivalProbability);

			AddToVec3D(&v_outgoingLightColor, ConusProduct(v_incomingLightColor, weight));
		}

		return v_outgoingLightColor;
	}

	bool NextIntersection(Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection, Vec3D* v_textureColor, Quaternion* q_normal, Material* material)
	{
		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, v_intersection, v_textureColor, q_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_start, v_direction, *v_intersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				*material = g_spheres[i].material;

				return true;
			}
		}

		for (int i = 0; i < g_triangles.size(); i++)
		{
			bool intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, v_intersection, v_textureColor, q_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_start, v_direction, *v_intersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				*material = g_triangles[i].material;

				return true;
			}
		}

		bool intersectionExists = GroundIntersection_RT(v_start, v_direction, v_intersection, v_textureColor, q_normal);

		if (intersectionExists)
		{
			*material = g_ground.material;

			return true;
		}

		return false;
	}

	bool IsRayBlocked(Vec3D v_start, Vec3D v_direction, Vec3D v_intersection)
	{
		Vec3D v_otherIntersection = ZERO_VEC3D;

		for (int j = 0; j < g_spheres.size(); j++)
		{
			bool otherIntersectionExists = SphereIntersection_RT(g_spheres[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		for (int j = 0; j < g_triangles.size(); j++)
		{
			bool otherIntersectionExists = TriangleIntersection_RT(g_triangles[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		bool otherIntersectionExists = GroundIntersection_RT(v_start, v_direction, &v_otherIntersection);

		// If there exists a closer intersection to the ray start vector it means the ray is blocked
		if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
		{
			return true;
		}

		// The ray is not blocked
		return false;
	}

	// Cook-Torrance BRDF with GGX distribution function and GGX geometry function
	Vec3D BRDF_COOKTORRANCE(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, Vec3D v_microscopicNormal, double refractionIndex1, double refractionIndex2, double roughness, Vec3D v_specularTint)
	{
		double fresnelFactor = Fresnel(v_incomingDirection, v_microscopicNormal, refractionIndex1, refractionIndex2);

		// Some terms are not included because they are cancelled out bt the PDF
		double specularTerm = Abs(DotProduct3D(v_incomingDirection, v_microscopicNormal)) * fresnelFactor * GeometryBidirectional(v_incomingDirection, v_outgoingDirection, v_normal, v_microscopicNormal, roughness) /
			(Abs(DotProduct3D(v_incomingDirection, v_normal)) * Abs(DotProduct3D(v_microscopicNormal, v_normal)));

		return VecScalarMultiplication3D(v_specularTint, specularTerm);
	}

	Vec3D BRDF_LAMBERTIAN(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, double refractionIndex1, double refractionIndex2, Vec3D v_diffuseTint)
	{
		Vec3D v_bisectorVector = ReturnNormalizedVec3D(Lerp3D(v_incomingDirection, v_outgoingDirection, 0.5));

		double fresnelFactor = Fresnel(v_incomingDirection, v_bisectorVector, refractionIndex1, refractionIndex2);

		double diffuseTerm = Chi(DotProduct3D(v_bisectorVector, v_normal)) * Square(1 - fresnelFactor) / PI;

		return VecScalarMultiplication3D(v_diffuseTint, diffuseTerm);
	}

	double Chi(double x)
	{
		return x > 0 ? 1 : 0;
	}

	double Fresnel(Vec3D v_incomingDirection, Vec3D v_microscopicNormal, double refractionIndex1, double refractionIndex2)
	{
		double c = Abs(DotProduct3D(v_incomingDirection, v_microscopicNormal));

		double g = sqrt(Max((refractionIndex2 * refractionIndex2) / (refractionIndex1 * refractionIndex1) - 1 + c * c, 0));

		return 0.5 * Square((g - c) / (g + c)) * (1 + Square(c * (g + c) - 1) / Square(c * (g - c) + 1));
	}

	double GeometryBidirectional(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, Vec3D v_microscopicNormal, double roughness)
	{
		return GeometryMonodirectional(v_incomingDirection, v_normal, v_microscopicNormal, roughness) * GeometryMonodirectional(v_outgoingDirection, v_normal, v_microscopicNormal, roughness);
	}

	double GeometryMonodirectional(Vec3D vec, Vec3D v_normal, Vec3D v_microscopicNormal, double roughness)
	{
		double VecDotNormal = DotProduct3D(vec, v_normal);
		double VecDotNormal2 = VecDotNormal * VecDotNormal;
		double a2 = VecDotNormal2 / (roughness * roughness * (1 - VecDotNormal2)); // a squared

		return Chi(DotProduct3D(vec, v_microscopicNormal) / DotProduct3D(vec, v_normal)) * 2 / (1 + sqrt(1 + 1 / a2));
	}

	Vec3D BTDF(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, Vec3D v_microscopicNormal, double refractionIndex1, double refractionIndex2, double roughness)
	{
		double fresnelFactor = Fresnel(v_incomingDirection, v_microscopicNormal, refractionIndex1, refractionIndex2);

		double btdf = Abs(DotProduct3D(v_incomingDirection, v_microscopicNormal)) * (1 - fresnelFactor) * GeometryBidirectional(v_incomingDirection, v_outgoingDirection, v_normal, v_microscopicNormal, roughness) / (Abs(DotProduct3D(v_incomingDirection, v_normal)) * Abs(DotProduct3D(v_microscopicNormal, v_normal)));

		return { btdf, btdf, btdf };
	}

	// computing the bisector vector (microscopic normal) used for importance sampling
	Vec3D MicroscopicNormal(Vec3D v_incomingDirection, Vec3D v_normal, double roughness, std::mt19937* randomEngine)
	{
		double randVariable = uniform_zero_to_one(*randomEngine);

		double cosTheta = sqrt((1 - randVariable) / (randVariable * (roughness * roughness - 1) + 1));
		double sinTheta = sqrt(1 - cosTheta * cosTheta);

		double randAngle = uniform_zero_to_one(*randomEngine) * TAU;

		Vec3D v_bisectorVector = { sinTheta * cos(randAngle), cosTheta, sinTheta * sin(randAngle) };

		Vec3D v_tangent = ReturnNormalizedVec3D(SubtractVec3D(v_incomingDirection, VecScalarMultiplication3D(v_normal, DotProduct3D(v_incomingDirection, v_normal))));

		Matrix3D transformationMatrix =
		{
			v_tangent,
			v_normal,
			CrossProduct(v_normal, v_tangent)
		};

		return VecMatrixMultiplication3D(v_bisectorVector, transformationMatrix);
	}

	/*Vec3D CalculateLighting_DistributionTracing(Vec3D v_objectColor, Material material, Vec3D v_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, int i_bounceCount)
	{
		Vec3D pixelColor = ZERO_VEC3D;

		// Temporary until refraction (it'll need to decide whether to offset in or out)
		AddToVec3D(&v_intersection, VecScalarMultiplication3D(v_surfaceNormal, OFFSET_DISTANCE));

		// Soft shadows
		for (int i = 0; i < g_lights.size(); i++)
		{
			double notBlockedProportion = 0;

			for (int j = 0; j < SAMPLES_PER_RAY; j++)
			{
				float randX = float(int64_t(randEngine()) - int64_t(randEngine.max()) / 2) / float(int64_t(randEngine.max()) / 2);
				float randY = float(int64_t(randEngine()) - int64_t(randEngine.max()) / 2) / float(int64_t(randEngine.max()) / 2);
				float randZ = float(int64_t(randEngine()) - int64_t(randEngine.max()) / 2) / float(int64_t(randEngine.max()) / 2);

				Vec3D v_displacement = { randX, randY, randZ };
				NormalizeVec3D(&v_displacement);
				v_displacement = VecScalarMultiplication3D(v_displacement, g_lights[i].radius);
				Vec3D randomPointLight = AddVec3D(g_lights[i].coords, v_displacement);

				Vec3D v_newDirection = ReturnNormalizedVec3D(SubtractVec3D(randomPointLight, v_intersection));

				notBlockedProportion += !IsRayBlocked(v_intersection, v_newDirection, g_lights[i].coords);
			}

			notBlockedProportion /= SAMPLES_PER_RAY;

			double distance = Distance3D(v_intersection, g_lights[i].coords) - g_lights[i].radius;

			//v_objectColor = VecScalarMultiplication3D(v_objectColor, material.emittance);
			Vec3D lightColor = VecScalarMultiplication3D(g_lights[i].tint, g_lights[i].emittance);

			// (objectColor + lightColor) * notBlockedProportion / (distance ^ 2)
			Vec3D v_shading = VecScalarMultiplication3D(VecScalarMultiplication3D(AddVec3D(v_objectColor, lightColor), notBlockedProportion), 1.0f / (distance * distance));

			AddToVec3D(&pixelColor, v_shading);
		}

		// Reflection


		// Refraction

		return pixelColor;
	}*/

	Vec3D RandomVec_InUnitSphere(std::mt19937* randomEngine)
	{
		Vec3D randPoint;

		do
		{
			double randX = uniformDistribution(*randomEngine);
			double randY = uniformDistribution(*randomEngine);
			double randZ = uniformDistribution(*randomEngine);

			randPoint = { randX, randY, randZ };
		} while (VecLengthSquared(randPoint) > 1);

		return randPoint;
	}
};

int main()
{
	Engine rayTracer;
	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}

#include "Controlls.h"