#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define SCREEN_WIDTH 800
#define SCREEN_HEIGHT 500
#define RENDER_DISTANCE 50
#define TOUCHING_DISTANCE 0.001f
#define OFFSET_DISTANCE 0.002f
#define MAX_BOUNCES 2
#define SAMPLES_PER_PIXEL 1
#define SAMPLES_PER_RAY 1

#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "olcPixelGameEngine.h"
#include "MathUtilities.cuh"
#include "WorldDatatypes.h"
#include "ParseOBJ.h"

// Global variables

Player g_player = { { 4, 6, -2 }, { 1, ZERO_VEC3D }, TAU * 0.25f };

Vec3D g_pixels[SCREEN_HEIGHT * SCREEN_WIDTH]; // Pixel buffer that contains all pixels that'll be drawn on screen
float g_depthBuffer[SCREEN_HEIGHT * SCREEN_WIDTH]; // Contains the distance to each point represented by a pixel

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;

olc::Sprite* g_textureAtlas;


namespace Options
{
	bool mcControls = false;
}


class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

public:
	bool OnUserCreate() override
	{
		g_textureAtlas = new olc::Sprite("../Assets/basketball.png");

		g_spheres = 
		{
			{ { -5, 6, 11 }, 10, { 100, 200, 255 }, 1, 0.75, g_textureAtlas, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ 1, 0, 1 }), PI / 4)},
			{ { 9, 6, 13 }, 3, { 255, 10, 100 }, 0.3, 0.8 }
		};

		g_triangles =
		{
			{ { { -2, 1, 3 }, { 0, 2, 3 }, { 1, 1.5, 3 } }, { { 0, 0 }, { 1, 1 }, { 0, 1 } } }
		};

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		Controlls(fElapsedTime);
		RayTracing();

		return true;
	}

	// Defined in Controlls.h
	void Controlls(float fElapsedTime);

	void RayTracing()
	{
		float zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (int y = -SCREEN_HEIGHT * 0.5f; y < SCREEN_HEIGHT * 0.5f; y++)
		{
			for (int x = -SCREEN_WIDTH * 0.5f; x < SCREEN_WIDTH * 0.5f; x++)
			{
				Vec3D v_direction = { x, y, zFar };
				NormalizeVec3D(&v_direction);

				Vec3D v_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, v_direction }, QuaternionConjugate(g_player.q_orientation)).vecPart;

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = (SCREEN_HEIGHT - 1) - (y + SCREEN_HEIGHT * 0.5f);

				Vec3D pixelColor = ZERO_VEC3D;

				for (int i = 0; i < SAMPLES_PER_PIXEL; i++)
				{
					// Clearing the buffers                        137, 250, 255
					g_pixels[SCREEN_WIDTH * screenY + screenX] = ZERO_VEC3D;
					g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = INFINITY;

					RenderGround(g_player.coords, v_newDirection, screenX, screenY);

					RenderSpheres(g_player.coords, v_newDirection, screenX, screenY);

					//RenderTriangles(g_player.coords, v_newDirection, screenX, screenY);

					AddToVec3D(&pixelColor, g_pixels[SCREEN_WIDTH * screenY + screenX]);
				}

				ScaleVec3D(&pixelColor, 1 / float(SAMPLES_PER_PIXEL));

				pixelColor.x = Min(pixelColor.x, 255.0f);
				pixelColor.y = Min(pixelColor.y, 255.0f);
				pixelColor.z = Min(pixelColor.z, 255.0f);

				Draw(screenX, screenY, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
		}
	}

	void RenderGround(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersectionColor = ZERO_VEC3D;

		bool intersectionExists;
		Vec3D v_intersection = ZERO_VEC3D;
		float depth = 0;

		float groundLevel = -1;
		VertexPair2D textureVertexPair = { { { 0, 0 }, { 1, 1 } } };
		float textureScalar = 10;

		intersectionExists = GroundIntersectionRT(groundLevel, textureVertexPair, textureScalar, v_start, v_direction, &v_intersection, &v_intersectionColor, &depth);

		if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
		{
			v_intersectionColor = CalculateLighting_PathTracing(v_intersectionColor, 0, 0.3, { 0, 1, 0 }, v_intersection, 0);

			g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
			g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
		}
	}

	bool GroundIntersectionRT(float groundLevel, VertexPair2D textureVertexPair, float textureScalar, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, float* depth = nullptr)
	{
		if (v_direction.y >= 0 || v_start.y < groundLevel)
		{
			return false;
		}

		if (v_intersection == nullptr)
		{
			return true;
		}

		ScaleVec3D(&v_direction, (groundLevel - v_start.y) / v_direction.y);

		Vec3D rayGroundIntersection = AddVec3D(v_start, v_direction);

		Vec3D v_offset = VecScalarMultiplication3D({ 0, 1, 0 }, OFFSET_DISTANCE);

		AddToVec3D(&rayGroundIntersection, v_offset);

		*v_intersection = rayGroundIntersection;

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, rayGroundIntersection);
		}

		if (v_intersectionColor == nullptr)
		{
			return true;
		}

		float signedTextureWidth = (textureVertexPair.vertices[1].x - textureVertexPair.vertices[0].x) * textureScalar;
		float signedTextureHeight = (textureVertexPair.vertices[1].y - textureVertexPair.vertices[0].y) * textureScalar;

		float textureX = fmod(rayGroundIntersection.x, signedTextureWidth) / signedTextureWidth;
		float textureY = fmod(rayGroundIntersection.z, signedTextureHeight) / signedTextureHeight;

		// if the textureCoordinates are negative, we need to flip them around the center of the texture and make them positive
		if (textureX < 0) textureX += 1;
		if (textureY < 0) textureY += 1;

		olc::Pixel texelColor = g_textureAtlas->Sample(textureX + textureVertexPair.vertices[0].x, textureY + textureVertexPair.vertices[0].y);

		*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };

		return true;
	}

	/*bool GroundIntersectionRM(float groundLevel, VertexPair2D textureVertexPair, float textureScalar, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr, Vec3D* pixelColor = nullptr)
	{
		float totalDistanceTravelled = 0;

		while (totalDistanceTravelled < RENDER_DISTANCE)
		{
			float distanceToGround = abs(v_start.y - groundLevel);

			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distanceToGround));

			if (distanceToGround < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				float signedTextureWidth = (textureVertexPair.vertices[1].x - textureVertexPair.vertices[0].x) * textureScalar;
				float signedTextureHeight = (textureVertexPair.vertices[1].y - textureVertexPair.vertices[0].y) * textureScalar;

				float textureX = fmod(v_start.x, signedTextureWidth) / signedTextureWidth;
				float textureY = fmod(v_start.z, signedTextureHeight) / signedTextureHeight;

				// if the textureCoordinates are negative, we need to flip them around the center of the texture and make them positive
				if (textureX < 0) textureX += 1;
				if (textureY < 0) textureY += 1;

				olc::Pixel texelColor = g_textureAtlas->Sample(textureX, textureY);

				*pixelColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };

				return true;
			}

			totalDistanceTravelled += distanceToGround;
		}

		return false;
	}*/

	void RenderSpheres(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_intersectionColor = ZERO_VEC3D;
		float depth = 0;
		Vec3D v_surfaceNormal = ZERO_VEC3D;

		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, &v_intersection, &v_intersectionColor, &depth, &v_surfaceNormal);

			//bool intersectionExists = SphereIntersection_RM(g_spheres[i], v_start, v_direction, &v_intersection, &depth);

			if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
			{
				v_intersectionColor = CalculateLighting_PathTracing(v_intersectionColor, g_spheres[i].emittance, g_spheres[i].reflectance, v_surfaceNormal, v_intersection, 0);

				g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
			}
		}
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, float* depth = nullptr, Vec3D* v_surfaceNormal = nullptr)
	{
		float dxdz = v_direction.x / v_direction.z;
		float dydz = v_direction.y / v_direction.z;

		float a = dxdz * dxdz + dydz * dydz + 1;
		
		float b = 
			2 * dxdz * (v_start.x - sphere.coords.x) +
			2 * dydz * (v_start.y - sphere.coords.y) +
			2 * (v_start.z - sphere.coords.z);

		float c = 
			(v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) +
			(v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) +
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// ISAK: There wasn't any need to recalculate this multiple times
		float rootContent = b * b - 4 * a * c;

		// There exists no intersections (no real answer)
		if (rootContent < 0) return false;

		float z1 = (-b + sqrt(rootContent)) / (2 * a);
		float z2 = (-b - sqrt(rootContent)) / (2 * a);

		Vec3D v_alternative1 = { z1 * dxdz, z1 * dydz, z1 };
		AddToVec3D(&v_alternative1, v_start);

		Vec3D v_alternative2 = { z2 * dxdz, z2 * dydz, z2 };
		AddToVec3D(&v_alternative2, v_start);

		// Check which intersection is the closest and choose that one
		float dist1 = DistanceSquared3D(v_alternative1, v_start);
		float dist2 = DistanceSquared3D(v_alternative2, v_start);

		Vec3D v_correctHit = (dist1 < dist2) ? v_alternative1 : v_alternative2;

		// Check if the intersection is behind the player. if so, discard it
		if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0) return false;

		Vec3D v_normal = SubtractVec3D(v_correctHit, sphere.coords);
		NormalizeVec3D(&v_normal);

		// There exists an intersection which is not behind the ray, but we don't care about returning where the intersection was
		if (v_intersection != nullptr)
		{
			Vec3D v_offset = VecScalarMultiplication3D(v_normal, OFFSET_DISTANCE);

			// ISAK: Better to offset the intersection here so we don't have to do it anywere else
			*v_intersection = AddVec3D(v_correctHit, v_offset);
		}

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, v_correctHit);
		}

		if (v_surfaceNormal != nullptr)
		{
			*v_surfaceNormal = v_normal;
		}

		if (v_intersectionColor != nullptr)
		{
			if (sphere.texture == nullptr) *v_intersectionColor = sphere.color;
			else *v_intersectionColor = SphereTexturing(sphere, v_normal);
		}

		return true;
	}

	Vec3D SphereTexturing(Sphere sphere, Vec3D v_normal)
	{
		Vec3D iHat = { 1, 0, 0 };
		Vec3D jHat = { 0, 1, 0 };
		Vec3D kHat = { 0, 0, 1 };

		// Rotating axies by sphere rotation quaternion
		iHat = QuaternionMultiplication(sphere.rotQuaternion, { 0, iHat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
		jHat = QuaternionMultiplication(sphere.rotQuaternion, { 0, jHat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
		kHat = QuaternionMultiplication(sphere.rotQuaternion, { 0, kHat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;

		// Translate normal into new coordinate system
		v_normal = { DotProduct3D(v_normal, iHat), DotProduct3D(v_normal, jHat), DotProduct3D(v_normal, kHat) };
		
		// UV coordinates
		float u = 0.5 + atan2(v_normal.x, v_normal.z) / TAU;
		float v = 0.5 - asin(v_normal.y) / PI;
		
		// Interpolate between assigned texture coordinates
		float textureX = Lerp(sphere.textureCorner1.x, sphere.textureCorner2.x, u);
		float textureY = Lerp(sphere.textureCorner1.y, sphere.textureCorner2.y, v);

		olc::Pixel texelColor = sphere.texture->Sample(textureX, textureY);

		return { (float)texelColor.r, (float)texelColor.g, (float)texelColor.b };
	}

	// Ray marching for spheres
	/*bool SphereIntersection_RM(Sphere sphere, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr)
	{
		float distanceTravelled = 0;

		while (distanceTravelled < RENDER_DISTANCE)
		{
			float distance = Distance3D(v_start, sphere.coords) - sphere.radius;

			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distance));
			distanceTravelled += distance;

			if (distance < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr) 
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				return true;
			}
		}

		return false;
	}*/

	void RenderTriangles(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersectionColor = ZERO_VEC3D;

		bool intersectionExists;
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_surfaceNormal = ZERO_VEC3D;
		float depth = 0;

		for (int i = 0; i < g_triangles.size(); i++)
		{
			intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, &v_intersection, &v_intersectionColor, &depth, &v_surfaceNormal);

			if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
			{
				g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
			}
		}
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, float* depth = nullptr, Vec3D* v_surfaceNormal = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if ((DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) ||
			(DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) ||
			(DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0))
		{
			return false;
		}

		Vec3D v_offset = VecScalarMultiplication3D(v_triangleNormal, OFFSET_DISTANCE);

		AddToVec3D(&v_trianglePlaneIntersection, v_offset);

		// if we don't care where the intersection is we just return true before setting v_intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_trianglePlaneIntersection;
		}

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, v_trianglePlaneIntersection);
		}

		if (v_surfaceNormal != nullptr)
		{
			*v_surfaceNormal = v_triangleNormal;
		}
		
		if (v_intersectionColor == nullptr)
		{
			return true;
		}

		// from here on we calculate the texture coordinates

		Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
		Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

		Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

		Matrix3D triangleMatrix =
		{
			v_triangleEdge1,
			v_triangleEdge2,
			v_triangleNormal
		};

		Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

		Vec2D textureCoordinates = { 0, 0 };

		AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
		AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
		AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

		olc::Pixel texelColor = g_textureAtlas->Sample(textureCoordinates.x, textureCoordinates.y);

		*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };
		
		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, float f_planeOffset)
	{
		float f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		float f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	/*bool TriangleIntersection_RM(Triangle triangle, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr, float* depth = nullptr, Vec3D* pixelColor = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);
		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		float f_totalDistanceTravelled = 0;

		while (f_totalDistanceTravelled < RENDER_DISTANCE)
		{
			float f_signedDistanceToPlane = f_trianglePlaneOffset - DotProduct3D(v_start, v_triangleNormal);

			// the start vector projected onto the trianglePlane
			Vec3D vecProjectedOnPlane = AddVec3D(v_start, VecScalarMultiplication3D(v_triangleNormal, f_signedDistanceToPlane));

			float f_distanceToTriangle;

			// If the projectedPoint is inside the triangle then the distance to the triangle is just the distance to the plane
			if (DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[0])) <= 0 &&
				DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[1])) <= 0 &&
				DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[2])) <= 0)
			{
				f_distanceToTriangle = abs(f_signedDistanceToPlane);
			}
			// Otherwise, the distance to the triangle is the distance to the closest edge of the triangle
			else
			{
				float distanceToEdge1 = DistanceToEdge(v_start, triangle.vertices[1], triangle.vertices[0]);
				float distanceToEdge2 = DistanceToEdge(v_start, triangle.vertices[2], triangle.vertices[1]);
				float distanceToEdge3 = DistanceToEdge(v_start, triangle.vertices[0], triangle.vertices[2]);

				float minDistance = distanceToEdge1;

				if (distanceToEdge2 < minDistance) minDistance = distanceToEdge2;
				if (distanceToEdge3 < minDistance) minDistance = distanceToEdge3;

				f_distanceToTriangle = minDistance;
			}
			
			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, f_distanceToTriangle));

			if (f_distanceToTriangle < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				// Calculating the texture coordinates

				Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
				Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

				Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_start, triangle.vertices[0]);

				Matrix3D triangleMatrix =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					v_triangleNormal
				};

				Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

				Vec2D textureCoordinates = { 0, 0 };

				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
				AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

				olc::Pixel texelColor = g_textureAtlas->Sample(textureCoordinates.x, textureCoordinates.y);

				*pixelColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };

				return true;
			}
			
			f_totalDistanceTravelled += f_distanceToTriangle;
		}

		return false;
	}

	float DistanceToEdge(Vec3D v_point, Vec3D v_vertex1, Vec3D v_vertex2)
	{
		Vec3D v_edgeDirection = SubtractVec3D(v_vertex2, v_vertex1);
		NormalizeVec3D(&v_edgeDirection);
		float f_edgeLength = Distance3D(v_vertex1, v_vertex2);

		float f_projectedPointOnEdgelength = DotProduct3D(SubtractVec3D(v_point, v_vertex1), v_edgeDirection);

		Vec3D v_closestPoint = VecScalarMultiplication3D(v_edgeDirection, Clamp(f_projectedPointOnEdgelength, 0, f_edgeLength));

		return Distance3D(v_point, v_closestPoint);
	}*/

	Vec3D CalculateLighting_PathTracing(Vec3D v_objectColor, float f_objectEmittance, float f_objectReflectance, Vec3D v_surfaceNormal, Vec3D v_start, int i_bounceCount)
	{
		Vec3D v_outgoingLightColor = VecScalarMultiplication3D(v_objectColor, f_objectEmittance);

		if (i_bounceCount > MAX_BOUNCES)
		{
			return v_outgoingLightColor;
		}

		Vec3D v_direction = ReturnNormalizedVec3D({ float(rand()), float(rand()), float(rand()) });

		// The direction vector is in the wrong hemisphere, so we need to flip it into the other hemisphere
		if (DotProduct3D(v_surfaceNormal, v_direction) < 0)
		{
			ScaleVec3D(&v_direction, -1);
		}

		for (int i = 0; i < g_spheres.size(); i++)
		{
			Vec3D v_intersection = ZERO_VEC3D;
			Vec3D v_intersectionColor = ZERO_VEC3D;
			Vec3D v_normal = ZERO_VEC3D;

			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, &v_intersection, &v_intersectionColor, nullptr, &v_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				for (int j = 0; j < g_spheres.size(); j++)
				{
					Vec3D v_otherIntersection = ZERO_VEC3D;

					bool otherIntersectionExists = SphereIntersection_RT(g_spheres[j], v_start, v_direction, &v_otherIntersection);

					// If there exists a closer intersection to the ray start vector it means the ray is blocked
					if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
					{
						b_rayIsBlocked = true;
						break;
					}
				}

				for (int j = 0; j < g_triangles.size(); j++)
				{
					Vec3D v_otherIntersection = ZERO_VEC3D;

					bool otherIntersectionExists = TriangleIntersection_RT(g_triangles[j], v_start, v_direction, &v_otherIntersection);

					// If there exists a closer intersection to the ray start vector it means the ray is blocked
					if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
					{
						b_rayIsBlocked = true;
						break;
					}
				}
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				Vec3D v_incomingLightColor = CalculateLighting_PathTracing(v_intersectionColor, g_spheres[i].emittance, g_spheres[i].reflectance, v_normal, v_intersection, i_bounceCount + 1);

				AddToVec3D(
					&v_outgoingLightColor,
					VecScalarMultiplication3D(v_incomingLightColor, 2 * f_objectReflectance * DotProduct3D(v_surfaceNormal, v_direction))
				);

				return v_outgoingLightColor;
			}
		}

		for (int i = 0; i < g_triangles.size(); i++)
		{
			Vec3D v_intersection = ZERO_VEC3D;
			Vec3D v_intersectionColor = ZERO_VEC3D;
			Vec3D v_normal = ZERO_VEC3D;

			bool intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, &v_intersection, &v_intersectionColor, nullptr, &v_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				for (int j = 0; j < g_spheres.size(); j++)
				{
					Vec3D v_otherIntersection = ZERO_VEC3D;

					bool otherIntersectionExists = SphereIntersection_RT(g_spheres[j], v_start, v_direction, &v_otherIntersection);

					// If there exists a closer intersection to the ray start vector it means the ray is blocked
					if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
					{
						b_rayIsBlocked = true;
						break;
					}
				}

				for (int j = 0; j < g_triangles.size(); j++)
				{
					Vec3D v_otherIntersection = ZERO_VEC3D;

					bool otherIntersectionExists = TriangleIntersection_RT(g_triangles[j], v_start, v_direction, &v_otherIntersection);

					// If there exists a closer intersection to the ray start vector it means the ray is blocked
					if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
					{
						b_rayIsBlocked = true;
						break;
					}
				}
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				Vec3D v_incomingLightColor = CalculateLighting_PathTracing(v_intersectionColor, g_triangles[i].emittance, g_triangles[i].reflectance, v_normal, v_intersection, i_bounceCount + 1);

				AddToVec3D(
					&v_outgoingLightColor,
					VecScalarMultiplication3D(v_incomingLightColor, 2 * f_objectReflectance * DotProduct3D(v_surfaceNormal, v_direction))
				);

				return v_outgoingLightColor;
			}
		}

		return v_outgoingLightColor;
	}
};

int main()
{
	Engine rayTracer;
	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}

#include "Controlls.h"