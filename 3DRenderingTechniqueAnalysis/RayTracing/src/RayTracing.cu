#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define SCREEN_WIDTH 800
#define SCREEN_HEIGHT 500
#define RENDER_DISTANCE 50
#define TOUCHING_DISTANCE 0.01f
#define OFFSET_DISTANCE 0.002f
#define MAX_BOUNCES 3
#define SAMPLES_PER_PIXEL 100
#define SAMPLES_PER_RAY 1
#define WHITE_COLOR { 255, 255, 255 }

#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <>
#include "olcPixelGameEngine.h"
#include "MathUtilities.cuh"
#include "WorldDatatypes.h"

// Global variables

Player g_player;

Vec3D g_pixels[SCREEN_HEIGHT * SCREEN_WIDTH]; // Pixel buffer that contains all pixels that'll be drawn on screen
float g_depthBuffer[SCREEN_HEIGHT * SCREEN_WIDTH]; // Contains the distance to each point represented by a pixel

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;

Ground g_ground;

olc::Sprite* g_textureAtlas;

enum ControlsType
{
	TYPE_1,
	TYPE_2
};

ControlsType g_controlsType;

std::default_random_engine randEngine;

class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

public:

	bool OnUserCreate() override
	{
		g_player = { { 1.5, 1.5, -1.5 }, { 1, ZERO_VEC3D }, TAU * 0.25f };

		g_textureAtlas = new olc::Sprite("../Assets/textureAtlas.png");

		g_spheres = 
		{ 
			{ { 1.5, 3, 1.5 }, 0.5, { 0.965, 0.795, 0.3333 }, 10, 0 }
		};

		g_triangles =
		{
			// Walls first face
			{ { { 0, 0, 3 }, { 0, 3, 3 }, { 3, 3, 3 } }, { 0.8, 1.2, 0.8 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },
			{ { { 0, 0, 3 }, { 3, 3, 3 }, { 3, 0, 3 } }, { 0.8, 1.2, 0.8 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },
			// Walls second face
			{ { { 0, 0, 0 }, { 0, 3, 0 }, { 0, 3, 3 } }, { 0.8, 1.1, 1.1 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },
			{ { { 0, 0, 0 }, { 0, 3, 3 }, { 0, 0, 3 } }, { 0.8, 1.1, 1.1 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },
			// Walls third face
			{ { { 3, 0, 3 }, { 3, 3, 3 }, { 3, 3, 0 } }, { 1.1, 0.8, 1.1 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },
			{ { { 3, 0, 3 }, { 3, 3, 0 }, { 3, 0, 0 } }, { 1.1, 0.8, 1.1 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },
			// Walls fourth face
			{ { { 0, 3, 0 }, { 3, 3, 3 }, { 0, 3, 3 } }, { 1, 1, 1 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },
			{ { { 0, 3, 0 }, { 3, 3, 0 }, { 3, 3, 3 } }, { 1, 1, 1 }, 0.1, 0.3, g_textureAtlas, { { 0.5, 0.5 }, { 0.5, 0 }, { 1, 0 } } },

			// Box first face
			{ { { 1, 0, 2 }, { 2, 1, 2 }, { 1, 1, 2 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0, 0 }, { 0.5, 0 } } },
			{ { { 1, 0, 2 }, { 2, 0, 2 }, { 2, 1, 2 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0.5, 0 }, { 0.5, 0.5 } } },
			// Box second face
			{ { { 1, 0, 1 }, { 1, 1, 1 }, { 2, 1, 1 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0, 0 }, { 0.5, 0 } } },
			{ { { 1, 0, 1 }, { 2, 1, 1 }, { 2, 0, 1 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0.5, 0 }, { 0.5, 0.5 } } },
			// Box third face
			{ { { 1, 0, 1 }, { 1, 1, 2 }, { 1, 1, 1 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0, 0 }, { 0.5, 0 } } },
			{ { { 1, 0, 1 }, { 1, 0, 2 }, { 1, 1, 2 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0.5, 0 }, { 0.5, 0.5 } } },
			// Box fourth face							   
			{ { { 2, 0, 1 }, { 2, 1, 1 }, { 2, 1, 2 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0, 0 }, { 0.5, 0 } } },
			{ { { 2, 0, 1 }, { 2, 1, 2 }, { 2, 0, 2 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0.5, 0 }, { 0.5, 0.5 } } },
			// Box fifth face							   
			{ { { 1, 1, 1 }, { 1, 1, 2 }, { 2, 1, 2 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0, 0 }, { 0.5, 0 } } },
			{ { { 1, 1, 1 }, { 2, 1, 2 }, { 2, 1, 1 } }, { 1, 1, 1 }, 0.1, 0.4, g_textureAtlas, { { 0, 0.5 }, { 0.5, 0 }, { 0.5, 0.5 } } }
		};

		g_ground = { 0, { 1, 1, 1 }, 0.1, 0.5, g_textureAtlas, { 0, 0.5 }, { 0.5, 1 }, 1 };

		g_controlsType = TYPE_2;

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		Controlls(fElapsedTime);
		RayTracing();

		return true;
	}

	void Controlls(float fElapsedTime)
	{
		float movementSpeed = 7 * fElapsedTime;
		float rotationSpeed = 2.5 * fElapsedTime;

		// Movement

		if (GetKey(olc::Key::W).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { 0, 0, 1 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}

		if (GetKey(olc::Key::A).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { -1, 0, 0 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}

		if (GetKey(olc::Key::S).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { 0, 0, -1 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}

		if (GetKey(olc::Key::D).bHeld)
		{
			Quaternion q_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, { 1, 0, 0 } }, ConjugateQuaternion(g_player.q_orientation));

			if (g_controlsType == TYPE_1)
			{
				q_newDirection.vecPart.y = 0;
			}

			NormalizeVec3D(&q_newDirection.vecPart);
			ScaleVec3D(&q_newDirection.vecPart, movementSpeed);

			AddToVec3D(&g_player.coords, q_newDirection.vecPart);
		}

		if (GetKey(olc::Key::SPACE).bHeld)
		{
			g_player.coords.y += movementSpeed;
		}

		if (GetKey(olc::Key::SHIFT).bHeld)
		{
			g_player.coords.y -= movementSpeed;
		}

		// Rotation

		if (GetKey(olc::Key::RIGHT).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion q_newRotationAxis = QuaternionMultiplication(ConjugateQuaternion(g_player.q_orientation), { 0, { 0, 1, 0 } }, g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion(q_newRotationAxis.vecPart, rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}

		if (GetKey(olc::Key::LEFT).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion q_newRotationAxis = QuaternionMultiplication(ConjugateQuaternion(g_player.q_orientation), { 0, { 0, 1, 0 } }, g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion(q_newRotationAxis.vecPart, -rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}

		if (GetKey(olc::Key::UP).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion({ 1, 0, 0 }, -rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}

		if (GetKey(olc::Key::DOWN).bHeld)
		{
			NormalizeQuaternion(&g_player.q_orientation);

			Quaternion rotationQuaternion = CreateRotationQuaternion({ 1, 0, 0 }, rotationSpeed);

			g_player.q_orientation = QuaternionMultiplication(g_player.q_orientation, rotationQuaternion);
		}
	}

	void RayTracing()
	{
		float zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (int y = -SCREEN_HEIGHT * 0.5f; y < SCREEN_HEIGHT * 0.5f; y++)
		{
			for (int x = -SCREEN_WIDTH * 0.5f; x < SCREEN_WIDTH * 0.5f; x++)
			{
				Vec3D v_direction = { x, y, zFar };
				NormalizeVec3D(&v_direction);

				Vec3D v_newDirection = QuaternionMultiplication(g_player.q_orientation, { 0, v_direction }, ConjugateQuaternion(g_player.q_orientation)).vecPart;

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = (SCREEN_HEIGHT - 1) - (y + SCREEN_HEIGHT * 0.5f);

				Vec3D pixelColor = ZERO_VEC3D;

				for (int i = 0; i < SAMPLES_PER_PIXEL; i++)
				{
					// Clearing the buffers
					g_pixels[SCREEN_WIDTH * screenY + screenX] = ZERO_VEC3D;
					g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = INFINITY;

					RenderGround(g_player.coords, v_newDirection, screenX, screenY);

					RenderSpheres(g_player.coords, v_newDirection, screenX, screenY);

					RenderTriangles(g_player.coords, v_newDirection, screenX, screenY);

					AddToVec3D(&pixelColor, g_pixels[SCREEN_WIDTH * screenY + screenX]);
				}

				ScaleVec3D(&pixelColor, 1 / float(SAMPLES_PER_PIXEL));

				pixelColor.x = Min(pixelColor.x, 255.0f);
				pixelColor.y = Min(pixelColor.y, 255.0f);
				pixelColor.z = Min(pixelColor.z, 255.0f);

				Draw(screenX, screenY, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
		}
	}

	void RenderGround(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersectionColor = ZERO_VEC3D;

		bool intersectionExists;
		Vec3D v_intersection = ZERO_VEC3D;
		float depth = 0;

		intersectionExists = GroundIntersection_RT(v_start, v_direction, &v_intersection, &v_intersectionColor, &depth);

		if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
		{
			v_intersectionColor = CalculateLighting_PathTracing(v_intersectionColor, g_ground.emittance, g_ground.reflectance, { 0, 1, 0 }, v_intersection, 0);

			g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
			g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
		}
	}

	bool GroundIntersection_RT(Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, float* depth = nullptr)
	{
		if (v_direction.y >= 0 || v_start.y < g_ground.level)
		{
			return false;
		}

		if (v_intersection == nullptr)
		{
			return true;
		}

		ScaleVec3D(&v_direction, (g_ground.level - v_start.y) / v_direction.y);

		Vec3D rayGroundIntersection = AddVec3D(v_start, v_direction);

		Vec3D v_offset = VecScalarMultiplication3D({ 0, 1, 0 }, OFFSET_DISTANCE);

		AddToVec3D(&rayGroundIntersection, v_offset);

		*v_intersection = rayGroundIntersection;

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, rayGroundIntersection);
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (g_ground.texture != nullptr)
		{
			float signedTextureWidth = (g_ground.textureCorner2.x - g_ground.textureCorner1.x) * g_ground.textureScalar;
			float signedTextureHeight = (g_ground.textureCorner2.y - g_ground.textureCorner1.y) * g_ground.textureScalar;

			float t1 = fmod(rayGroundIntersection.x, signedTextureWidth) / signedTextureWidth;
			float t2 = fmod(rayGroundIntersection.z, signedTextureHeight) / signedTextureHeight;

			// if the textureCoordinates are negative, we need to flip them around the center of the texture and make them positive
			if (t1 < 0) t1 += 1;
			if (t2 < 0) t2 += 1;

			olc::Pixel texelColor = g_textureAtlas->Sample(
				Lerp(g_ground.textureCorner1.x, g_ground.textureCorner2.x, t1),
				Lerp(g_ground.textureCorner1.y, g_ground.textureCorner2.y, t2)
			);

			*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };
		}

		// Proof that the ConusProduct is the most useful function

		// Tint the color
		*v_intersectionColor = ConusProduct(*v_intersectionColor, g_ground.tint);

		return true;
	}

	/*bool GroundIntersection_RM(float groundLevel, VertexPair2D textureVertexPair, float textureScalar, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr, Vec3D* pixelColor = nullptr)
	{
		float totalDistanceTravelled = 0;

		while (totalDistanceTravelled < RENDER_DISTANCE)
		{
			float distanceToGround = abs(v_start.y - groundLevel);

			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distanceToGround));

			if (distanceToGround < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				float signedTextureWidth = (textureVertexPair.vertices[1].x - textureVertexPair.vertices[0].x) * textureScalar;
				float signedTextureHeight = (textureVertexPair.vertices[1].y - textureVertexPair.vertices[0].y) * textureScalar;

				float textureX = fmod(v_start.x, signedTextureWidth) / signedTextureWidth;
				float textureY = fmod(v_start.z, signedTextureHeight) / signedTextureHeight;

				// if the textureCoordinates are negative, we need to flip them around the center of the texture and make them positive
				if (textureX < 0) textureX += 1;
				if (textureY < 0) textureY += 1;

				olc::Pixel texelColor = g_textureAtlas->Sample(textureX, textureY);

				*pixelColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };

				return true;
			}

			totalDistanceTravelled += distanceToGround;
		}

		return false;
	}*/

	void RenderSpheres(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_intersectionColor = ZERO_VEC3D;
		float depth = 0;
		Vec3D v_surfaceNormal = ZERO_VEC3D;

		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, &v_intersection, &v_intersectionColor, &depth, &v_surfaceNormal);

			//bool intersectionExists = SphereIntersection_RM(g_spheres[i], v_start, v_direction, &v_intersection, &depth);

			if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
			{
				v_intersectionColor = CalculateLighting_PathTracing(v_intersectionColor, g_spheres[i].emittance, g_spheres[i].reflectance, v_surfaceNormal, v_intersection, 0);

				g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
			}
		}
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, float* depth = nullptr, Vec3D* v_surfaceNormal = nullptr)
	{
		float dxdz = v_direction.x / v_direction.z;
		float dydz = v_direction.y / v_direction.z;

		float a = dxdz * dxdz + dydz * dydz + 1;
		
		float b = 
			2 * dxdz * (v_start.x - sphere.coords.x) +
			2 * dydz * (v_start.y - sphere.coords.y) +
			2 * (v_start.z - sphere.coords.z);

		float c = 
			(v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) +
			(v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) +
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// ISAK: There wasn't any need to recalculate this multiple times
		float rootContent = b * b - 4 * a * c;

		// There exists no intersections (no real answer)
		if (rootContent < 0) return false;

		float z1 = (-b + sqrt(rootContent)) / (2 * a);
		float z2 = (-b - sqrt(rootContent)) / (2 * a);

		Vec3D v_alternative1 = { z1 * dxdz, z1 * dydz, z1 };
		AddToVec3D(&v_alternative1, v_start);

		Vec3D v_alternative2 = { z2 * dxdz, z2 * dydz, z2 };
		AddToVec3D(&v_alternative2, v_start);

		// Check which intersection is the closest and choose that one
		float dist1 = DistanceSquared3D(v_alternative1, v_start);
		float dist2 = DistanceSquared3D(v_alternative2, v_start);

		Vec3D v_correctHit = (dist1 < dist2) ? v_alternative1 : v_alternative2;

		// Check if the intersection is behind the ray. if so, discard it
		if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0) return false;

		Vec3D v_normal = SubtractVec3D(v_correctHit, sphere.coords);
		NormalizeVec3D(&v_normal);

		// There exists an intersection which is not behind the ray, but we don't care about returning where the intersection was
		if (v_intersection != nullptr)
		{
			Vec3D v_offset = VecScalarMultiplication3D(v_normal, OFFSET_DISTANCE);

			// ISAK: Better to offset the intersection here so we don't have to do it anywere else
			*v_intersection = AddVec3D(v_correctHit, v_offset);
		}

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, v_correctHit);
		}

		if (v_surfaceNormal != nullptr)
		{
			*v_surfaceNormal = v_normal;
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;
		
		// Tint the color
		*v_intersectionColor = ConusProduct(*v_intersectionColor, sphere.tint);

		return true;
	}

	// Ray marching for spheres
	/*bool SphereIntersection_RM(Sphere sphere, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, float* depth = nullptr)
	{
		float distanceTravelled = 0;

		while (distanceTravelled < RENDER_DISTANCE)
		{
			float distance = Distance3D(v_start, sphere.coords) - sphere.radius;

			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, distance));
			distanceTravelled += distance;

			if (distance < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr) 
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				return true;
			}
		}

		return false;
	}*/

	void RenderTriangles(Vec3D v_start, Vec3D v_direction, int screenX, int screenY)
	{
		Vec3D v_intersectionColor = ZERO_VEC3D;

		bool intersectionExists;
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_surfaceNormal = ZERO_VEC3D;
		float depth = 0;

		for (int i = 0; i < g_triangles.size(); i++)
		{
			intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, &v_intersection, &v_intersectionColor, &depth, &v_surfaceNormal);

			if (intersectionExists && depth < g_depthBuffer[SCREEN_WIDTH * screenY + screenX])
			{
				v_intersectionColor = CalculateLighting_PathTracing(v_intersectionColor, g_triangles[i].emittance, g_triangles[i].reflectance, v_surfaceNormal, v_intersection, 0);

				g_pixels[SCREEN_WIDTH * screenY + screenX] = v_intersectionColor;
				g_depthBuffer[SCREEN_WIDTH * screenY + screenX] = depth;
			}
		}
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction, 
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, float* depth = nullptr, Vec3D* v_surfaceNormal = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if ((DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) ||
			(DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) ||
			(DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0))
		{
			return false;
		}

		Vec3D v_offset = VecScalarMultiplication3D(v_triangleNormal, OFFSET_DISTANCE);

		AddToVec3D(&v_trianglePlaneIntersection, v_offset);

		if (DotProduct3D(SubtractVec3D(v_trianglePlaneIntersection, v_start), v_direction) < 0)
		{
			return false;
		}

		// if we don't care where the intersection is we just return true before setting v_intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_trianglePlaneIntersection;
		}

		if (depth != nullptr)
		{
			*depth = Distance3D(g_player.coords, v_trianglePlaneIntersection);
		}

		if (v_surfaceNormal != nullptr)
		{
			*v_surfaceNormal = v_triangleNormal;
		}
		
		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (triangle.texture != nullptr)
		{
			// from here on we calculate the texture coordinates

			Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
			Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

			Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

			Matrix3D triangleMatrix =
			{
				v_triangleEdge1,
				v_triangleEdge2,
				v_triangleNormal
			};

			Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

			Vec2D textureCoordinates = { 0, 0 };

			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
			AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

			olc::Pixel texelColor = g_textureAtlas->Sample(textureCoordinates.x, textureCoordinates.y);

			*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };
		}
		
		// Tint the color
		*v_intersectionColor = ConusProduct(*v_intersectionColor, triangle.tint);
		
		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, float f_planeOffset)
	{
		float f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		float f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	/*bool TriangleIntersection_RM(Triangle triangle, Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection = nullptr, float* depth = nullptr, Vec3D* pixelColor = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);
		NormalizeVec3D(&v_triangleNormal);

		// the triangle is facing away from the ray, so we return no intersection
		if (DotProduct3D(v_triangleNormal, v_direction) > 0) return false;

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		float f_totalDistanceTravelled = 0;

		while (f_totalDistanceTravelled < RENDER_DISTANCE)
		{
			float f_signedDistanceToPlane = f_trianglePlaneOffset - DotProduct3D(v_start, v_triangleNormal);

			// the start vector projected onto the trianglePlane
			Vec3D vecProjectedOnPlane = AddVec3D(v_start, VecScalarMultiplication3D(v_triangleNormal, f_signedDistanceToPlane));

			float f_distanceToTriangle;

			// If the projectedPoint is inside the triangle then the distance to the triangle is just the distance to the plane
			if (DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[0])) <= 0 &&
				DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[1])) <= 0 &&
				DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(vecProjectedOnPlane, triangle.vertices[2])) <= 0)
			{
				f_distanceToTriangle = abs(f_signedDistanceToPlane);
			}
			// Otherwise, the distance to the triangle is the distance to the closest edge of the triangle
			else
			{
				float distanceToEdge1 = DistanceToEdge(v_start, triangle.vertices[1], triangle.vertices[0]);
				float distanceToEdge2 = DistanceToEdge(v_start, triangle.vertices[2], triangle.vertices[1]);
				float distanceToEdge3 = DistanceToEdge(v_start, triangle.vertices[0], triangle.vertices[2]);

				float minDistance = distanceToEdge1;

				if (distanceToEdge2 < minDistance) minDistance = distanceToEdge2;
				if (distanceToEdge3 < minDistance) minDistance = distanceToEdge3;

				f_distanceToTriangle = minDistance;
			}
			
			AddToVec3D(&v_start, VecScalarMultiplication3D(v_direction, f_distanceToTriangle));

			if (f_distanceToTriangle < TOUCHING_DISTANCE)
			{
				if (v_intersection == nullptr)
				{
					return true;
				}

				*v_intersection = v_start;
				*depth = Distance3D(g_player.coords, v_start);

				if (pixelColor == nullptr)
				{
					return true;
				}

				// Calculating the texture coordinates

				Vec2D v_textureTriangleEdge1 = SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]);
				Vec2D v_textureTriangleEdge2 = SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]);

				Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_start, triangle.vertices[0]);

				Matrix3D triangleMatrix =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					v_triangleNormal
				};

				Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

				Vec2D textureCoordinates = { 0, 0 };

				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge1, triangleEdgeScalars.x));
				AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(v_textureTriangleEdge2, triangleEdgeScalars.y));
				AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

				olc::Pixel texelColor = g_textureAtlas->Sample(textureCoordinates.x, textureCoordinates.y);

				*pixelColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };

				return true;
			}
			
			f_totalDistanceTravelled += f_distanceToTriangle;
		}

		return false;
	}

	float DistanceToEdge(Vec3D v_point, Vec3D v_vertex1, Vec3D v_vertex2)
	{
		Vec3D v_edgeDirection = SubtractVec3D(v_vertex2, v_vertex1);
		NormalizeVec3D(&v_edgeDirection);
		float f_edgeLength = Distance3D(v_vertex1, v_vertex2);

		float f_projectedPointOnEdgelength = DotProduct3D(SubtractVec3D(v_point, v_vertex1), v_edgeDirection);

		Vec3D v_closestPoint = VecScalarMultiplication3D(v_edgeDirection, Clamp(f_projectedPointOnEdgelength, 0, f_edgeLength));

		return Distance3D(v_point, v_closestPoint);
	}*/

	Vec3D CalculateLighting_PathTracing(Vec3D v_objectColor, float f_objectEmittance, float f_objectReflectance, Vec3D v_surfaceNormal, Vec3D v_start, int i_bounceCount)
	{
		Vec3D v_outgoingLightColor = VecScalarMultiplication3D(v_objectColor, f_objectEmittance);

		if (i_bounceCount > MAX_BOUNCES)
		{
			return v_outgoingLightColor;
		}

		float randX = int64_t(randEngine()) - int64_t(randEngine.max()) / 2;
		float randY = int64_t(randEngine()) - int64_t(randEngine.max()) / 2;
		float randZ = int64_t(randEngine()) - int64_t(randEngine.max()) / 2;

		Vec3D v_direction = ReturnNormalizedVec3D({ randX, randY, randZ });

		if (DotProduct3D(v_surfaceNormal, v_direction) < 0)
		{
			// The direction vector is in the wrong hemisphere, so we need to flip it into the other hemisphere
			ScaleVec3D(&v_direction, -1);
		}

		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_intersectionColor = ZERO_VEC3D;
		Vec3D v_normal = ZERO_VEC3D;

		// Checking for an intersection with any of the spheres

		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, &v_intersection, &v_intersectionColor, nullptr, &v_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_start, v_direction, v_intersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				Vec3D v_incomingLightColor = CalculateLighting_PathTracing(v_intersectionColor, g_spheres[i].emittance, g_spheres[i].reflectance, v_normal, v_intersection, i_bounceCount + 1);

				AddToVec3D(
					&v_outgoingLightColor,
					VecScalarMultiplication3D(v_incomingLightColor, 2 * f_objectReflectance * DotProduct3D(v_surfaceNormal, v_direction))
				);

				return v_outgoingLightColor;
			}
		}

		// Checking for an intersection with any of the triangles

		for (int i = 0; i < g_triangles.size(); i++)
		{
			bool intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, &v_intersection, &v_intersectionColor, nullptr, &v_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_start, v_direction, v_intersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				Vec3D v_incomingLightColor = CalculateLighting_PathTracing(v_intersectionColor, g_triangles[i].emittance, g_triangles[i].reflectance, v_normal, v_intersection, i_bounceCount + 1);

				AddToVec3D(
					&v_outgoingLightColor,
					VecScalarMultiplication3D(v_incomingLightColor, 2 * f_objectReflectance * DotProduct3D(v_surfaceNormal, v_direction))
				);

				return v_outgoingLightColor;
			}
		}

		// Checking for an intersection with the ground

		bool intersectionExists = GroundIntersection_RT(v_start, v_direction, &v_intersection, &v_intersectionColor);

		// The ground is always pointing upwards
		v_normal = { 0, 1, 0 };

		bool b_rayIsBlocked = false;

		if (intersectionExists)
		{
			b_rayIsBlocked = IsRayBlocked(v_start, v_direction, v_intersection);
		}

		if (intersectionExists && b_rayIsBlocked == false)
		{
			Vec3D v_incomingLightColor = CalculateLighting_PathTracing(v_intersectionColor, g_ground.emittance, g_ground.reflectance, v_normal, v_intersection, i_bounceCount + 1);

			AddToVec3D(
				&v_outgoingLightColor,
				VecScalarMultiplication3D(v_incomingLightColor, 2 * f_objectReflectance * DotProduct3D(v_surfaceNormal, v_direction))
			);

			return v_outgoingLightColor;
		}

		// Nothing was hit
		return v_outgoingLightColor;
	}

	bool IsRayBlocked(Vec3D v_start, Vec3D v_direction, Vec3D v_intersection)
	{
		Vec3D v_otherIntersection = ZERO_VEC3D;

		for (int j = 0; j < g_spheres.size(); j++)
		{
			bool otherIntersectionExists = SphereIntersection_RT(g_spheres[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		for (int j = 0; j < g_triangles.size(); j++)
		{
			bool otherIntersectionExists = TriangleIntersection_RT(g_triangles[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		bool otherIntersectionExists = GroundIntersection_RT(v_start, v_direction, &v_otherIntersection);

		// If there exists a closer intersection to the ray start vector it means the ray is blocked
		if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
		{
			return true;
		}

		// The ray is not blocked
		return false;
	}
};

int main()
{
	Engine rayTracer;
	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}

//#include "Controlls.h"