#include "hip/hip_runtime.h"
#define OLC_PGE_APPLICATION
#define RAY_TRACER

// Startup settings (cannot be changed during runtime)
#define ASYNC 1
#define THREAD_COUNT 4
#define SCREEN_WIDTH 900
#define SCREEN_HEIGHT 720
#define TOUCHING_DISTANCE 0.01f
#define OFFSET_DISTANCE 0.00001f
#define MAX_BOUNCES 5
#define SAMPLES_PER_PIXEL 1 // for path tracing
#define SAMPLES_PER_RAY 5 // for distribution ray tracing
#define WHITE_COLOR { 255, 255, 255 }
#define REFRACTION_INDEX_AIR 1

#include <iostream>
#include <random>
#include <future>

#include <hip/hip_runtime.h>
#include <>

#include "olcPixelGameEngine.h"

#include "MathUtilities.cuh"
#include "WorldDatatypes.h"
#include "ParseOBJ.h"

// Global variables

Player g_player;

std::vector<Sphere> g_spheres;
std::vector<Triangle> g_triangles;
std::vector<Light> g_lights;

Ground g_ground;

// Textures
olc::Sprite* g_basketball_texture;
olc::Sprite* g_planks_texture;
olc::Sprite* g_concrete_texture;
olc::Sprite* g_tiledfloor_texture;
olc::Sprite* g_worldmap_texture;
olc::Sprite* g_bricks_texture;

olc::Sprite* g_basketball_normalmap;
olc::Sprite* g_planks_normalmap;
olc::Sprite* g_concrete_normalmap;
olc::Sprite* g_tiledfloor_normalmap;
olc::Sprite* g_worldmap_normalmap;
olc::Sprite* g_bricks_normalmap;

std::random_device seedEngine;
std::uniform_real_distribution<> uniformDistribution(-1, 1);



// Ingame options (can be changed during runtime)
namespace Options
{
	bool mcControls = true;
}

class Engine : public olc::PixelGameEngine
{
public:
	Engine()
	{
		sAppName = "Ray_Tracing_Engine";
	}

	bool OnUserCreate() override
	{
		g_player = { { 1.5, 1.5, -2.064 }, { 1, ZERO_VEC3D }, TAU * 0.2f };

		g_basketball_texture = new olc::Sprite("../Assets/basketball.png");
		g_planks_texture = new olc::Sprite("../Assets/planks.png");
		g_concrete_texture = new olc::Sprite("../Assets/concrete.png");
		g_tiledfloor_texture = new olc::Sprite("../Assets/tiledfloor.png");
		g_worldmap_texture = new olc::Sprite("../Assets/worldmap.png");
		g_bricks_texture = new olc::Sprite("../Assets/bricks.png");

		g_basketball_normalmap = new olc::Sprite("../Assets/basketball_normalmap.png");
		g_planks_normalmap = new olc::Sprite("../Assets/planks_normalmap.png");
		g_concrete_normalmap = new olc::Sprite("../Assets/concrete_normalmap.png");
		g_tiledfloor_normalmap = new olc::Sprite("../Assets/tiledfloor_normalmap.png");
		g_worldmap_normalmap = new olc::Sprite("../Assets/tiledfloor_normalmap.png");
		g_bricks_normalmap = new olc::Sprite("../Assets/bricks_normalmap.png");

		g_spheres =
		{
			// Lightsource
			//{ { 1.5, 3, 1.5 }, 0.5, { { 45, 40, 30 }, { 0.9, 0.7, 0.1 }, { 0.9, 0.7, 0.1 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Glossy ball
			{ { 1.5, 1.4, 1.5 }, 0.4, { { 0, 0, 0 }, { 1, 1, 1 }, { 1, 1, 1 }, 0.2, 10, { 500, 500, 500 } } },
			// Other lightsource
			{ { 0.6, 0.3, 0.85 }, 0.3, { { 30, 5, 10 }, { 0.9, 0.2, 0.4 }, { 0.9, 0.2, 0.4 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Other lightsource
			{ { 1.9, 0.3, 0.5 }, 0.3, { { 2.25, 13.1, 18.7 }, { 0.9, 0.2, 0.4 }, { 0.9, 0.2, 0.4 }, 0.6, 1.6, { 500, 500, 500 } } },
			// Refractive ball
			//{ { 2.5, 0.5, 2.2 }, 0.5, { { 0, 0, 0 }, { 0.2, 0.2, 0.2 }, { 0.2, 0.2, 0.2 }, 0.3, 1.52, { 0, 0, 0 } } }
			// Refractive ball
			{ { 1.5, 2.3, 0.3 }, 0.5, { { 0, 0, 0 }, { 0.2, 0.2, 0.2 }, { 0.2, 0.2, 0.2 }, 0.3, 1.52, { 0, 0, 0 } } }
			// Basket ball
			//{ { 2.5, 0.5, 0.8 }, 0.5, { 1, 1, 1 }, { 0.2, 0.6, 0.8, 0.9, { -1, 0, 0 }, 500, 2 }, g_basketball_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ 1, 0, 1 }), PI / 2) },
			// World atlas globe
			//{ { 1.75, 0.3, 0.5 }, 0.3, { 1, 1, 1 }, { 0.35, 0.7, 0.7, 0.9, { 1, 0, 0 }, 500, 1.45 }, g_worldmap_texture, { 0, 0 }, { 1, 1 }, CreateRotationQuaternion(ReturnNormalizedVec3D({ -1, 0.5, -2 }), PI / 2) },
			// Magenta lightsource
			//{ { 0.5, 0.4, 0.8 }, 0.4, { 1, 0.2, 0.4157 }, { 35, 0.2, 0.5, 0.95, { -1, 0, 0 }, 500, 1.6 } },
			// Refractive ball
			//{ { 1.1, 0.3, 0.4 }, 0.3, { 1, 1, 1 }, { 0.2, 0.2, 0.2, 0.95, { 1, 0, 0 }, 0.5, 1.4 } },
			// Cyan lightsource
			//{ { 2.4, 0.3, 1.75 }, 0.3, { 0.3, 1.15, 1.15 }, { 45, 0.2, 0.5, 0.95, { 1, 0, 0 }, 500, 1.6 } }
		};

		g_triangles =
		{
			// Walls first face
			{ { { 0, 0, 3 }, { 0, 3, 3 }, { 3, 3, 3 } }, { { 0, 0, 0 }, { 0.3, 0.2, 0.2 }, { 0.3, 0.2, 0.2 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_bricks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_bricks_normalmap },
			{ { { 0, 0, 3 }, { 3, 3, 3 }, { 3, 0, 3 } }, { { 0, 0, 0 }, { 0.3, 0.2, 0.2 }, { 0.3, 0.2, 0.2 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_bricks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_bricks_normalmap },
			// Walls second face														   
			{ { { 0, 0, 0 }, { 0, 3, 0 }, { 0, 3, 3 } }, { { 0, 0, 0 }, { 0.2, 0.4, 0.4 }, { 0.2, 0.4, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap },
			{ { { 0, 0, 0 }, { 0, 3, 3 }, { 0, 0, 3 } }, { { 0, 0, 0 }, { 0.2, 0.4, 0.4 }, { 0.2, 0.4, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap },
			// Walls third face															   
			{ { { 3, 0, 3 }, { 3, 3, 3 }, { 3, 3, 0 } }, { { 0, 0, 0 }, { 0.4, 0.2, 0.4 }, { 0.4, 0.2, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap },
			{ { { 3, 0, 3 }, { 3, 3, 0 }, { 3, 0, 0 } }, { { 0, 0, 0 }, { 0.4, 0.2, 0.4 }, { 0.4, 0.2, 0.4 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap },
			// Walls fourth face														   
			{ { { 0, 3, 0 }, { 3, 3, 3 }, { 0, 3, 3 } }, { { 0, 0, 0 }, { 0.3, 0.3, 0.3 }, { 0.3, 0.3, 0.3 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_concrete_normalmap },
			{ { { 0, 3, 0 }, { 3, 3, 0 }, { 3, 3, 3 } }, { { 0, 0, 0 }, { 0.3, 0.3, 0.3 }, { 0.3, 0.3, 0.3 }, 0.975, 1.3, { 500, 500, 500 } }, "", g_concrete_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_concrete_normalmap },

			// Box first face															   
			{ { { 1, 0, 2 }, { 2, 1, 2 }, { 1, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 2 }, { 2, 0, 2 }, { 2, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box second face											  				     
			{ { { 1, 0, 1 }, { 1, 1, 1 }, { 2, 1, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 1 }, { 2, 1, 1 }, { 2, 0, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box third face											 				    
			{ { { 1, 0, 1 }, { 1, 1, 2 }, { 1, 1, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 0, 1 }, { 1, 0, 2 }, { 1, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box fourth face							   				  				     
			{ { { 2, 0, 1 }, { 2, 1, 1 }, { 2, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 2, 0, 1 }, { 2, 1, 2 }, { 2, 0, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },
			// Box fifth face							   				  				     
			{ { { 1, 1, 1 }, { 1, 1, 2 }, { 2, 1, 2 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 0, 0 }, { 1, 0 } }, g_planks_normalmap },
			{ { { 1, 1, 1 }, { 2, 1, 2 }, { 2, 1, 1 } }, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.9, 1.7, { 500, 500, 500 } }, "", g_planks_texture, { { 0, 1 }, { 1, 0 }, { 1, 1 } }, g_planks_normalmap },

			// refractive pyramid
			/*{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.1, 0 + 0.01, 2.9 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.1, 0 + 0.01, 2.9 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.1, 0 + 0.01, 2.1 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.9, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.1 - 0.7 }, { 0.5, 1.4 + 0.01, 2.5 - 0.7 }, { 0.9, 0 + 0.01, 2.9 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.1, 0 + 0.01, 2.9 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.52 } },
			{ { { 0.9, 0 + 0.01, 2.9 - 0.7 }, { 0.9, 0 + 0.01, 2.1 - 0.7 }, { 0.1, 0 + 0.01, 2.1 - 0.7 } }, { 1, 1, 1 }, { 0.25, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.52 } },

			// other refractive pyramid
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.1 + 2, 0 + 0.01, 2.9 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.1 + 2, 0 + 0.01, 2.9 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.1 + 2, 0 + 0.01, 2.1 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.9 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.1 }, { 0.5 + 2, 1.4 + 0.01, 2.5 }, { 0.9 + 2, 0 + 0.01, 2.9 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 0, 1, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.1 + 2, 0 + 0.01, 2.9 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.7 } },
			{ { { 0.9 + 2, 0 + 0.01, 2.9 }, { 0.9 + 2, 0 + 0.01, 2.1 }, { 0.1 + 2, 0 + 0.01, 2.1 } }, { 0.6, 0.6, 1.5 }, { 0.3, 0.4, 0.02, 0.95, { 1, 0, 0 }, 0, 1.7 } }*/
		};

		g_ground = { 0, { { 0, 0, 0 }, { 0.4, 0.4, 0.4 }, { 0.4, 0.4, 0.4 }, 0.6, 2, 500 }, g_tiledfloor_texture, { 0, 0 }, { 1, 1 }, 1, g_tiledfloor_normalmap };

		g_lights =
		{
			{ { 1.5, 3, 1.5 }, 0.5, 500, { 1, 0.8, 0.6 } }
		};

		for (int i = 0; i < g_lights.size(); i++)
			g_spheres.push_back({ g_lights[i].coords, g_lights[i].radius, g_lights[i].tint, { g_lights[i].emittance, 0, 0, 0, 0, 1, 0 } });

#if ASYNC == 1
	//std::async(std::launch::async, ImportScene, &g_triangles, "../Assets/RubberDuck.obj", 0.4, Vec3D({ 0.8, 0.5, 0.5 }));
#else
	//ImportScene(&g_triangles, "../Assets/RubberDuck.obj", 0.4, { 0.8, 0.5, 0.5 });
#endif

		return true;
	}

	bool OnUserUpdate(float fElapsedTime) override
	{
		Timer timer("Rendering");

		Controlls(fElapsedTime);

#if ASYNC == 1
		// Screen split up into 4 quadrants running in parallell on seperate threads

		std::future<void> returnValues[THREAD_COUNT];

		for (int i = 0; i < THREAD_COUNT; i++)
		{
			int startX = i * ceil(SCREEN_WIDTH / float(THREAD_COUNT));
			int endX = (i + 1) * ceil(SCREEN_WIDTH / float(THREAD_COUNT));

			if (startX >= SCREEN_WIDTH)
			{
				break;
			}

			endX = Min(endX, SCREEN_WIDTH);

			std::mt19937 randomEngine(seedEngine());

			returnValues[i] = std::async(std::launch::async, &Engine::RayTracing, this, startX, endX, randomEngine);
		}
#else
		RayTracing();
#endif
		//std::cout << "\a" << std::endl;

		return true;
	}

private:
	// Defined in Controlls.h
	void Controlls(float fElapsedTime);

	void RayTracing(int startX, int endX, std::mt19937 randomEngine)
	{
		const float zFar = (SCREEN_WIDTH * 0.5f) / tan(g_player.FOV * 0.5f);

		for (float y = -SCREEN_HEIGHT * 0.5f + 0.5f; y < SCREEN_HEIGHT * 0.5f + 0.5f; y++)
		{
			for (float x = -SCREEN_WIDTH * 0.5f + 0.5f + startX; x < -SCREEN_WIDTH * 0.5f + 0.5f + endX; x++)
			{
				Vec3D v_direction = { x, y, zFar };

				Vec3D v_orientedDirection = QuaternionMultiplication(g_player.q_orientation, { 0, v_direction }, QuaternionConjugate(g_player.q_orientation)).vecPart;

				int screenX = x + SCREEN_WIDTH * 0.5f;
				int screenY = SCREEN_HEIGHT - (y + SCREEN_HEIGHT * 0.5f);

				Vec3D pixelColor = ZERO_VEC3D;

				for (int i = 0; i < SAMPLES_PER_PIXEL; i++)
				{
					// For anti-aliasing
					Vec3D v_jitteredDirection = AddVec3D(v_orientedDirection, RandomVec_InUnitSphere(&randomEngine));

					NormalizeVec3D(&v_jitteredDirection);

					AddToVec3D(&pixelColor, RenderPixel(g_player.coords, v_jitteredDirection, &randomEngine));
				}

				ScaleVec3D(&pixelColor, 1 / float(SAMPLES_PER_PIXEL));

				pixelColor.x = Min(pixelColor.x, 255.0f);
				pixelColor.y = Min(pixelColor.y, 255.0f);
				pixelColor.z = Min(pixelColor.z, 255.0f);

				ScaleVec3D(&pixelColor, 1.0f / 255);

				pixelColor = { LINEAR_TO_SRGB(pixelColor.x), LINEAR_TO_SRGB(pixelColor.y), LINEAR_TO_SRGB(pixelColor.z) };

				ScaleVec3D(&pixelColor, 255.0f);

				Draw(screenX, screenY, { uint8_t(pixelColor.x), uint8_t(pixelColor.y), uint8_t(pixelColor.z) });
			}
#if PATH_TRACING == 1
			std::cout << ((y + SCREEN_HEIGHT * 0.5f) / SCREEN_HEIGHT) * 100 << "%" << std::endl;
#endif
		}
	}

	Vec3D RenderPixel(Vec3D v_start, Vec3D v_direction, std::mt19937* randomEngine)
	{
		Vec3D v_intersection = ZERO_VEC3D;
		Vec3D v_textureColor = ZERO_VEC3D;
		Quaternion q_surfaceNormal = IDENTITY_QUATERNION;
		Material material;

		bool intersectionExists = NextIntersection(v_start, v_direction, &v_intersection, &v_textureColor, &q_surfaceNormal, &material);

		if (intersectionExists)
		{
#if PATH_TRACING == 1
			v_textureColor = CalculateLighting_PathTracing(
				v_textureColor, material, q_surfaceNormal, v_direction, v_intersection, 0, randomEngine
			);
#else
			v_textureColor = CalculateLighting_DistributionTracing(
				v_textureColor, material, q_surfaceNormal.vecPart, v_direction, v_intersection, 0
			);
#endif
		}

		return v_textureColor;
	}

	float LINEAR_TO_SRGB(float l)
	{
		if (l <= 0.0031308)
		{
			return l * 12.92;
		}

		return 1.055 * pow(l, 0.41666) - 0.055;
	}

	bool GroundIntersection_RT(Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		if (v_direction.y >= 0 || v_start.y < g_ground.level)
		{
			return false;
		}

		ScaleVec3D(&v_direction, (g_ground.level - v_start.y) / v_direction.y);

		Vec3D rayGroundIntersection = AddVec3D(v_start, v_direction);

		if (v_intersection != nullptr)
		{
			*v_intersection = rayGroundIntersection;
		}

		if (q_surfaceNormal != nullptr)
		{
			*q_surfaceNormal = { 1, { 0, 1, 0 } };
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (g_ground.texture != nullptr || g_ground.normalMap != nullptr)
		{
			float signedTextureWidth = (g_ground.textureCorner2.x - g_ground.textureCorner1.x) * g_ground.textureScalar;
			float signedTextureHeight = (g_ground.textureCorner2.y - g_ground.textureCorner1.y) * g_ground.textureScalar;

			float t1 = fmod(rayGroundIntersection.x, signedTextureWidth) / signedTextureWidth;
			float t2 = fmod(rayGroundIntersection.z, signedTextureHeight) / signedTextureHeight;

			// if the t values are negative, we need to flip them around the center of the texture and make them positive
			if (t1 < 0) t1 += 1;
			if (t2 < 0) t2 += 1;

			float textureX = Lerp(g_ground.textureCorner1.x, g_ground.textureCorner2.x, t1);
			float textureY = Lerp(g_ground.textureCorner1.y, g_ground.textureCorner2.y, t2);

			if (g_ground.texture != nullptr)
			{
				olc::Pixel texelColor = g_ground.texture->Sample(textureX, textureY);

				*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };
			}
			if (g_ground.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = g_ground.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				q_surfaceNormal->vecPart = ReturnNormalizedVec3D({ float(normalMapColor.r) * 2 - 255.0f, float(normalMapColor.b) * 2 - 255.0f, float(normalMapColor.g) * 2 - 255.0f });
			}
		}

		return true;
	}

	// Ray tracing for spheres
	bool SphereIntersection_RT(Sphere sphere, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		float dxdz = v_direction.x / v_direction.z;
		float dydz = v_direction.y / v_direction.z;

		float a = dxdz * dxdz + dydz * dydz + 1;
		
		float b = 
			2 * dxdz * (v_start.x - sphere.coords.x) +
			2 * dydz * (v_start.y - sphere.coords.y) +
			2 * (v_start.z - sphere.coords.z);

		float c = 
			(v_start.x - sphere.coords.x) * (v_start.x - sphere.coords.x) +
			(v_start.y - sphere.coords.y) * (v_start.y - sphere.coords.y) +
			(v_start.z - sphere.coords.z) * (v_start.z - sphere.coords.z) - sphere.radius * sphere.radius;

		// ISAK: There wasn't any need to recalculate this multiple times
		float rootContent = b * b - 4 * a * c;

		// There exists no intersections (no real answer)
		if (rootContent < 0) return false;

		float z1 = (-b + sqrt(rootContent)) / (2 * a);
		float z2 = (-b - sqrt(rootContent)) / (2 * a);

		Vec3D v_alternative1 = { z1 * dxdz, z1 * dydz, z1 };
		AddToVec3D(&v_alternative1, v_start);

		Vec3D v_alternative2 = { z2 * dxdz, z2 * dydz, z2 };
		AddToVec3D(&v_alternative2, v_start);

		// Check which intersection is the closest and choose that one
		float dist1 = DistanceSquared3D(v_alternative1, v_start);
		float dist2 = DistanceSquared3D(v_alternative2, v_start);

		bool dist1Closest = dist1 < dist2;

		Vec3D v_correctHit = dist1Closest ? v_alternative1 : v_alternative2;
		Vec3D v_otherHit = dist1Closest ? v_alternative2 : v_alternative1;

		// Check if the intersection is behind the ray. If so, choose the other one.
		if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0)
		{
			v_correctHit = v_otherHit;

			// Check if the other intersection is behind the ray. If so, discard it.
			if (DotProduct3D(SubtractVec3D(v_correctHit, v_start), v_direction) < 0) return false;
		}

		// Checks whether or not to return the intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_correctHit;
		}

		// Calculating the normal of the sphere (without normalmap)
		Vec3D v_normal = SubtractVec3D(v_correctHit, sphere.coords);
		NormalizeVec3D(&v_normal);

		if (q_surfaceNormal != nullptr)
		{
			q_surfaceNormal->vecPart = v_normal;
			q_surfaceNormal->realPart = 1;

			if (DistanceSquared3D(v_start, sphere.coords) < sphere.radius * sphere.radius)
			{
				q_surfaceNormal->realPart = -1;
			}
		}

		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (sphere.texture != nullptr || sphere.normalMap != nullptr)
		{
			Vec3D i_Hat = { 1, 0, 0 };
			Vec3D j_Hat = { 0, 1, 0 };
			Vec3D k_Hat = { 0, 0, 1 };

			// Rotating axies by sphere rotation quaternion
			i_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, i_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			j_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, j_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;
			k_Hat = QuaternionMultiplication(sphere.rotQuaternion, { 0, k_Hat }, QuaternionConjugate(sphere.rotQuaternion)).vecPart;

			// Translate normal into new coordinate system
			v_normal = { DotProduct3D(v_normal, i_Hat), DotProduct3D(v_normal, j_Hat), DotProduct3D(v_normal, k_Hat) };

			// UV coordinates
			float u = 0.5 + atan2(v_normal.x, v_normal.z) / TAU;
			float v = 0.5 - asin(v_normal.y) / PI;

			float textureX = Lerp(sphere.textureCorner1.x, sphere.textureCorner2.x, u);
			float textureY = Lerp(sphere.textureCorner1.y, sphere.textureCorner2.y, v);

			if (sphere.texture != nullptr)
			{
				// Interpolating between assigned texture coordinates
				olc::Pixel texelColor = sphere.texture->Sample(textureX, textureY);

				*v_intersectionColor = { (float)texelColor.r, (float)texelColor.g, (float)texelColor.b };
			}
			if (sphere.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = sphere.normalMap->Sample(textureX, textureY);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ float(normalMapColor.r) * 2 - 255.0f, float(normalMapColor.b) * 2 - 255.0f, float(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the sphere
				Vec3D v_sidewaysTangent = ReturnNormalizedVec3D({ -v_normal.z, 0, v_normal.x });
				Vec3D v_forwardTangent = ReturnNormalizedVec3D({ 0, -v_normal.z, v_normal.y });

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					v_sidewaysTangent,
					v_normal,
					v_forwardTangent
				};

				q_surfaceNormal->vecPart = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		return true;
	}

	// Ray tracing for triangles
	bool TriangleIntersection_RT(Triangle triangle, Vec3D v_start, Vec3D v_direction,
		Vec3D* v_intersection = nullptr, Vec3D* v_intersectionColor = nullptr, Quaternion* q_surfaceNormal = nullptr)
	{
		Vec3D v_triangleEdge1 = SubtractVec3D(triangle.vertices[1], triangle.vertices[0]);
		Vec3D v_triangleEdge2 = SubtractVec3D(triangle.vertices[2], triangle.vertices[0]);

		Vec3D v_triangleNormal = CrossProduct(v_triangleEdge1, v_triangleEdge2);

		NormalizeVec3D(&v_triangleNormal);

		// how much the plane is offseted in the direction of the planeNormal
		// a negative value means it's offseted in the opposite direction of the planeNormal
		float f_trianglePlaneOffset = DotProduct3D(v_triangleNormal, triangle.vertices[0]);

		Vec3D v_trianglePlaneIntersection = LinePlaneIntersection(v_start, v_direction, v_triangleNormal, f_trianglePlaneOffset);

		if (DotProduct3D(SubtractVec3D(v_trianglePlaneIntersection, v_start), v_direction) < 0) return false;

		// these normals aren't actually normalized, but that doesn't matter for this use-case
		Vec3D v_triangleEdge1_normal = CrossProduct(SubtractVec3D(triangle.vertices[1], triangle.vertices[0]), v_triangleNormal);
		Vec3D v_triangleEdge2_normal = CrossProduct(SubtractVec3D(triangle.vertices[2], triangle.vertices[1]), v_triangleNormal);
		Vec3D v_triangleEdge3_normal = CrossProduct(SubtractVec3D(triangle.vertices[0], triangle.vertices[2]), v_triangleNormal);

		// check if the intersection is outside of the triangle
		if ((DotProduct3D(v_triangleEdge1_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[1])) > 0) ||
			(DotProduct3D(v_triangleEdge2_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[2])) > 0) ||
			(DotProduct3D(v_triangleEdge3_normal, SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0])) > 0))
		{
			return false;
		}

		// Checks whether or not to return the intersection
		if (v_intersection != nullptr)
		{
			*v_intersection = v_trianglePlaneIntersection;
		}

		if (q_surfaceNormal != nullptr)
		{
			q_surfaceNormal->vecPart = v_triangleNormal;

			q_surfaceNormal->realPart = 1;

			if (DotProduct3D(v_triangleNormal, v_direction) > 0)
			{
				// The triangle face is inside of the mesh, so the normal must be flipped
				q_surfaceNormal->realPart = -1;
			}
		}
		
		if (v_intersectionColor == nullptr)
		{
			// Don't return any color
			return true;
		}

		*v_intersectionColor = WHITE_COLOR;

		if (triangle.texture != nullptr || triangle.normalMap != nullptr)
		{
			// from here on we calculate the texture coordinates

			Vec3D v_intersectionRelativeToTriangle = SubtractVec3D(v_trianglePlaneIntersection, triangle.vertices[0]);

			Matrix3D triangleMatrix =
			{
				v_triangleEdge1,
				v_triangleEdge2,
				v_triangleNormal
			};

			Vec3D triangleEdgeScalars = VecMatrixMultiplication3D(v_intersectionRelativeToTriangle, InverseMatrix3D(triangleMatrix));

			Vec2D textureCoordinates = { 0, 0 };

			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[1], triangle.textureVertices[0]), triangleEdgeScalars.x));
			AddToVec2D(&textureCoordinates, VecScalarMultiplication2D(SubtractVec2D(triangle.textureVertices[2], triangle.textureVertices[0]), triangleEdgeScalars.y));
			AddToVec2D(&textureCoordinates, triangle.textureVertices[0]);

			if (triangle.texture != nullptr)
			{
				olc::Pixel texelColor = triangle.texture->Sample(textureCoordinates.x, textureCoordinates.y);

				*v_intersectionColor = { float(texelColor.r), float(texelColor.g), float(texelColor.b) };
			}
			if (triangle.normalMap != nullptr)
			{
				olc::Pixel normalMapColor = triangle.normalMap->Sample(textureCoordinates.x, textureCoordinates.y);

				// Converting the color in the normalMap to an actual unit vector
				Vec3D v_normalMapNormal = ReturnNormalizedVec3D({ float(normalMapColor.r) * 2 - 255.0f, float(normalMapColor.b) * 2 - 255.0f, float(normalMapColor.g) * 2 - 255.0f });

				// Calculating tangents of the triangle for finding the normal in object space

				// { u1, v1 }, { u2, v2 }, { u3, v3 } are the normalMapVertices
				// T is the tangent
				// B is the bitangent
				
				//                       | T.x  B.x  0 |   
				// { v_triangleEdge1 } = | T.y  B.y  0 | * { u2 - u1, v2 - v1, 0 }
				//                       | T.z  B.z  0 |   

				//                       | T.x  B.x  0 |   
				// { v_triangleEdge2 } = | T.y  B.y  0 | * { u3 - u1, v3 - v1, 0 }
				//                       | T.z  B.z  0 |   

				// | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | T.x  B.x  0 |   | u2 - u1  u3 - u1  0 |
				// | v_triangleEdge1.y  v_triangleEdge2.y  0 | = | T.y  B.y  0 | * | v2 - v1  v3 - v1  0 |
				// | v_triangleEdge1.z  v_triangleEdge2.z  0 |   | T.z  B.z  0 |   |    0        0     1 |

				//                                                                                       -1
				// | T.x  B.x  0 |   | v_triangleEdge1.x  v_triangleEdge2.x  0 |   | u2 - u1  u3 - u1  0 |
				// | T.y  B.y  0 | = | v_triangleEdge1.y  v_triangleEdge2.y  0 | * | v2 - v1  v3 - v1  0 |
				// | T.z  B.z  0 |	 | v_triangleEdge1.z  v_triangleEdge2.z  0 |   |    0        0     1 |

				Matrix3D m1 =
				{
					v_triangleEdge1,
					v_triangleEdge2,
					ZERO_VEC3D
				};

				Matrix3D m2 =
				{
					{ triangle.textureVertices[1].x - triangle.textureVertices[0].x, triangle.textureVertices[1].y - triangle.textureVertices[0].y, 0 },
					{ triangle.textureVertices[2].x - triangle.textureVertices[0].x, triangle.textureVertices[2].y - triangle.textureVertices[0].y, 0 },
					{ 0, 0, 1 }
				};

				Matrix3D tangentsMatrix = MatrixMultiplication3D(InverseMatrix3D(m2), m1);

				// Takes the normal in the normalMap and transforms it into the actual normal of the object
				Matrix3D normalMatrix =
				{
					ReturnNormalizedVec3D(tangentsMatrix.i_Hat),
					v_triangleNormal,
					ReturnNormalizedVec3D(tangentsMatrix.j_Hat)
				};

				q_surfaceNormal->vecPart = VecMatrixMultiplication3D(v_normalMapNormal, normalMatrix);
			}
		}
		
		return true;
	}

	Vec3D LinePlaneIntersection(Vec3D v_start, Vec3D v_direction, Vec3D v_planeNormal, float f_planeOffset)
	{
		float f_deltaOffset = DotProduct3D(v_start, v_planeNormal);

		f_planeOffset -= f_deltaOffset;

		float f_scalingFactor = f_planeOffset / DotProduct3D(v_direction, v_planeNormal);

		return AddVec3D(VecScalarMultiplication3D(v_direction, f_scalingFactor), v_start);
	}

	Vec3D CalculateLighting_PathTracing(Vec3D v_textureColor, Material material, Quaternion q_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, int i_bounceCount, std::mt19937* randomEngine)
	{
		Vec3D v_outgoingLightColor = ConusProduct(v_textureColor, material.emittance);

		float refractionIndex1 = REFRACTION_INDEX_AIR;
		float refractionIndex2 = material.refractionIndex;
		Vec3D attenuation = { 0, 0, 0 };

		if (q_surfaceNormal.realPart == -1)
		{
			refractionIndex1 = material.refractionIndex;
			refractionIndex2 = REFRACTION_INDEX_AIR;
			v_outgoingLightColor = ZERO_VEC3D;
		}

		if (i_bounceCount > MAX_BOUNCES)
		{
			return v_outgoingLightColor;
		}

		Vec3D v_outgoingDirection = ReturnNormalizedVec3D(RandomVec_InUnitSphere(randomEngine));

		AddToVec3D(&v_intersection, VecScalarMultiplication3D(v_outgoingDirection, OFFSET_DISTANCE));

		if (DotProduct3D(v_outgoingDirection, q_surfaceNormal.vecPart) < 0)
		{
			// The ray is going into the object
			attenuation = material.attenuation;
		}

		// Scale the normal to be oriented in the hemisphere the material was hit from
		ScaleVec3D(&(q_surfaceNormal.vecPart), q_surfaceNormal.realPart);

		Vec3D v_nextIntersection = ZERO_VEC3D;
		Vec3D v_nextTextureColor = ZERO_VEC3D;
		Quaternion q_nextNormal = IDENTITY_QUATERNION;
		Material nextMaterial;

		bool intersectionExists = NextIntersection(v_intersection, v_outgoingDirection, &v_nextIntersection, &v_nextTextureColor, &q_nextNormal, &nextMaterial);

		if (intersectionExists)
		{
			Vec3D v_incomingLightColor = CalculateLighting_PathTracing(
				v_nextTextureColor, nextMaterial, q_nextNormal, v_outgoingDirection, v_nextIntersection, i_bounceCount + 1, randomEngine
			);

			Vec3D v_diffuseTint = VecScalarMultiplication3D(ConusProduct(v_textureColor, material.diffuseTint), 1.0f / 255);
			Vec3D v_specularTint = VecScalarMultiplication3D(ConusProduct(v_textureColor, material.specularTint), 1.0f / 255);

			float distance = Distance3D(v_intersection, v_nextIntersection);

			attenuation = { exp(-attenuation.x * distance), exp(-attenuation.y * distance), exp(-attenuation.z * distance) };

			v_incomingLightColor = ConusProduct(v_incomingLightColor, attenuation);

			//reflection
			AddToVec3D(
				&v_outgoingLightColor,
				VecScalarMultiplication3D(
					ConusProduct(v_incomingLightColor, BRDF(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, refractionIndex1, refractionIndex2, material.roughness, v_diffuseTint, v_specularTint)),
					DotProduct3D(v_outgoingDirection, q_surfaceNormal.vecPart) * 2 * TAU
				)
			);
			
			//transmission
			AddToVec3D(
				&v_outgoingLightColor,
				VecScalarMultiplication3D(
					v_incomingLightColor, BTDF(v_incomingDirection, v_outgoingDirection, q_surfaceNormal.vecPart, refractionIndex1, refractionIndex2, material.roughness) * Abs(DotProduct3D(v_outgoingDirection, q_surfaceNormal.vecPart)) * 2 * TAU
				)
			);
		}

		return v_outgoingLightColor;
	}

	bool NextIntersection(Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection, Vec3D* v_textureColor, Quaternion* q_normal, Material* material)
	{
		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool intersectionExists = SphereIntersection_RT(g_spheres[i], v_start, v_direction, v_intersection, v_textureColor, q_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_start, v_direction, *v_intersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				*material = g_spheres[i].material;

				return true;
			}
		}

		for (int i = 0; i < g_triangles.size(); i++)
		{
			bool intersectionExists = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, v_intersection, v_textureColor, q_normal);

			bool b_rayIsBlocked = false;

			if (intersectionExists)
			{
				b_rayIsBlocked = IsRayBlocked(v_start, v_direction, *v_intersection);
			}

			if (intersectionExists && b_rayIsBlocked == false)
			{
				*material = g_triangles[i].material;

				return true;
			}
		}

		bool intersectionExists = GroundIntersection_RT(v_start, v_direction, v_intersection, v_textureColor, q_normal);

		if (intersectionExists)
		{
			*material = g_ground.material;

			return true;
		}

		return false;
	}

	bool IsRayBlocked(Vec3D v_start, Vec3D v_direction, Vec3D v_intersection)
	{
		Vec3D v_otherIntersection = ZERO_VEC3D;

		for (int j = 0; j < g_spheres.size(); j++)
		{
			bool otherIntersectionExists = SphereIntersection_RT(g_spheres[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		for (int j = 0; j < g_triangles.size(); j++)
		{
			bool otherIntersectionExists = TriangleIntersection_RT(g_triangles[j], v_start, v_direction, &v_otherIntersection);

			// If there exists a closer intersection to the ray start vector it means the ray is blocked
			if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
			{
				return true;
			}
		}

		bool otherIntersectionExists = GroundIntersection_RT(v_start, v_direction, &v_otherIntersection);

		// If there exists a closer intersection to the ray start vector it means the ray is blocked
		if (otherIntersectionExists && DistanceSquared3D(v_start, v_otherIntersection) < DistanceSquared3D(v_start, v_intersection))
		{
			return true;
		}

		// The ray is not blocked
		return false;
	}

	// Cook-Torrance BRDF with GGX distribution function and GGX geometry function
	Vec3D BRDF(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, float refractionIndex1, float refractionIndex2, float roughness, Vec3D v_diffuseTint, Vec3D v_specularTint)
	{
		ScaleVec3D(&v_incomingDirection, -1);

		Vec3D v_bisectorVector = ReturnNormalizedVec3D(Lerp3D(v_incomingDirection, v_outgoingDirection, 0.5));

		float fresnelFactor = Fresnel(v_incomingDirection, v_bisectorVector, refractionIndex1, refractionIndex2);

		float diffuseTerm = Chi(DotProduct3D(v_bisectorVector, v_normal)) * (1 - fresnelFactor) / PI;

		float specularTerm = fresnelFactor * GeometryBidirectional(v_incomingDirection, v_outgoingDirection, v_normal, v_bisectorVector, roughness) * Distribution(v_normal, v_bisectorVector, roughness) /
			(4 * DotProduct3D(v_incomingDirection, v_normal) * DotProduct3D(v_outgoingDirection, v_normal));

		return AddVec3D(VecScalarMultiplication3D(v_diffuseTint, diffuseTerm), VecScalarMultiplication3D(v_specularTint, specularTerm));
	}

	float Chi(float x)
	{
		return x > 0 ? 1 : 0;
	}

	float Distribution(Vec3D v_normal, Vec3D v_bisectorVector, float roughness)
	{
		float bisectDotNormal = DotProduct3D(v_bisectorVector, v_normal);
		float bisectDotNormal2 = bisectDotNormal * bisectDotNormal;
		float roughness2 = roughness * roughness;

		return (Chi(DotProduct3D(v_bisectorVector, v_normal)) * roughness2) / (PI * Square(bisectDotNormal2 * (roughness2 + (1 - bisectDotNormal2) / bisectDotNormal2)));
	}

	float Fresnel(Vec3D v_incomingDirection, Vec3D v_bisectorVector, float refractionIndex1, float refractionIndex2)
	{
		float c = DotProduct3D(v_incomingDirection, v_bisectorVector);

		float g = sqrt(Max((refractionIndex2 * refractionIndex2) / (refractionIndex1 * refractionIndex1) - 1 + c * c, 0));

		return 0.5 * (Square(g - c) / Square(g + c)) * (1 + Square(c * (g + c) - 1) / Square(c * (g - c) + 1));
	}

	float GeometryBidirectional(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, Vec3D v_bisectorVector, float roughness)
	{
		return GeometryMonodirectional(v_incomingDirection, v_normal, v_bisectorVector, roughness) * GeometryMonodirectional(v_outgoingDirection, v_normal, v_bisectorVector, roughness);
	}

	float GeometryMonodirectional(Vec3D vec, Vec3D v_normal, Vec3D v_bisectorVector, float roughness)
	{
		float VecDotNormal = DotProduct3D(vec, v_normal);
		float VecDotNormal2 = VecDotNormal * VecDotNormal;
		float a = 1.0f / (roughness * sqrt(1 - VecDotNormal2) / VecDotNormal);
		float a2 = a * a;

		return Chi(DotProduct3D(vec, v_bisectorVector) / DotProduct3D(vec, v_normal)) * (a < 1.59 ? (3.535 * a + 2.181 * a2) / (1 + 2.276 * a + 2.577 * a2) : 1);
	}

	float BTDF(Vec3D v_incomingDirection, Vec3D v_outgoingDirection, Vec3D v_normal, float refractionIndex1, float refractionIndex2, float roughness)
	{
		ScaleVec3D(&v_normal, -1);

		Vec3D v_bisectorVector = ReturnNormalizedVec3D(Lerp3D(v_incomingDirection, v_outgoingDirection, 0.5));

		float incomingDotBisector = DotProduct3D(v_incomingDirection, v_bisectorVector);
		float outgoingDotBisector = DotProduct3D(v_outgoingDirection, v_bisectorVector);

		return (incomingDotBisector * outgoingDotBisector) / (DotProduct3D(v_incomingDirection, v_normal) * DotProduct3D(v_outgoingDirection, v_normal)) *
			(refractionIndex2 * refractionIndex2 * (1 - Fresnel(v_incomingDirection, v_bisectorVector, refractionIndex1, refractionIndex2)) * GeometryBidirectional(v_incomingDirection, v_outgoingDirection, v_normal, v_bisectorVector, roughness) *
				Distribution(v_normal, v_bisectorVector, roughness)) / Square(refractionIndex1 * incomingDotBisector + refractionIndex2 * outgoingDotBisector);
	}

	Vec3D CalculateLighting_DistributionTracing(Vec3D v_objectColor, Material material, Vec3D v_surfaceNormal, Vec3D v_incomingDirection, Vec3D v_intersection, int i_bounceCount)
	{
		Vec3D v_pixelColor = ZERO_VEC3D;

		if (i_bounceCount > MAX_BOUNCES)
			return v_pixelColor;

		// Temporary until refraction (it'll need to decide whether to offset in or out)
		AddToVec3D(&v_intersection, VecScalarMultiplication3D(v_surfaceNormal, OFFSET_DISTANCE));

		// Soft shadows
		for (int i = 0; i < g_lights.size(); i++)
		{
			float notBlockedProportion = 0;

			for (int j = 0; j < SAMPLES_PER_RAY; j++)
			{
				Vec3D v_displacement = ReturnNormalizedVec3D(RandomVec_InUnitSphere());
				v_displacement = VecScalarMultiplication3D(v_displacement, g_lights[i].radius);
				Vec3D randomPointLight = AddVec3D(g_lights[i].coords, v_displacement);

				Vec3D v_newDirection = ReturnNormalizedVec3D(SubtractVec3D(randomPointLight, v_intersection));

				notBlockedProportion += !IsRayBlocked(v_intersection, v_newDirection, g_lights[i].coords);
			}

			notBlockedProportion /= SAMPLES_PER_RAY;

			float distance = Distance3D(v_intersection, g_lights[i].coords) - g_lights[i].radius;

			v_objectColor = VecScalarMultiplication3D(v_objectColor, 1 + material.emittance);
			Vec3D v_lightColor = VecScalarMultiplication3D(g_lights[i].tint, g_lights[i].emittance);

			// (objectColor + lightColor) * notBlockedProportion / (distance ^ 2)
			Vec3D v_shading = VecScalarMultiplication3D(VecScalarMultiplication3D(AddVec3D(v_objectColor, v_lightColor), notBlockedProportion), 1 / (distance * distance));

			AddToVec3D(&v_pixelColor, v_shading);
		}


		// Reflections
		Vec3D v_specularDirecion = SubtractVec3D(v_incomingDirection, VecScalarMultiplication3D(v_surfaceNormal, 2 * DotProduct3D(v_incomingDirection, v_surfaceNormal)));

		Vec3D v_reflectionIntersection, v_reflectionColor;
		Quaternion q_reflectionIntersectionNormal;
		Material newMaterial;

		if (material.reflectiveRoughness == 0)
		{
			// Specular reflections
			bool b_foundIntersection = FindIntersection(v_intersection, v_specularDirecion, &v_reflectionIntersection, &v_reflectionColor, &q_reflectionIntersectionNormal, &newMaterial);

			if (b_foundIntersection)
			{
				v_reflectionColor = CalculateLighting_DistributionTracing(
					v_reflectionColor, newMaterial, q_reflectionIntersectionNormal.vecPart, v_specularDirecion, v_intersection, i_bounceCount + 1
				);
			}
		}
		else
		{
			// Diffuse reflections
			int hitCount = 0;

			for (int i = 0; i < SAMPLES_PER_RAY; i++)
			{
				Vec3D v_lambertianDirection = ReturnNormalizedVec3D(RandomVec_InUnitSphere());
				if (DotProduct3D(v_lambertianDirection, v_surfaceNormal) < 0)
					v_lambertianDirection = VecScalarMultiplication3D(v_lambertianDirection, -1);

				Vec3D v_diffuseDirection = Lerp3D(v_specularDirecion, v_lambertianDirection, material.reflectiveRoughness);

				bool b_foundIntersection = FindIntersection(v_intersection, v_diffuseDirection, &v_reflectionIntersection, &v_reflectionColor, &q_reflectionIntersectionNormal, &newMaterial);

				if (b_foundIntersection)
				{
					AddToVec3D(&v_reflectionColor, CalculateLighting_DistributionTracing(
						v_reflectionColor, newMaterial, q_reflectionIntersectionNormal.vecPart, v_diffuseDirection, v_intersection, i_bounceCount + 1
					));
					hitCount++;
				}
			}

			if (hitCount > 0) v_reflectionColor = VecScalarMultiplication3D(v_reflectionColor, 1 / hitCount);
		}


		// Refraction
		// Tangent inside of the plane defined by v_surfaceNormal and v_incomingDirection
		Vec3D v_surfaceTangent = CrossProduct(ReturnNormalizedVec3D(CrossProduct(v_surfaceNormal, v_incomingDirection)), v_surfaceNormal);

		float sinIncomingAngle = DotProduct3D(v_incomingDirection, v_surfaceTangent);

		float sinRefractedAngle = Min(REFRACTION_INDEX_AIR * sinIncomingAngle / material.refractionIndex, 1.0f);

		float cosRefractedAngle = sqrt(1 - sinRefractedAngle * sinRefractedAngle); // Pythagorean identity

		//bool b_foundIntersection = FindIntersection(v_intersection, v_reflectedDirecion, &v_reflectionIntersection, &v_reflectionColor, &q_reflectionIntersectionNormal, &newMaterial);

		//if (b_foundIntersection)
		//	v_reflectionColor = CalculateLighting_DistributionTracing(
		//		v_reflectionColor, newMaterial, q_reflectionIntersectionNormal.vecPart, v_reflectedDirecion, v_intersection, ++i_bounceCount
		//	);


		// Fresnel for weighing reflection and refraction color
		float cosIncomingAngle = -DotProduct3D(v_incomingDirection, v_surfaceNormal);

		// Average of the s-polarized reflectance and p-polarized reflectance probabilities
		float fresnel = (
			Square((REFRACTION_INDEX_AIR * cosIncomingAngle - material.refractionIndex * cosRefractedAngle) / (REFRACTION_INDEX_AIR * cosIncomingAngle + material.refractionIndex * cosRefractedAngle)) +
			Square((REFRACTION_INDEX_AIR * cosRefractedAngle - material.refractionIndex * cosIncomingAngle) / (REFRACTION_INDEX_AIR * cosRefractedAngle + material.refractionIndex * cosIncomingAngle))
		) * 0.5f;

		float reflectance = Lerp(material.minReflectance, material.maxReflectance, fresnel);

		 //((1 - reflectance) * v_objectColor + reflectance * v_reflectionColor) / 2
		v_pixelColor = VecScalarMultiplication3D(AddVec3D(VecScalarMultiplication3D(v_objectColor, (1 - reflectance)), VecScalarMultiplication3D(v_reflectionColor, reflectance)), 0.5f);
		//v_pixelColor = VecScalarMultiplication3D(AddVec3D(v_objectColor, v_reflectionColor), 0.5f);

		return v_pixelColor;
	}

	bool FindIntersection(Vec3D v_start, Vec3D v_direction, Vec3D* v_intersection, Vec3D* v_color, Quaternion* q_normal, Material* material)
	{
		// Check ground
		bool groundIntersect = GroundIntersection_RT(v_start, v_direction, v_intersection, v_color, q_normal);

		if (groundIntersect && !IsRayBlocked(v_start, v_direction, *v_intersection))
		{
			*material = g_ground.material;
			return true;
		}

		// Check all spheres
		for (int i = 0; i < g_spheres.size(); i++)
		{
			bool sphereIntersect = SphereIntersection_RT(g_spheres[i], v_start, v_direction, v_intersection, v_color, q_normal);

			if (sphereIntersect && !IsRayBlocked(v_start, v_direction, *v_intersection))
			{
				*material = g_spheres[i].material;
				return true;
			}
		}

		// Check all triangles
		for (int i = 0; i < g_triangles.size(); i++)
		{
			bool triangleIntersect = TriangleIntersection_RT(g_triangles[i], v_start, v_direction, v_intersection, v_color, q_normal);

			if (triangleIntersect && !IsRayBlocked(v_start, v_direction, *v_intersection))
			{
				*material = g_triangles[i].material;
				return true;
			}
		}

		return pixelColor;
	}*/

	Vec3D RandomVec_InUnitSphere(std::mt19937* randomEngine)
	{
		Vec3D randPoint;

		do
		{
			float randX = uniformDistribution(*randomEngine);
			float randY = uniformDistribution(*randomEngine);
			float randZ = uniformDistribution(*randomEngine);

			randPoint = { randX, randY, randZ };
		} while (VecLengthSquared(randPoint) > 1);

		return randPoint;
	}
};

int main()
{
	Engine rayTracer;
	if (rayTracer.Construct(SCREEN_WIDTH, SCREEN_HEIGHT, 1, 1))
		rayTracer.Start();
	return 0;
}

#include "Controlls.h"